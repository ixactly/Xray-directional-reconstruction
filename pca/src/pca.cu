#include "hip/hip_runtime.h"
//
// Created by tomokimori on 22/10/24.
//

#include "pca.cuh"
#include "Params.h"
#include <Eigen/Dense>

void calcEigenVector(const Volume<float> *ct, Volume<float> *md, int x, int y, int z) {

    Eigen::Matrix3f varMatrix;
    varMatrix << 0, 0, 0,
            0, 0, 0,
            0, 0, 0;
    // calclate VarianceCovariance Matrix
    float mu_mean = 0.0f;
    for (int i = 0; i < NUM_BASIS_VECTOR; i++) {
        float mu = (ct[i])(x, y, z);
        mu_mean += mu;
        Eigen::Matrix<float, 3, 1> scat(mu * basisVector[3 * i + 0], mu * basisVector[3 * i + 1],
                                         mu * basisVector[3 * i + 2]);
        varMatrix += scat * scat.transpose();
    }

    mu_mean /= static_cast<float>(NUM_BASIS_VECTOR);
    varMatrix /= static_cast<float>(NUM_BASIS_VECTOR);

    Eigen::SelfAdjointEigenSolver<Eigen::Matrix3f> ES(varMatrix);

    // Eigen::Vector3f values = ES.eigenvalues();
    Eigen::Matrix3f vectors = ES.eigenvectors();

    // (temporary) pick up minimum eigenvector, then normalization
    Eigen::Vector3f min = vectors.col(0).normalized();

    // md norm -> need elipsoid volume
    md[0](x, y, z) = mu_mean * min.x();
    md[1](x, y, z) = mu_mean * min.y();
    md[2](x, y, z) = mu_mean * min.z();
    // std::cout << md[0](x, y, z) << " " << md[1](x, y, z) << " " << md[2](x, y, z) << std::endl;
    // std::cout << varMatrix << std::endl;
}
