//
// Created by tomokimori on 23/08/17.
//

#include <json.h>
#include <params.h>
#include <iostream>
#include <string>
#include <fstream>

// global variables are now not constant due to implement problem
using json = nlohmann::json;

std::string PROJ_PATH;
std::string VOLUME_PATH;
std::string DIRECTION_PATH;

int BLOCK_SIZE;
__managed__ int NUM_BASIS_VECTOR;
__managed__ int NUM_PROJ_COND;

float SRC_OBJ_DISTANCE;
float SRC_DETECT_DISTANCE;
int NUM_PROJ;
int NUM_DETECT_U;
int NUM_DETECT_V;
float DETECTOR_SIZE;
int NUM_VOXEL;
int LOAD_INDEX[100];

__constant__ float elemR[117];
__constant__ float elemT[39];
__constant__ float INIT_OFFSET[39];
__managed__ int proj_arr[20];

__managed__ float basisVector[21];
__constant__ float fdThresh = 0.99f;
const float scatter_angle_xy = 0.0f;
__managed__ float d_loss_proj;
__managed__ float d_loss_norm;

void init_params(const std::string& tag) {
    std::ifstream f("../utility/settings.json");
    json data = json::parse(f);

    PROJ_PATH = data[tag]["proj_path"];
    VOLUME_PATH = data[tag]["vol_path"];
    DIRECTION_PATH = data[tag]["direc_path"];

    BLOCK_SIZE = data["recon_variable"]["blockSize"];
    NUM_BASIS_VECTOR = data["recon_variable"]["vector"];
    NUM_PROJ_COND = data["recon_variable"]["condition"];

    SRC_OBJ_DISTANCE = data[tag]["sod"];
    SRC_DETECT_DISTANCE = data[tag]["sdd"];
    NUM_PROJ = data[tag]["proj"];
    NUM_DETECT_U = data[tag]["num_det_u"];
    NUM_DETECT_V = data[tag]["num_det_v"];
    DETECTOR_SIZE = data[tag]["det_size"];
    NUM_VOXEL = data[tag]["num_voxel"];

    int proj_cond = data[tag]["rot"];

    std::vector<float> mRot = data[tag]["matrixRot"];
    std::vector<float> vTrans = data[tag]["vecTrans"];
    std::vector<float> recon = data[tag]["areaTrans"];
    std::vector<float> offset = data[tag]["offset"];
    std::vector<float> base = data["recon_variable"]["base"];
    bool sorting = data[tag]["sorting"];

    for (int i = 0; i < 3 * NUM_BASIS_VECTOR; i++) {
        basisVector[i] = base[i];
    }
    for (int i = 0; i < proj_cond; i++) {
        vTrans[3 * i + 0] += recon[0];
        vTrans[3 * i + 1] += recon[1];
        vTrans[3 * i + 2] += recon[2];
    }

    if (sorting) {
        std::vector<int> index = data[tag]["index"];
        for (int i = 0; i < NUM_PROJ_COND; i++) {
            LOAD_INDEX[i] = index[i] + 1;
            hipMemcpyToSymbol(HIP_SYMBOL(elemR), &(mRot[9 * index[i]]), 9 * sizeof(float), 9 * i * sizeof(float));
            hipMemcpyToSymbol(HIP_SYMBOL(elemT), &(vTrans[3 * index[i]]), 3 * sizeof(float), 3 * i * sizeof(float));
            hipMemcpyToSymbol(HIP_SYMBOL(INIT_OFFSET), &(offset[3 * index[i]]), 3 * sizeof(float), 3 * i * sizeof(float));
        }
    } else {
        for (int i = 0; i < NUM_PROJ_COND; i++) {
            LOAD_INDEX[i] = i + 1;
        }
        hipMemcpyToSymbol(HIP_SYMBOL(elemR), &(mRot[0]), mRot.size() * sizeof(float));
        hipMemcpyToSymbol(HIP_SYMBOL(elemT), &(vTrans[0]), vTrans.size() * sizeof(float));
        hipMemcpyToSymbol(HIP_SYMBOL(INIT_OFFSET), &(offset[0]), offset.size() * sizeof(float));
    }

    /*
     * __constant__ float elemR[117];
     * __constant__ float elemT[39];
     * __constant__ float INIT_OFFSET[39];
     */
    // std::cout << mRot[0] << std::endl;
    // std::cout <<  typeid(decltype(data[tag]["matrixRot"])).name();
}