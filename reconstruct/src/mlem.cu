#include "hip/hip_runtime.h"
//
// Created by tomokimori on 22/07/20.
//
#include <Geometry.h>
#include <mlem.cuh>
#include <random>
#include <Params.h>
#include <reconstruct.cuh>

__global__ void
forwardProjXTT(float *devProj, float *devVoxel, Geometry *geom, int cond,
               int y, int n) {
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int z = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= geom->voxel || z >= geom->voxel) return;

    const int coord[4] = {x, y, z, n};
    forwardXTTonDevice(coord, devProj, devVoxel, *geom, cond);
}

__global__ void
backwardProjXTT(float *devProj, float *devVoxelTmp, float *devVoxelFactor, Geometry *geom, int cond,
                const int y, const int n) {
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int z = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= geom->voxel || z >= geom->voxel) return;

    const int coord[4] = {x, y, z, n};
    backwardXTTonDevice(coord, devProj, devVoxelTmp, devVoxelFactor, *geom, cond);
}

__global__ void
forwardProj(float *devProj, float *devVoxel, Geometry *geom, int cond, int y, int n) {
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int z = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= geom->voxel || z >= geom->voxel) return;

    const int coord[4] = {x, y, z, n};
    forwardonDevice(coord, devProj, devVoxel, *geom, cond);
}

__global__ void
backwardProj(float *devProj, float *devVoxelTmp, float *devVoxelFactor, Geometry *geom, int cond,
             int y, int n) {
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int z = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= geom->voxel || z >= geom->voxel) return;

    const int coord[4] = {x, y, z, n};
    backwardonDevice(coord, devProj, devVoxelTmp, devVoxelFactor, *geom, cond);
}

__global__ void projRatio(float *devProj, const float *devSino, const Geometry *geom, const int n) {
    const int u = blockIdx.x * blockDim.x + threadIdx.x;
    const int v = blockIdx.y * blockDim.y + threadIdx.y;
    if (u >= geom->detect || v >= geom->detect) return;

    const int idx = u + geom->detect * v + geom->detect * geom->detect * abs(n);
    atomicAdd(&loss, abs(devSino[idx] - devProj[idx]));
    // const float div = devSino[idx] / devProj[idx];
    if (devProj[idx] != 0.0f)
        devProj[idx] = devSino[idx] / (devProj[idx] + 0.02f * (1.0f - exp(-abs(1.0f - devSino[idx] / devProj[idx]))));
    // a = b / c;
}

__global__ void
voxelProduct(float *devVoxel, const float *devVoxelTmp, const float *devVoxelFactor, const Geometry *geom,
             const int y) {
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int z = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= geom->voxel || z >= geom->voxel) return;

    for (int i = 0; i < NUM_BASIS_VECTOR; i++) {
        const int idxVoxel =
                x + geom->voxel * y + geom->voxel * geom->voxel * z + (geom->voxel * geom->voxel * geom->voxel) * i;
        const int idxOnPlane = x + geom->voxel * z + geom->voxel * geom->voxel * i;
        devVoxel[idxVoxel] = (devVoxelFactor[idxOnPlane] == 0.0f) ? 1e-10f : devVoxel[idxVoxel] *
                                                                             devVoxelTmp[idxOnPlane] /
                                                                             devVoxelFactor[idxOnPlane];
        /*
        if (devVoxelFactor[idxOnPlane] == 0.0f) {
            devVoxel[idxVoxel] = 0.0f;
        }
        else {
            if (devVoxel[idxVoxel] == 0.0f) {
                if (1 < x && x < geom->voxel - 1 && 1 < y && y < geom->voxel - 1 && 1 < z && z < geom->voxel - 1) {
                    devVoxel[idxVoxel] = (devVoxel[x - 1 + geom->voxel * y + geom->voxel * geom->voxel * z +
                                                   (geom->voxel * geom->voxel * geom->voxel) * i]
                                          + devVoxel[x + 1 + geom->voxel * y + geom->voxel * geom->voxel * z +
                                                     (geom->voxel * geom->voxel * geom->voxel) * i]
                                          + devVoxel[x + geom->voxel * (y - 1) + geom->voxel * geom->voxel * z +
                                                     (geom->voxel * geom->voxel * geom->voxel) * i]
                                          + devVoxel[x + geom->voxel * (y + 1) + geom->voxel * geom->voxel * z +
                                                     (geom->voxel * geom->voxel * geom->voxel) * i]
                                          + devVoxel[x + geom->voxel * y + geom->voxel * geom->voxel * (z - 1) +
                                                     (geom->voxel * geom->voxel * geom->voxel) * i]
                                          + devVoxel[x + geom->voxel * y + geom->voxel * geom->voxel * (z + 1) +
                                                     (geom->voxel * geom->voxel * geom->voxel) * i]) / 6.0f;
                }
            }
            devVoxel[idxVoxel] = devVoxel[idxVoxel] * devVoxelTmp[idxOnPlane] / devVoxelFactor[idxOnPlane];
        }
         */
    }
}

__global__ void sqrtVoxel(float *devVoxel, const Geometry *geom, const int y) {
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int z = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= geom->voxel || z >= geom->voxel) return;

    for (int i = 0; i < NUM_BASIS_VECTOR; i++) {
        const int idxVoxel =
                x + geom->voxel * y + geom->voxel * geom->voxel * z + (geom->voxel * geom->voxel * geom->voxel) * i;

        devVoxel[idxVoxel] = sqrt(devVoxel[idxVoxel]);

    }
}

__device__ void
forwardonDevice(const int coord[4], float *devProj, const float *devVoxel,
                const Geometry &geom, int cond) {

    int sizeV[3] = {geom.voxel, geom.voxel, geom.voxel};
    int sizeD[3] = {geom.detect, geom.detect, geom.nProj};

    float u, v;
    Vector3f B, G;
    rayCasting(u, v, B, G, cond, coord, geom);

    if (!(0.55f < u && u < (float) sizeD[0] - 0.55f && 0.55f < v && v < (float) sizeD[1] - 0.55f))
        return;

    float u_tmp = u - 0.5f, v_tmp = v - 0.5f;
    int intU = floor(u_tmp), intV = floor(v_tmp);
    float c1 = (1.0f - (u_tmp - (float) intU)) * (v_tmp - (float) intV),
            c2 = (u_tmp - (float) intU) * (v_tmp - (float) intV),
            c3 = (u_tmp - (float) intU) * (1.0f - (v_tmp - (float) intV)),
            c4 = (1.0f - (u_tmp - (float) intU)) * (1.0f - (v_tmp - (float) intV));

    const int n = abs(coord[3]);

    const int idxVoxel =
            coord[0] + sizeV[0] * coord[1] + sizeV[0] * sizeV[1] * coord[2] + cond * (sizeV[0] * sizeV[1] * sizeV[2]);
    atomicAdd(&devProj[intU + sizeD[0] * (intV + 1) + sizeD[0] * sizeD[1] * n],
              c1 * (float) geom.voxel * devVoxel[idxVoxel]);
    atomicAdd(&devProj[(intU + 1) + sizeD[0] * (intV + 1) + sizeD[0] * sizeD[1] * n],
              c2 * (float) geom.voxel * devVoxel[idxVoxel]);
    atomicAdd(&devProj[(intU + 1) + sizeD[0] * intV + sizeD[0] * sizeD[1] * n],
              c3 * (float) geom.voxel * devVoxel[idxVoxel]);
    atomicAdd(&devProj[intU + sizeD[0] * intV + sizeD[0] * sizeD[1] * n],
              c4 * (float) geom.voxel * devVoxel[idxVoxel]);

}

__device__ void
backwardonDevice(const int coord[4], const float *devProj, float *devVoxelTmp, float *devVoxelFactor,
                 const Geometry &geom, int cond) {

    int sizeV[3] = {geom.voxel, geom.voxel, geom.voxel};
    int sizeD[3] = {geom.detect, geom.detect, geom.nProj};

    float u, v;
    Vector3f B, G;
    rayCasting(u, v, B, G, cond, coord, geom);

    if (!(0.55f < u && u < (float) sizeD[0] - 0.55f && 0.55f < v && v < (float) sizeD[1] - 0.55f))
        return;

    const int n = abs(coord[3]);

    float u_tmp = u - 0.5f, v_tmp = v - 0.5f;
    int intU = floor(u_tmp), intV = floor(v_tmp);
    float c1 = (1.0f - (u_tmp - (float) intU)) * (v_tmp - (float) intV), c2 =
            (u_tmp - (float) intU) * (v_tmp - (float) intV),
            c3 = (u_tmp - (float) intU) * (1.0f - (v_tmp - (float) intV)), c4 =
            (1.0f - (u_tmp - (float) intU)) * (1.0f - (v_tmp - (float) intV));

    const int idxVoxel = coord[0] + sizeV[0] * coord[2] + cond * (sizeV[0] * sizeV[1]);
    const float numBack = c1 * devProj[intU + sizeD[0] * (intV + 1) + sizeD[0] * sizeD[1] * n] +
                          c2 * devProj[(intU + 1) + sizeD[0] * (intV + 1) + sizeD[0] * sizeD[1] * n] +
                          c3 * devProj[(intU + 1) + sizeD[0] * intV + sizeD[0] * sizeD[1] * n] +
                          c4 * devProj[intU + sizeD[0] * intV + sizeD[0] * sizeD[1] * n];

    devVoxelFactor[idxVoxel] += 1.0f * geom.voxSize;
    devVoxelTmp[idxVoxel] += numBack;
}

__device__ void
forwardXTTonDevice(const int coord[4], float *devProj, const float *devVoxel,
                   const Geometry &geom, int cond) {

    int sizeV[3] = {geom.voxel, geom.voxel, geom.voxel};
    int sizeD[3] = {geom.detect, geom.detect, geom.nProj};

    float u = 0.0f, v = 0.0f;
    Vector3f B(0.0f, 0.0f, 0.0f), G(0.0f, 0.0f, 0.0f);
    rayCasting(u, v, B, G, cond, coord, geom);

    if (!(0.55f < u && u < (float) sizeD[0] - 0.55f && 0.55f < v && v < (float) sizeD[1] - 0.55f))
        return;

    const int n = abs(coord[3]);

    float u_tmp = u - 0.5f, v_tmp = v - 0.5f;
    int intU = floor(u_tmp), intV = floor(v_tmp);
    float c1 = (1.0f - (u_tmp - (float) intU)) * (v_tmp - (float) intV),
            c2 = (u_tmp - (float) intU) * (v_tmp - (float) intV),
            c3 = (u_tmp - (float) intU) * (1.0f - (v_tmp - (float) intV)),
            c4 = (1.0f - (u_tmp - (float) intU)) * (1.0f - (v_tmp - (float) intV));

    for (int i = 0; i < NUM_BASIS_VECTOR; i++) {
        // add scattering coefficient (read paper)
        // B->beam direction unit vector (src2voxel)
        // S->scattering base vector
        // G->grating sensivity vector
        Vector3f S(basisVector[3 * i + 0], basisVector[3 * i + 1], basisVector[3 * i + 2]);
        float vkm = B.cross(S).norm2() * abs(S * G);
        const int idxVoxel =
                coord[0] + sizeV[0] * coord[1] + sizeV[0] * sizeV[1] * coord[2] + i * (sizeV[0] * sizeV[1] * sizeV[2]);
        atomicAdd(&devProj[intU + sizeD[0] * (intV + 1) + sizeD[0] * sizeD[1] * n],
                  vkm * vkm * c1 * devVoxel[idxVoxel]);
        atomicAdd(&devProj[(intU + 1) + sizeD[0] * (intV + 1) + sizeD[0] * sizeD[1] * n],
                  vkm * vkm * c2 * devVoxel[idxVoxel]);
        atomicAdd(&devProj[(intU + 1) + sizeD[0] * intV + sizeD[0] * sizeD[1] * n],
                  vkm * vkm * c3 * devVoxel[idxVoxel]);
        atomicAdd(&devProj[intU + sizeD[0] * intV + sizeD[0] * sizeD[1] * n], vkm * vkm * c4 * devVoxel[idxVoxel]);
        // printf("%d: %lf\n", i+1, vkm);
        // printf("sinogram: %lf\n", devSino[intU + sizeD[0] * intV + sizeD[0] * sizeD[1] * n]);
    }
}

// change to class
__device__ void
backwardXTTonDevice(const int coord[4], const float *devProj, float *devVoxelTmp, float *devVoxelFactor,
                    const Geometry &geom, int cond) {

    int sizeV[3] = {geom.voxel, geom.voxel, geom.voxel};
    int sizeD[3] = {geom.detect, geom.detect, geom.nProj};

    float u = 0.0f, v = 0.0f;
    Vector3f B(0.0f, 0.0f, 0.0f), G(0.0f, 0.0f, 0.0f);
    rayCasting(u, v, B, G, cond, coord, geom);

    if (!(0.55f < u && u < (float) sizeD[0] - 0.55f && 0.55f < v && v < (float) sizeD[1] - 0.55f))
        return;

    const int n = abs(coord[3]);
    float u_tmp = u - 0.5f, v_tmp = v - 0.5f;
    int intU = floor(u_tmp), intV = floor(v_tmp);
    float c1 = (1.0f - (u_tmp - (float) intU)) * (v_tmp - (float) intV), c2 =
            (u_tmp - (float) intU) * (v_tmp - (float) intV),
            c3 = (u_tmp - (float) intU) * (1.0f - (v_tmp - (float) intV)), c4 =
            (1.0f - (u_tmp - (float) intU)) * (1.0f - (v_tmp - (float) intV));

    for (int i = 0; i < NUM_BASIS_VECTOR; i++) {
        // calculate immutable geometry
        // add scattering coefficient (read paper)
        // B->beam direction unit vector (src2voxel)
        // S->scattering base vector
        // G->grating sensivity vector
        // v_km = (|B_m x S_k|<S_k*G>)^2
        Vector3f S(basisVector[3 * i + 0], basisVector[3 * i + 1], basisVector[3 * i + 2]);
        float vkm = B.cross(S).norm2() * abs(S * G);
        const int idxVoxel = coord[0] + sizeV[0] * coord[2] + i * (sizeV[0] * sizeV[1]);
        const float backForward = vkm * vkm * c1 * devProj[intU + sizeD[0] * (intV + 1) + sizeD[0] * sizeD[1] * n] +
                                  vkm * vkm * c2 *
                                  devProj[(intU + 1) + sizeD[0] * (intV + 1) + sizeD[0] * sizeD[1] * n] +
                                  vkm * vkm * c3 * devProj[(intU + 1) + sizeD[0] * intV + sizeD[0] * sizeD[1] * n] +
                                  vkm * vkm * c4 * devProj[intU + sizeD[0] * intV + sizeD[0] * sizeD[1] * n];

        devVoxelFactor[idxVoxel] += (vkm * vkm);
        devVoxelTmp[idxVoxel] += backForward;
    }
}


