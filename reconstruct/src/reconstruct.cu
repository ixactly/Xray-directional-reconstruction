#include "hip/hip_runtime.h"
//
// Created by tomokimori on 22/08/30.
//
#include <Geometry.h>
#include <ir.cuh>
#include <fdk.cuh>
#include <fiber.cuh>
#include <random>
#include <memory>
#include <Pbar.h>
#include <Params.h>
#include <Volume.h>
#include <omp.h>
#include <pca.cuh>
#include <reconstruct.cuh>

namespace IR {
    void
    reconstruct(Volume<float> *sinogram, Volume<float> *voxel, const Geometry &geom, int epoch, int batch, Rotate dir,
                Method method, float lambda) {
        std::cout << "starting reconstruct(IR)..." << std::endl;
        for (int i = 0; i < NUM_BASIS_VECTOR; i++) {
            voxel[i].forEach([](float value) -> float { return 0.01; });
        }

        int rotation = (dir == Rotate::CW) ? 1 : -1;

        int sizeV[3] = {voxel[0].x(), voxel[0].y(), voxel[0].z()};
        int sizeD[3] = {sinogram[0].x(), sinogram[0].y(), sinogram[0].z()};
        int nProj = sizeD[2];

        // hipMalloc
        float *devSino, *devProj, *devVoxel, *devVoxelFactor, *devVoxelTmp;
        const long lenV = sizeV[0] * sizeV[1] * sizeV[2];
        const long lenD = sizeD[0] * sizeD[1] * sizeD[2];

        hipMalloc(&devSino, sizeof(float) * lenD * NUM_PROJ_COND);
        hipMalloc(&devProj, sizeof(float) * lenD * NUM_PROJ_COND); // memory can be small to subsetSize
        hipMalloc(&devVoxel, sizeof(float) * lenV * NUM_BASIS_VECTOR);
        hipMalloc(&devVoxelFactor, sizeof(float) * sizeV[0] * sizeV[1] * NUM_BASIS_VECTOR);
        hipMalloc(&devVoxelTmp, sizeof(float) * sizeV[0] * sizeV[1] * NUM_BASIS_VECTOR);

        for (int i = 0; i < NUM_PROJ_COND; i++)
            hipMemcpy(&devSino[i * lenD], sinogram[i].get(), sizeof(float) * lenD, hipMemcpyHostToDevice);
        for (int i = 0; i < NUM_BASIS_VECTOR; i++)
            hipMemcpy(&devVoxel[i * lenV], voxel[i].get(), sizeof(float) * lenV, hipMemcpyHostToDevice);

        Geometry *devGeom;
        hipMalloc(&devGeom, sizeof(Geometry));
        hipMemcpy(devGeom, &geom, sizeof(Geometry), hipMemcpyHostToDevice);

        // define blocksize
        const int blockSize = 16;
        dim3 blockV(blockSize, blockSize, 1);
        dim3 gridV((sizeV[0] + blockSize - 1) / blockSize, (sizeV[2] + blockSize - 1) / blockSize, 1);
        dim3 blockD(blockSize, blockSize, 1);
        dim3 gridD((sizeD[0] + blockSize - 1) / blockSize, (sizeD[1] + blockSize - 1) / blockSize, 1);

        // forwardProj, divide, backwardProj proj
        int subsetSize = (nProj + batch - 1) / batch;
        std::vector<int> subsetOrder(batch);
        for (int i = 0; i < batch; i++) {
            subsetOrder[i] = i;
        }

        std::vector<float> losses(epoch);

        // progress bar
        progressbar pbar(epoch * batch * NUM_PROJ_COND * (subsetSize + sizeV[1]));

        // set scattering vector direction
        // setScatterDirecOn4D(2.0f * (float) M_PI * scatter_angle_xy / 360.0f, basisVector);

        // main routine
        for (int ep = 0; ep < epoch; ep++) {
            std::mt19937_64 get_rand_mt; // fixed seed
            std::shuffle(subsetOrder.begin(), subsetOrder.end(), get_rand_mt);
            hipMemset(&loss, 0.0f, sizeof(float));
            hipMemset(devProj, 0.0f, sizeof(float) * lenD * NUM_PROJ_COND);
            for (int &sub: subsetOrder) {
                // forwardProj and ratio
                for (int cond = 0; cond < NUM_PROJ_COND; cond++) {
                    for (int subOrder = 0; subOrder < subsetSize; subOrder++) {
                        int n = rotation * ((sub + batch * subOrder) % nProj);
                        // !!care!! judge from vecSod which plane we chose
                        pbar.update();

                        // forwardProj process
                        for (int y = 0; y < sizeV[1]; y++) {
                            forwardProj<<<gridV, blockV>>>(&devProj[lenD * cond], devVoxel, devGeom, cond, y, n);
                            hipDeviceSynchronize();
                        }
                        // ratio process
                        if (method == Method::ART) {
                            projSubtract<<<gridD, blockD>>>(&devProj[lenD * cond], &devSino[lenD * cond], devGeom, n);
                        } else {
                            projRatio<<<gridD, blockD>>>(&devProj[lenD * cond], &devSino[lenD * cond], devGeom, n);
                        }
                        hipDeviceSynchronize();
                    }
                }

                // backwardProj process
                for (int y = 0; y < sizeV[1]; y++) {
                    hipMemset(devVoxelFactor, 0, sizeof(float) * sizeV[0] * sizeV[1] * NUM_BASIS_VECTOR);
                    hipMemset(devVoxelTmp, 0, sizeof(float) * sizeV[0] * sizeV[1] * NUM_BASIS_VECTOR);
                    for (int cond = 0; cond < NUM_PROJ_COND; cond++) {
                        pbar.update();
                        for (int subOrder = 0; subOrder < subsetSize; subOrder++) {
                            int n = rotation * ((sub + batch * subOrder) % nProj);
                            backwardProj<<<gridV, blockV>>>(&devProj[lenD * cond], devVoxelTmp, devVoxelFactor, devGeom,
                                                            cond, y, n);
                            hipDeviceSynchronize();
                        }
                    }
                    if (method == Method::ART) {
                        voxelPlus<<<gridV, blockV>>>(devVoxel, devVoxelTmp, lambda / (float) subsetSize, devGeom, y);
                    } else {
                        voxelProduct<<<gridV, blockV>>>(devVoxel, devVoxelTmp, devVoxelFactor, devGeom, y);
                    }
                    hipDeviceSynchronize();
                }
            }

            loss /= static_cast<float>(NUM_DETECT_V * NUM_DETECT_U * NUM_PROJ);
            hipMemcpy(losses.data() + ep, &loss, sizeof(float), hipMemcpyDeviceToHost); // loss
        }

        for (int i = 0; i < NUM_PROJ_COND; i++)
            hipMemcpy(sinogram[i].get(), &devProj[i * lenD], sizeof(float) * lenD, hipMemcpyDeviceToHost);
        for (int i = 0; i < NUM_BASIS_VECTOR; i++)
            hipMemcpy(voxel[i].get(), &devVoxel[i * lenV], sizeof(float) * lenV, hipMemcpyDeviceToHost);

        hipFree(devProj);
        hipFree(devSino);
        hipFree(devVoxel);
        hipFree(devGeom);
        hipFree(devVoxelFactor);
        hipFree(devVoxelTmp);

        std::ofstream ofs("../python/loss.csv");
        for (auto &e: losses)
            ofs << e << ",";
    }
}

namespace XTT {
    void orthReconstruct(Volume<float> *sinogram, Volume<float> voxel[3], Volume<float> md[3], const Geometry &geom,
                         int iter1, int iter2, int batch, Rotate dir, Method method, float lambda) {
        std::cout << "starting reconstruct(orth)..." << std::endl;

        // int rotation = (dir == Rotate::CW) ? -1 : 1;
        int rotation = (dir == Rotate::CW) ? 1 : -1;

        int sizeV[3] = {voxel[0].x(), voxel[0].y(), voxel[0].z()};
        int sizeD[3] = {sinogram[0].x(), sinogram[0].y(), sinogram[0].z()};
        int nProj = sizeD[2];

        // hipMalloc
        float *devSino, *devProj, *devVoxel, *devVoxelFactor, *devVoxelTmp;
        const long lenV = sizeV[0] * sizeV[1] * sizeV[2];
        const long lenD = sizeD[0] * sizeD[1] * sizeD[2];

        hipMalloc(&devSino, sizeof(float) * lenD * NUM_PROJ_COND);
        hipMalloc(&devProj, sizeof(float) * lenD * NUM_PROJ_COND); // memory can be small to subsetSize
        hipMalloc(&devVoxel, sizeof(float) * lenV * NUM_BASIS_VECTOR);
        hipMalloc(&devVoxelFactor, sizeof(float) * sizeV[0] * sizeV[1] * NUM_BASIS_VECTOR);
        hipMalloc(&devVoxelTmp, sizeof(float) * sizeV[0] * sizeV[1] * NUM_BASIS_VECTOR);

        if (method == Method::MLEM) {
            for (int i = 0; i < NUM_BASIS_VECTOR; i++)
                voxel[i].forEach([](float value) -> float { return 0.01f; });
        } else {
            for (int i = 0; i < NUM_BASIS_VECTOR; i++)
                voxel[i].forEach([](float value) -> float { return 0.0f; });
        }
        for (int i = 0; i < NUM_PROJ_COND; i++)
            hipMemcpy(&devSino[i * lenD], sinogram[i].get(), sizeof(float) * lenD, hipMemcpyHostToDevice);
        for (int i = 0; i < NUM_BASIS_VECTOR; i++)
            hipMemcpy(&devVoxel[i * lenV], voxel[i].get(), sizeof(float) * lenV, hipMemcpyHostToDevice);

        // store theta, phi on polar coordination to devDirection
        float *devCoef;
        hipMalloc(&devCoef, sizeof(float) * lenV * 4);
        std::vector<float> hCoef1(lenV);
        for (auto &e: hCoef1)
            e = 1.0f;

        hipMemcpy(&devCoef[lenV * 0], hCoef1.data(), sizeof(float) * lenV, hipMemcpyHostToDevice);
        hipMemcpy(&devCoef[lenV * 2], hCoef1.data(), sizeof(float) * lenV, hipMemcpyHostToDevice);

        Geometry *devGeom;
        hipMalloc(&devGeom, sizeof(Geometry));
        hipMemcpy(devGeom, &geom, sizeof(Geometry), hipMemcpyHostToDevice);

        // define blocksize
        const int blockSize = 16;
        dim3 blockV(blockSize, blockSize, 1);
        dim3 gridV((sizeV[0] + blockSize - 1) / blockSize, (sizeV[2] + blockSize - 1) / blockSize, 1);
        dim3 blockD(blockSize, blockSize, 1);
        dim3 gridD((sizeD[0] + blockSize - 1) / blockSize, (sizeD[1] + blockSize - 1) / blockSize, 1);

        // forwardProj, divide, backwardProj proj
        int subsetSize = (nProj + batch - 1) / batch;
        std::vector<int> subsetOrder(batch);
        for (int i = 0; i < batch; i++) {
            subsetOrder[i] = i;
        }

        std::vector<float> losses(iter1);

        // progress bar
        progressbar pbar(iter1 * iter2 * batch * NUM_PROJ_COND * (subsetSize + sizeV[1]));

        // set scattering vector direction
        // setScatterDirecOn4D(2.0f * (float) M_PI * scatter_angle_xy / 360.0f, basisVector);

        // main routine
        for (int ep1 = 0; ep1 < iter1; ep1++) {
            for (int i = 0; i < 3; i++)
                hipMemcpy(&devVoxel[i * lenV], voxel[i].get(), sizeof(float) * lenV, hipMemcpyHostToDevice);

            for (int ep2 = 0; ep2 < iter2; ep2++) {
                std::mt19937_64 get_rand_mt; // fixed seed
                std::shuffle(subsetOrder.begin(), subsetOrder.end(), get_rand_mt);
                hipMemset(&loss, 0.0f, sizeof(float));
                hipMemset(devProj, 0.0f, sizeof(float) * lenD * NUM_PROJ_COND);
                for (int &sub: subsetOrder) {
                    // forwardProj and ratio
                    for (int cond = 0; cond < NUM_PROJ_COND; cond++) {
                        for (int subOrder = 0; subOrder < subsetSize; subOrder++) {
                            int n = rotation * ((sub + batch * subOrder) % nProj);
                            // !!care!! judge from vecSod which plane we chose
                            pbar.update();

                            // forwardProj process
                            for (int y = 0; y < sizeV[1]; y++) {
                                // iterate basis vector in forwardProjXTT

                                forwardOrth<<<gridV, blockV>>>(&devProj[lenD * cond], devVoxel, devCoef,
                                                               cond, y, n, ep1, devGeom);
                                hipDeviceSynchronize();
                            }

                            // ratio process
                            if (method == Method::ART)
                                projSubtract<<<gridD, blockD>>>(&devProj[lenD * cond],
                                                                &devSino[lenD * cond], devGeom, n);
                            else
                                projRatio<<<gridD, blockD>>>(&devProj[lenD * cond], &devSino[lenD * cond], devGeom, n);
                            hipDeviceSynchronize();
                        }
                    }

                    // backwardProj process
                    for (int y = 0; y < sizeV[1]; y++) {
                        hipMemset(devVoxelFactor, 0, sizeof(float) * sizeV[0] * sizeV[1] * NUM_BASIS_VECTOR);
                        hipMemset(devVoxelTmp, 0, sizeof(float) * sizeV[0] * sizeV[1] * NUM_BASIS_VECTOR);
                        for (int cond = 0; cond < NUM_PROJ_COND; cond++) {
                            pbar.update();
                            for (int subOrder = 0; subOrder < subsetSize; subOrder++) {
                                int n = rotation * ((sub + batch * subOrder) % nProj);

                                backwardOrth<<<gridV, blockV>>>(&devProj[lenD * cond], devCoef, devVoxelTmp,
                                                                devVoxelFactor, devGeom, cond, y, n, ep1);
                                hipDeviceSynchronize();
                            }
                        }
                        if (method == Method::ART) {
                            voxelPlus<<<gridV, blockV>>>(devVoxel, devVoxelTmp, lambda / (float) subsetSize,
                                                         devGeom, y);
                        } else {
                            voxelProduct<<<gridV, blockV>>>(devVoxel, devVoxelTmp, devVoxelFactor, devGeom, y);
                        }
                        hipDeviceSynchronize();
                    }
                }
            }

            // out iter1
            for (int y = 0; y < sizeV[1]; y++) {
                voxelSqrt<<<gridV, blockV>>>(devVoxel, devGeom, y);
                hipDeviceSynchronize();
                calcNormalVector<<<gridV, blockV>>>(devVoxel, devCoef, y, ep1, devGeom);
                hipDeviceSynchronize();
            }

            loss /= static_cast<float>(NUM_DETECT_V * NUM_DETECT_U * NUM_PROJ);
            hipMemcpy(losses.data() + ep1, &loss, sizeof(float), hipMemcpyDeviceToHost); // loss
        }

        for (int i = 0; i < NUM_PROJ_COND; i++)
            hipMemcpy(sinogram[i].get(), &devProj[i * lenD], sizeof(float) * lenD, hipMemcpyDeviceToHost);
        for (int i = 0; i < NUM_BASIS_VECTOR; i++)
            hipMemcpy(voxel[i].get(), &devVoxel[i * lenV], sizeof(float) * lenV, hipMemcpyDeviceToHost);

        Volume<float> coef[4];
        for (int i = 0; i < 4; i++) {
            coef[i] = Volume<float>(NUM_VOXEL, NUM_VOXEL, NUM_VOXEL);
            hipMemcpy(coef[i].get(), &devCoef[i * lenV], sizeof(float) * lenV, hipMemcpyDeviceToHost);
        }

        convertNormVector(voxel, md, coef);
        // need convert phi, theta to direction(size<-mu1 + mu2 / 2)

        hipFree(devProj);
        hipFree(devSino);
        hipFree(devVoxel);
        hipFree(devGeom);
        hipFree(devVoxelFactor);
        hipFree(devVoxelTmp);
        hipFree(devCoef);

        std::ofstream ofs("../python/loss.csv");
        for (auto &e: losses)
            ofs << e << ",";

    }

    void newReconstruct(Volume<float> *sinogram, Volume<float> *voxel, Volume<float> *md, const Geometry &geom,
                        int iter1, int iter2, int batch, Rotate dir, Method method, float lambda) {
        std::cout << "starting reconstruct(XTT)..." << std::endl;
        if (method == Method::MLEM) {
            for (int i = 0; i < NUM_BASIS_VECTOR; i++) {
                voxel[i].forEach([](float value) -> float { return 0.01; });
            }
        }

        // int rotation = (dir == Rotate::CW) ? -1 : 1;
        int rotation = (dir == Rotate::CW) ? 1 : -1;

        int sizeV[3] = {voxel[0].x(), voxel[0].y(), voxel[0].z()};
        int sizeD[3] = {sinogram[0].x(), sinogram[0].y(), sinogram[0].z()};
        int nProj = sizeD[2];

        // hipMalloc
        float *devSino, *devProj, *devVoxel, *hostVoxel, *devVoxelFactor, *devVoxelTmp;
        const long lenV = sizeV[0] * sizeV[1] * sizeV[2];
        const long lenD = sizeD[0] * sizeD[1] * sizeD[2];

        hipMalloc(&devSino, sizeof(float) * lenD * NUM_PROJ_COND);
        hipMalloc(&devProj, sizeof(float) * lenD * NUM_PROJ_COND); // memory can be small to subsetSize
        hipMalloc(&devVoxel, sizeof(float) * lenV * NUM_BASIS_VECTOR);
        hipMalloc(&hostVoxel, sizeof(float) * lenV * NUM_BASIS_VECTOR);
        hipMalloc(&devVoxelFactor, sizeof(float) * sizeV[0] * sizeV[1] * NUM_BASIS_VECTOR);
        hipMalloc(&devVoxelTmp, sizeof(float) * sizeV[0] * sizeV[1] * NUM_BASIS_VECTOR);

        // direction, phi, theta
        float *devDirection;
        hipMalloc(&devDirection, sizeof(float) * lenV * 2);
        hipMemset(devDirection, 0.0f, sizeof(float) * lenV * 2);
        for (int i = 0; i < 3; i++) {
            md[i] = Volume<float>(NUM_VOXEL, NUM_VOXEL, NUM_VOXEL);
        }

        for (int i = 0; i < NUM_PROJ_COND; i++)
            hipMemcpy(&devSino[i * lenD], sinogram[i].get(), sizeof(float) * lenD, hipMemcpyHostToDevice);
        for (int i = 0; i < NUM_BASIS_VECTOR; i++)
            hipMemcpy(&devVoxel[i * lenV], voxel[i].get(), sizeof(float) * lenV, hipMemcpyHostToDevice);

        Geometry *devGeom;
        hipMalloc(&devGeom, sizeof(Geometry));
        hipMemcpy(devGeom, &geom, sizeof(Geometry), hipMemcpyHostToDevice);

        // define blocksize
        const int blockSize = 16;
        dim3 blockV(blockSize, blockSize, 1);
        dim3 gridV((sizeV[0] + blockSize - 1) / blockSize, (sizeV[2] + blockSize - 1) / blockSize, 1);
        dim3 blockD(blockSize, blockSize, 1);
        dim3 gridD((sizeD[0] + blockSize - 1) / blockSize, (sizeD[1] + blockSize - 1) / blockSize, 1);

        // forwardProj, divide, backwardProj proj
        int subsetSize = (nProj + batch - 1) / batch;
        std::vector<int> subsetOrder(batch);
        for (int i = 0; i < batch; i++) {
            subsetOrder[i] = i;
        }

        std::vector<float> losses(iter1 * iter2);

        // progress bar
        progressbar pbar(iter1 * iter2 * batch * NUM_PROJ_COND * (subsetSize + sizeV[1]));

        // set scattering vector direction
        // setScatterDirecOn4D(2.0f * (float) M_PI * scatter_angle_xy / 360.0f, basisVector);

        // main routine
        for (int it2 = 0; it2 < iter1; it2++) {
            for (int ep = 0; ep < iter2; ep++) {
                std::mt19937_64 get_rand_mt; // fixed seed
                std::shuffle(subsetOrder.begin(), subsetOrder.end(), get_rand_mt);
                hipMemset(&loss, 0.0f, sizeof(float));
                hipMemset(devProj, 0.0f, sizeof(float) * lenD * NUM_PROJ_COND);

                for (int &sub: subsetOrder) {
                    // forwardProj and ratio
                    for (int cond = 0; cond < NUM_PROJ_COND; cond++) {
                        for (int subOrder = 0; subOrder < subsetSize; subOrder++) {
                            int n = rotation * ((sub + batch * subOrder) % nProj);
                            // !!care!! judge from vecSod which plane we chose
                            pbar.update();

                            // forwardProj process
                            for (int y = 0; y < sizeV[1]; y++) {
                                // iterate basis vector in forwardProjXTT
                                forwardProjXTTbyFiber<<<gridV, blockV>>>(&devProj[lenD * cond], devVoxel, *devGeom,
                                                                         cond, y, n, devDirection);
                                hipDeviceSynchronize();
                            }

                            // ratio process
                            if (method == Method::ART) {
                                projSubtract<<<gridD, blockD>>>(&devProj[lenD * cond], &devSino[lenD * cond],
                                                                devGeom, n);
                            } else {
                                projRatio<<<gridD, blockD>>>(&devProj[lenD * cond], &devSino[lenD * cond], devGeom, n);
                            }
                            hipDeviceSynchronize();
                        }
                    }

                    // backwardProj process
                    for (int y = 0; y < sizeV[1]; y++) {
                        hipMemset(devVoxelFactor, 0, sizeof(float) * sizeV[0] * sizeV[1] * NUM_BASIS_VECTOR);
                        hipMemset(devVoxelTmp, 0, sizeof(float) * sizeV[0] * sizeV[1] * NUM_BASIS_VECTOR);
                        for (int cond = 0; cond < NUM_PROJ_COND; cond++) {
                            pbar.update();
                            for (int subOrder = 0; subOrder < subsetSize; subOrder++) {
                                int n = rotation * ((sub + batch * subOrder) % nProj);
                                backwardProjXTTbyFiber<<<gridV, blockV>>>(&devProj[lenD * cond], devVoxelTmp,
                                                                          devVoxelFactor,
                                                                          *devGeom, cond, y, n, devDirection);
                                hipDeviceSynchronize();
                            }
                        }
                        if (method == Method::ART) {
                            voxelPlus<<<gridV, blockV>>>(devVoxel, devVoxelTmp, lambda / (float) subsetSize,
                                                         devGeom, y);
                        } else {
                            voxelProduct<<<gridV, blockV>>>(devVoxel, devVoxelTmp, devVoxelFactor, devGeom, y);
                        }
                        hipDeviceSynchronize();
                    }
                }

                loss /= static_cast<float>(NUM_DETECT_V * NUM_DETECT_U * NUM_PROJ);
                hipMemcpy(losses.data() + ep, &loss, sizeof(float), hipMemcpyDeviceToHost); // loss

                // record sqrt of voxel val to host memory
                for (int y = 0; y < sizeV[1]; y++) {
                    voxelSqrtFromSrc<<<gridV, blockV>>>(hostVoxel, devVoxel, devGeom, y); // host
                    hipDeviceSynchronize();
                }

                for (int i = 0; i < NUM_BASIS_VECTOR; i++) {
                    hipMemcpy(voxel[i].get(), &hostVoxel[i * lenV], sizeof(float) * lenV, hipMemcpyDeviceToHost);
                    hipDeviceSynchronize();
                }

                // calc main direction
                Volume<float> tmp[3];
                for (auto &e: tmp) {
                    e = Volume<float>(NUM_VOXEL, NUM_VOXEL, NUM_VOXEL);
                }

                for (int z = 0; z < NUM_VOXEL; z++) {
#pragma parallel omp for
                    for (int y = 0; y < NUM_VOXEL; y++) {
                        for (int x = 0; x < NUM_VOXEL; x++) {
                            calcEigenVector(voxel, md, tmp, y, z, x);
                        }
                    }
                }
                for (int i = 0; i < 3; i++)
                    hipMemcpy(&devDirection[i * lenV], md[i].get(), sizeof(float) * lenV, hipMemcpyHostToDevice);
            }
            // copy md to devMD
            for (int i = 0; i < 3; i++)
                hipMemcpy(md[i].get(), &devDirection[i * lenV], sizeof(float) * lenV, hipMemcpyDeviceToHost);
        }

        for (int i = 0; i < NUM_PROJ_COND; i++)
            hipMemcpy(sinogram[i].get(), &devProj[i * lenD], sizeof(float) * lenD, hipMemcpyDeviceToHost);
        /*
        for (int i = 0; i < NUM_BASIS_VECTOR; i++)
            hipMemcpy(voxel[i].get(), &hostVoxel[i * lenV], sizeof(float) * lenV, hipMemcpyDeviceToHost);
*/
        hipFree(devProj);
        hipFree(devSino);
        hipFree(devVoxel);
        hipFree(devGeom);
        hipFree(devVoxelFactor);
        hipFree(devVoxelTmp);
        hipFree(hostVoxel);
        hipFree(devDirection);

        std::ofstream ofs("../python/loss.csv");
        for (auto &e: losses)
            ofs << e << ",";
    }

    void
    reconstruct(Volume<float> *sinogram, Volume<float> *voxel, Volume<float> *md, const Geometry &geom,
                int epoch, int batch, Rotate dir, Method method, float lambda) {
        std::cout << "starting reconstruct(XTT)..." << std::endl;
        if (method == Method::MLEM) {
            for (int i = 0; i < NUM_BASIS_VECTOR; i++) {
                voxel[i].forEach([](float value) -> float { return 0.01; });
            }
        }

        // int rotation = (dir == Rotate::CW) ? -1 : 1;
        int rotation = (dir == Rotate::CW) ? 1 : -1;

        int sizeV[3] = {voxel[0].x(), voxel[0].y(), voxel[0].z()};
        int sizeD[3] = {sinogram[0].x(), sinogram[0].y(), sinogram[0].z()};
        int nProj = sizeD[2];

        // hipMalloc
        float *devSino, *devProj, *devVoxel, *devVoxelFactor, *devVoxelTmp;
        const long lenV = sizeV[0] * sizeV[1] * sizeV[2];
        const long lenD = sizeD[0] * sizeD[1] * sizeD[2];

        hipMalloc(&devSino, sizeof(float) * lenD * NUM_PROJ_COND);
        hipMalloc(&devProj, sizeof(float) * lenD * NUM_PROJ_COND); // memory can be small to subsetSize
        hipMalloc(&devVoxel, sizeof(float) * lenV * NUM_BASIS_VECTOR);
        hipMalloc(&devVoxelFactor, sizeof(float) * sizeV[0] * sizeV[1] * NUM_BASIS_VECTOR);
        hipMalloc(&devVoxelTmp, sizeof(float) * sizeV[0] * sizeV[1] * NUM_BASIS_VECTOR);

        for (int i = 0; i < NUM_PROJ_COND; i++)
            hipMemcpy(&devSino[i * lenD], sinogram[i].get(), sizeof(float) * lenD, hipMemcpyHostToDevice);
        for (int i = 0; i < NUM_BASIS_VECTOR; i++)
            hipMemcpy(&devVoxel[i * lenV], voxel[i].get(), sizeof(float) * lenV, hipMemcpyHostToDevice);

        Geometry *devGeom;
        hipMalloc(&devGeom, sizeof(Geometry));
        hipMemcpy(devGeom, &geom, sizeof(Geometry), hipMemcpyHostToDevice);

        // define blocksize
        const int blockSize = 16;
        dim3 blockV(blockSize, blockSize, 1);
        dim3 gridV((sizeV[0] + blockSize - 1) / blockSize, (sizeV[2] + blockSize - 1) / blockSize, 1);
        dim3 blockD(blockSize, blockSize, 1);
        dim3 gridD((sizeD[0] + blockSize - 1) / blockSize, (sizeD[1] + blockSize - 1) / blockSize, 1);

        // forwardProj, divide, backwardProj proj
        int subsetSize = (nProj + batch - 1) / batch;
        std::vector<int> subsetOrder(batch);
        for (int i = 0; i < batch; i++) {
            subsetOrder[i] = i;
        }

        std::vector<float> losses(epoch);

        // progress bar
        progressbar pbar(epoch * batch * NUM_PROJ_COND * (subsetSize + sizeV[1]));

        // set scattering vector direction
        // setScatterDirecOn4D(2.0f * (float) M_PI * scatter_angle_xy / 360.0f, basisVector);

        // main routine
        for (int ep = 0; ep < epoch; ep++) {
            std::mt19937_64 get_rand_mt; // fixed seed
            std::shuffle(subsetOrder.begin(), subsetOrder.end(), get_rand_mt);
            hipMemset(&loss, 0.0f, sizeof(float));
            hipMemset(devProj, 0.0f, sizeof(float) * lenD * NUM_PROJ_COND);
            for (int &sub: subsetOrder) {
                // forwardProj and ratio
                for (int cond = 0; cond < NUM_PROJ_COND; cond++) {
                    for (int subOrder = 0; subOrder < subsetSize; subOrder++) {
                        int n = rotation * ((sub + batch * subOrder) % nProj);
                        // !!care!! judge from vecSod which plane we chose
                        pbar.update();

                        // forwardProj process
                        for (int y = 0; y < sizeV[1]; y++) {
                            // iterate basis vector in forwardProjXTT
                            forwardProjXTT<<<gridV, blockV>>>(&devProj[lenD * cond], devVoxel, devGeom, cond, y, n);
                            hipDeviceSynchronize();
                        }

                        // ratio process
                        if (method == Method::ART) {
                            projSubtract<<<gridD, blockD>>>(&devProj[lenD * cond], &devSino[lenD * cond], devGeom,
                                                            n);
                        } else {
                            projRatio<<<gridD, blockD>>>(&devProj[lenD * cond], &devSino[lenD * cond], devGeom, n);
                        }
                        hipDeviceSynchronize();
                    }
                }

                // backwardProj process
                for (int y = 0; y < sizeV[1]; y++) {
                    hipMemset(devVoxelFactor, 0, sizeof(float) * sizeV[0] * sizeV[1] * NUM_BASIS_VECTOR);
                    hipMemset(devVoxelTmp, 0, sizeof(float) * sizeV[0] * sizeV[1] * NUM_BASIS_VECTOR);
                    for (int cond = 0; cond < NUM_PROJ_COND; cond++) {
                        pbar.update();
                        for (int subOrder = 0; subOrder < subsetSize; subOrder++) {
                            int n = rotation * ((sub + batch * subOrder) % nProj);
                            backwardProjXTT<<<gridV, blockV>>>(&devProj[lenD * cond], devVoxelTmp, devVoxelFactor,
                                                               devGeom, cond, y, n);
                            hipDeviceSynchronize();
                        }
                    }
                    if (method == Method::ART) {
                        voxelPlus<<<gridV, blockV>>>(devVoxel, devVoxelTmp, lambda / (float) subsetSize, devGeom,
                                                     y);
                    } else {
                        voxelProduct<<<gridV, blockV>>>(devVoxel, devVoxelTmp, devVoxelFactor, devGeom, y);
                    }
                    hipDeviceSynchronize();
                }
            }

            loss /= static_cast<float>(NUM_DETECT_V * NUM_DETECT_U * NUM_PROJ);
            hipMemcpy(losses.data() + ep, &loss, sizeof(float), hipMemcpyDeviceToHost); // loss
        }

        for (int y = 0; y < sizeV[1]; y++) {
            voxelSqrt<<<gridV, blockV>>>(devVoxel, devGeom, y);
            hipDeviceSynchronize();
        }

        for (int i = 0; i < NUM_PROJ_COND; i++)
            hipMemcpy(sinogram[i].get(), &devProj[i * lenD], sizeof(float) * lenD, hipMemcpyDeviceToHost);
        for (int i = 0; i < NUM_BASIS_VECTOR; i++)
            hipMemcpy(voxel[i].get(), &devVoxel[i * lenV], sizeof(float) * lenV, hipMemcpyDeviceToHost);

        std::cout << "\ncalculate main direction\n";
        Volume<float> tmp[3];
        for (auto &e: tmp) {
            e = Volume<float>(NUM_VOXEL, NUM_VOXEL, NUM_VOXEL);
        }
        // calc main direction
        for (int z = 0; z < NUM_VOXEL; z++) {
#pragma parallel omp for
            for (int y = 0; y < NUM_VOXEL; y++) {
                for (int x = 0; x < NUM_VOXEL; x++) {
                    calcEigenVector(voxel, md, tmp, y, z, x);
                }
            }
        }

        hipFree(devProj);
        hipFree(devSino);
        hipFree(devVoxel);
        hipFree(devGeom);
        hipFree(devVoxelFactor);
        hipFree(devVoxelTmp);

        std::ofstream ofs("../python/loss.csv");
        for (auto &e: losses)
            ofs << e << ",";
    }

    void
    fiberModelReconstruct(Volume<float> *sinogram, Volume<float> *voxel, const Geometry &geom, int epoch, int batch,
                          Rotate dir,
                          Method method, float lambda) {
        std::cout << "starting reconstruct(XTT), use fiber model..." << std::endl;
        for (int i = 0; i < NUM_BASIS_VECTOR; i++) {
            voxel[i].forEach([](float value) -> float { return 0.01; });
        }

        int rotation = (dir == Rotate::CW) ? 1 : -1;

        int sizeV[3] = {voxel[0].x(), voxel[0].y(), voxel[0].z()};
        int sizeD[3] = {sinogram[0].x(), sinogram[0].y(), sinogram[0].z()};
        int nProj = sizeD[2];

        // hipMalloc
        float *devSino, *devProj, *devVoxel, *devVoxelFactor, *devVoxelTmp;
        const long lenV = sizeV[0] * sizeV[1] * sizeV[2];
        const long lenD = sizeD[0] * sizeD[1] * sizeD[2];

        hipMalloc(&devSino, sizeof(float) * lenD * NUM_PROJ_COND);
        hipMalloc(&devProj, sizeof(float) * lenD * NUM_PROJ_COND); // memory can be small to subsetSize
        hipMalloc(&devVoxel, sizeof(float) * lenV * NUM_BASIS_VECTOR);
        hipMalloc(&devVoxelFactor, sizeof(float) * sizeV[0] * sizeV[1] * NUM_BASIS_VECTOR);
        hipMalloc(&devVoxelTmp, sizeof(float) * sizeV[0] * sizeV[1] * NUM_BASIS_VECTOR);

        for (int i = 0; i < NUM_PROJ_COND; i++)
            hipMemcpy(&devSino[i * lenD], sinogram[i].get(), sizeof(float) * lenD, hipMemcpyHostToDevice);
        for (int i = 0; i < NUM_BASIS_VECTOR; i++)
            hipMemcpy(&devVoxel[i * lenV], voxel[i].get(), sizeof(float) * lenV, hipMemcpyHostToDevice);

        Geometry *devGeom;
        hipMalloc(&devGeom, sizeof(Geometry));
        hipMemcpy(devGeom, &geom, sizeof(Geometry), hipMemcpyHostToDevice);

        // define blocksize
        const int blockSize = 16;
        dim3 blockV(blockSize, blockSize, 1);
        dim3 gridV((sizeV[0] + blockSize - 1) / blockSize, (sizeV[2] + blockSize - 1) / blockSize, 1);
        dim3 blockD(blockSize, blockSize, 1);
        dim3 gridD((sizeD[0] + blockSize - 1) / blockSize, (sizeD[1] + blockSize - 1) / blockSize, 1);

        // forwardProj, divide, backwardProj proj
        int subsetSize = (nProj + batch - 1) / batch;
        std::vector<int> subsetOrder(batch);
        for (int i = 0; i < batch; i++) {
            subsetOrder[i] = i;
        }

        std::vector<float> losses(epoch);

        // progress bar
        progressbar pbar(epoch * batch * NUM_PROJ_COND * (subsetSize + sizeV[1]));

        // set scattering vector direction
        // setScatterDirecOn4D(2.0f * (float) M_PI * scatter_angle_xy / 360.0f, basisVector);

        // main routine
        for (int ep = 0; ep < epoch; ep++) {
            std::mt19937_64 get_rand_mt; // fixed seed
            std::shuffle(subsetOrder.begin(), subsetOrder.end(), get_rand_mt);
            hipMemset(&loss, 0.0f, sizeof(float));
            hipMemset(devProj, 0.0f, sizeof(float) * lenD * NUM_PROJ_COND);
            for (int &sub: subsetOrder) {
                // forwardProj and ratio
                for (int cond = 0; cond < NUM_PROJ_COND; cond++) {
                    for (int subOrder = 0; subOrder < subsetSize; subOrder++) {
                        int n = rotation * ((sub + batch * subOrder) % nProj);
                        // !!care!! judge from vecSod which plane we chose
                        pbar.update();

                        // forwardProj process
                        for (int y = 0; y < sizeV[1]; y++) {
                            // iterate basis vector in forwardProjXTT
                            forwardProjFiber<<<gridV, blockV>>>(&devProj[lenD * cond], devVoxel, devGeom, cond, y,
                                                                n);
                            hipDeviceSynchronize();
                        }

                        // ratio process
                        if (method == Method::ART) {
                            projSubtract<<<gridD, blockD>>>(&devProj[lenD * cond], &devSino[lenD * cond], devGeom,
                                                            n);
                        } else {
                            projRatio<<<gridD, blockD>>>(&devProj[lenD * cond], &devSino[lenD * cond], devGeom, n);
                        }
                        hipDeviceSynchronize();
                    }
                }

                // backwardProj process
                for (int y = 0; y < sizeV[1]; y++) {
                    hipMemset(devVoxelFactor, 0, sizeof(float) * sizeV[0] * sizeV[1] * NUM_BASIS_VECTOR);
                    hipMemset(devVoxelTmp, 0, sizeof(float) * sizeV[0] * sizeV[1] * NUM_BASIS_VECTOR);
                    for (int cond = 0; cond < NUM_PROJ_COND; cond++) {
                        pbar.update();
                        for (int subOrder = 0; subOrder < subsetSize; subOrder++) {
                            int n = rotation * ((sub + batch * subOrder) % nProj);
                            backwardProjFiber<<<gridV, blockV>>>(&devProj[lenD * cond], devVoxel, devVoxelTmp,
                                                                 devVoxelFactor, devGeom, cond, y, n);
                            hipDeviceSynchronize();
                        }
                    }
                    if (method == Method::ART) {
                        voxelPlus<<<gridV, blockV>>>(devVoxel, devVoxelTmp, lambda / (float) subsetSize, devGeom,
                                                     y);
                    } else {
                        voxelProduct<<<gridV, blockV>>>(devVoxel, devVoxelTmp, devVoxelFactor, devGeom, y);
                    }
                    hipDeviceSynchronize();
                }
            }

            loss /= static_cast<float>(NUM_DETECT_V * NUM_DETECT_U * NUM_PROJ);
            hipMemcpy(losses.data() + ep, &loss, sizeof(float), hipMemcpyDeviceToHost); // loss
        }

        for (int i = 0; i < NUM_PROJ_COND; i++)
            hipMemcpy(sinogram[i].get(), &devProj[i * lenD], sizeof(float) * lenD, hipMemcpyDeviceToHost);
        for (int i = 0; i < NUM_BASIS_VECTOR; i++)
            hipMemcpy(voxel[i].get(), &devVoxel[i * lenV], sizeof(float) * lenV, hipMemcpyDeviceToHost);

        hipFree(devProj);
        hipFree(devSino);
        hipFree(devVoxel);
        hipFree(devGeom);
        hipFree(devVoxelFactor);
        hipFree(devVoxelTmp);

        std::ofstream ofs("../python/loss.csv");
        for (auto &e: losses)
            ofs << e << ",";
    }
}

namespace FDK {
    void reconstruct(Volume<float> *sinogram, Volume<float> *voxel, const Geometry &geom, Rotate dir) {
        std::cout << "starting reconstruct(FDK)..." << std::endl;
        for (int i = 0; i < NUM_BASIS_VECTOR; i++) {
            voxel[i].forEach([](float value) -> float { return 0.0; });
        }

        int rotation = (dir == Rotate::CW) ? 1 : -1;

        int sizeV[3] = {voxel[0].x(), voxel[0].y(), voxel[0].z()};
        int sizeD[3] = {sinogram[0].x(), sinogram[0].y(), sinogram[0].z()};
        int nProj = sizeD[2];

        // hipMalloc
        float *devSino, *devSinoFilt, *devVoxel, *weight, *filt;
        const long lenV = sizeV[0] * sizeV[1] * sizeV[2];
        const long lenD = sizeD[0] * sizeD[1] * sizeD[2];

        hipMalloc(&devSino, sizeof(float) * lenD * NUM_PROJ_COND);
        hipMalloc(&devSinoFilt, sizeof(float) * lenD * NUM_PROJ_COND); // memory can be small to subsetSize
        hipMalloc(&devVoxel, sizeof(float) * lenV * NUM_BASIS_VECTOR);
        hipMalloc(&weight, sizeof(float) * sizeD[0] * sizeD[1]);
        hipMallocManaged(&filt, sizeof(float) * geom.detect);

        for (int i = 0; i < NUM_PROJ_COND; i++)
            hipMemcpy(&devSino[i * lenD], sinogram[i].get(), sizeof(float) * lenD, hipMemcpyHostToDevice);

        Geometry *devGeom;
        hipMalloc(&devGeom, sizeof(Geometry));
        hipMemcpy(devGeom, &geom, sizeof(Geometry), hipMemcpyHostToDevice);

        // define blocksize
        const int blockSize = 16;
        dim3 blockV(blockSize, blockSize, 1);
        dim3 gridV((sizeV[0] + blockSize - 1) / blockSize, (sizeV[2] + blockSize - 1) / blockSize, 1);
        dim3 blockD(blockSize, blockSize, 1);
        dim3 gridD((sizeD[0] + blockSize - 1) / blockSize, (sizeD[1] + blockSize - 1) / blockSize, 1);

        // progress bar

        progressbar pbar(nProj);
        calcWeight<<<gridD, blockD>>>(weight, devGeom);
        hipDeviceSynchronize();
        // make Shepp-Logan fliter

        float d = geom.detSize * (geom.sod / geom.sdd);
        // float d = geom.detSize * (geom.sod / geom.sdd);
        for (int v = 0; v < geom.detect; v++) {
            filt[v] = 1.0f / (float) (M_PI * M_PI * d * (1.0f - 4.0f * (float) (v * v)));
        }

        for (int cond = 0; cond < NUM_PROJ_COND; cond++) {
            for (int n = 0; n < nProj; n++) {
                // convolution
                // hogeTmpWakaran<<<gridD, blockD>>>();
                projConv<<<gridD, blockD>>>(&devSinoFilt[lenD * cond], &devSino[lenD * cond], devGeom, n, filt,
                                            weight);
                hipDeviceSynchronize();
                for (int y = 0; y < geom.voxel; y++) {
                    filteredBackProj<<<gridV, blockV>>>(devSinoFilt, devVoxel, devGeom, cond, y, rotation * n);
                }
            }
        }

        for (int i = 0; i < NUM_PROJ_COND; i++)
            hipMemcpy(sinogram[i].get(), &devSinoFilt[i * lenD], sizeof(float) * lenD, hipMemcpyDeviceToHost);
        for (int i = 0; i < NUM_BASIS_VECTOR; i++)
            hipMemcpy(voxel[i].get(), &devVoxel[i * lenV], sizeof(float) * lenV, hipMemcpyDeviceToHost);

        hipFree(devSinoFilt);
        hipFree(devSino);
        hipFree(devVoxel);
        hipFree(devGeom);
        hipFree(filt);
        hipFree(weight);
    }
}

void forwardProjOnly(Volume<float> *sinogram, Volume<float> *voxel, const Geometry &geom, Rotate dir) {
    std::cout << "starting forward projection..." << std::endl;

    int rotation = (dir == Rotate::CW) ? 1 : -1;

    int sizeV[3] = {voxel[0].x(), voxel[0].y(), voxel[0].z()};
    int sizeD[3] = {sinogram[0].x(), sinogram[0].y(), sinogram[0].z()};
    int nProj = sizeD[2];

    // hipMalloc
    float *devProj, *devVoxel;
    const long lenV = sizeV[0] * sizeV[1] * sizeV[2];
    const long lenD = sizeD[0] * sizeD[1] * sizeD[2];

    hipMalloc(&devProj, sizeof(float) * lenD * NUM_PROJ_COND); // memory can be small to subsetSize
    hipMalloc(&devVoxel, sizeof(float) * lenV * NUM_BASIS_VECTOR);

    for (int i = 0; i < NUM_BASIS_VECTOR; i++)
        hipMemcpy(&devVoxel[i * lenV], voxel[i].get(), sizeof(float) * lenV, hipMemcpyHostToDevice);

    Geometry *devGeom;
    hipMalloc(&devGeom, sizeof(Geometry));
    hipMemcpy(devGeom, &geom, sizeof(Geometry), hipMemcpyHostToDevice);

    // define blocksize
    const int blockSize = 16;
    dim3 blockV(blockSize, blockSize, 1);
    dim3 gridV((sizeV[0] + blockSize - 1) / blockSize, (sizeV[2] + blockSize - 1) / blockSize, 1);

    // forwardProj, divide, backwardProj proj
    // progress bar
    progressbar pbar(NUM_PROJ * NUM_PROJ_COND);

    // set scattering vector direction
    // setScatterDirecOn4D(2.0f * (float) M_PI * scatter_angle_xy / 360.0f, basisVector);

    // main routine
    hipMemset(devProj, 0.0f, sizeof(float) * lenD * NUM_PROJ_COND);
    // forwardProj and ratio
    for (int cond = 0; cond < NUM_PROJ_COND; cond++) {
        for (int n = 0; n < NUM_PROJ; n++) {
            // !!care!! judge from vecSod which plane we chose
            pbar.update();
            // forwardProj process
            for (int y = 0; y < sizeV[1]; y++) {
                forwardProj<<<gridV, blockV>>>(&devProj[lenD * cond], devVoxel, devGeom, cond, y, n * rotation);
                hipDeviceSynchronize();
            }
        }
    }

    for (int i = 0; i < NUM_PROJ_COND; i++)
        hipMemcpy(sinogram[i].get(), &devProj[i * lenD], sizeof(float) * lenD, hipMemcpyDeviceToHost);
    for (int i = 0; i < NUM_BASIS_VECTOR; i++)
        hipMemcpy(voxel[i].get(), &devVoxel[i * lenV], sizeof(float) * lenV, hipMemcpyDeviceToHost);

    hipFree(devProj);
    hipFree(devVoxel);
    hipFree(devGeom);
}

void compareXYZTensorVolume(Volume<float> *voxel, const Geometry &geom) {
    for (int i = 0; i < geom.voxel; i++) {
        for (int j = 0; j < geom.voxel; j++) {
            for (int k = 0; k < geom.voxel; k++) {
                float min = voxel[0](i, j, k);
                int idx = 0;
                for (int n = 1; n < NUM_BASIS_VECTOR; n++) {
                    if (min > voxel[n](i, j, k)) {
                        min = voxel[n](i, j, k);
                        idx = n;
                    }
                }
                for (int n = 0; n < NUM_BASIS_VECTOR; n++) {
                    if (n != idx) {
                        voxel[n](i, j, k) = 0.0f;
                    }
                }
            }
        }
    }
}

/*
__host__ void
reconstructDebugHost(Volume<float> &sinogram, Volume<float> &voxel, const Geometry &geom, const int epoch,
                     const int batch, bool dir) {

    printf("pass");
    CudaVolume<float> sino(sinogram);
    CudaVolume<float> vox(voxel);

    int sizeV[3] = {voxel.x(), voxel.y(), voxel.z()};
    int sizeD[3] = {sinogram.x(), sinogram.y(), sinogram.z()};
    int nProj = sizeD[2];


    // forward, divide, backwardProj proj
    int subsetSize = (nProj + batch - 1) / batch;
    std::vector<int> subsetOrder(batch);
    for (int i = 0; i < batch; i++) {
        subsetOrder[i] = i;
    }

    std::mt19937_64 get_rand_mt; // fixed seed
    std::shuffle(subsetOrder.begin(), subsetOrder.end(), get_rand_mt);

    // main routine
    for (int ep = 0; ep < epoch; ep++) {
        // forward
        for (int n = 15; n < nProj; n++) {

            // forwardProj
            for (int x = 0; x < sizeV[0]; x++) {
                for (int y = 0; y < sizeV[1]; y++) {
                    for (int z = 0; z < sizeV[2]; z++) {
                        int coord[4] = {x, y, z, n};
                        forwardXTTonDevice(coord, sino, &vox, geom);
                    }
                }
            }
        }
    }
}
 */
