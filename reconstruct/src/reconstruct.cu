#include "hip/hip_runtime.h"
//
// Created by tomokimori on 22/08/30.
//
#include <geometry.h>
#include <ir.cuh>
#include <fdk.cuh>
#include <fiber.cuh>
#include <random>
#include <memory>
#include <progressbar.h>
#include <params.h>
#include <volume.h>
#include <algorithm>
#include <omp.h>
#include <pca.cuh>
#include <reconstruct.cuh>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <omp.h>
#include "quadfilt.h"

namespace IR {
    void
    reconstruct(Volume<float> *sinogram, Volume<float> *voxel, const Geometry &geom, int epoch, int batch, Rotate dir,
                Method method, float lambda) {
        std::cout << "starting reconstruct(IR)..." << std::endl;

        int rotation = (dir == Rotate::CW) ? 1 : -1;

        int64_t sizeV[3] = {voxel[0].x(), voxel[0].y(), voxel[0].z()};
        int64_t sizeD[3] = {sinogram[0].x(), sinogram[0].y(), sinogram[0].z()};
        int64_t nProj = sizeD[2];

        // hipMalloc
        float *devSino, *devProj, *devVoxel, *devVoxelFactor, *devVoxelTmp;
        const int64_t lenV = sizeV[0] * sizeV[1] * sizeV[2];
        const int64_t lenD = sizeD[0] * sizeD[1] * sizeD[2];

        hipMalloc(&devSino, sizeof(float) * lenD);
        hipMalloc(&devProj, sizeof(float) * lenD); // memory can be small to subsetSize
        hipMalloc(&devVoxel, sizeof(float) * lenV);
        hipMalloc(&devVoxelFactor, sizeof(float) * sizeV[0] * sizeV[1]);
        hipMalloc(&devVoxelTmp, sizeof(float) * sizeV[0] * sizeV[1]);

        float *loss1;
        hipMalloc(&loss1, sizeof(float));

        Geometry *devGeom;
        hipMalloc(&devGeom, sizeof(Geometry));
        hipMemcpy(devGeom, &geom, sizeof(Geometry), hipMemcpyHostToDevice);

        // define blocksize
        dim3 blockV(BLOCK_SIZE, BLOCK_SIZE, 1);
        dim3 gridV((sizeV[0] + BLOCK_SIZE - 1) / BLOCK_SIZE, (sizeV[2] + BLOCK_SIZE - 1) / BLOCK_SIZE, 1);
        dim3 blockD(BLOCK_SIZE, BLOCK_SIZE, 1);
        dim3 gridD((sizeD[0] + BLOCK_SIZE - 1) / BLOCK_SIZE, (sizeD[1] + BLOCK_SIZE - 1) / BLOCK_SIZE, 1);

        // forwardProj, divide, backwardProj proj
        int subsetSize = (nProj + batch - 1) / batch;
        std::vector<int> subsetOrder(batch);
        for (int i = 0; i < batch; i++) {
            subsetOrder[i] = i;
        }

        std::vector<float> losses(epoch);

        // progress bar
        progressbar pbar(epoch * batch * NUM_PROJ_COND * (subsetSize + sizeV[1]));

        // set scattering vector direction
        // setScatterDirecOn4D(2.0f * (float) M_PI * scatter_angle_xy / 360.0f, basisVector);

        // main routine
        std::mt19937_64 get_rand_mt; // fixed seed
        for (int cond = 0; cond < NUM_PROJ_COND; cond++) {
            hipMemcpy(devVoxel, voxel[cond].get(), sizeof(float) * lenV, hipMemcpyHostToDevice);
            hipMemcpy(devSino, sinogram[cond].get(), sizeof(float) * lenD, hipMemcpyHostToDevice);

            for (int ep = 0; ep < epoch; ep++) {
                std::shuffle(subsetOrder.begin(), subsetOrder.end(), get_rand_mt);
                hipMemset(devProj, 0.0f, sizeof(float) * lenD);
                for (int &sub: subsetOrder) {
                    // forwardProj and ratio
                    for (int subOrder = 0; subOrder < subsetSize; subOrder++) {
                        int n = rotation * ((sub + batch * subOrder) % nProj);
                        // !!care!! judge from vecSod which plane we chose
                        pbar.update();

                        // forwardProj process
                        for (int y = 0; y < sizeV[1]; y++) {
                            forwardProj<<<gridV, blockV>>>(devProj, devVoxel, devGeom, cond, y, n);
                            hipDeviceSynchronize();
                        }

                        // ratio process
                        if (method == Method::ART) {
                            projSubtract<<<gridD, blockD>>>(devProj, devSino, devGeom, n, loss1);
                        } else {
                            projRatio<<<gridD, blockD>>>(devProj, devSino, devGeom, n, loss1);
                        }
                        hipDeviceSynchronize();
                    }

                    // backwardProj process
                    for (int y = 0; y < sizeV[1]; y++) {
                        hipMemset(devVoxelFactor, 0, sizeof(float) * sizeV[0] * sizeV[1]);
                        hipMemset(devVoxelTmp, 0, sizeof(float) * sizeV[0] * sizeV[1]);

                        pbar.update();
                        for (int subOrder = 0; subOrder < subsetSize; subOrder++) {
                            int n = rotation * ((sub + batch * subOrder) % nProj);
                            backwardProj<<<gridV, blockV>>>(devProj, devVoxelTmp, devVoxelFactor, devGeom, cond, y, n);
                            hipDeviceSynchronize();
                        }

                        if (method == Method::ART) {
                            voxelPlus<<<gridV, blockV>>>(devVoxel, devVoxelTmp, lambda / (float) subsetSize, devGeom, y);
                        } else {
                            voxelProduct<<<gridV, blockV>>>(devVoxel, devVoxelTmp, devVoxelFactor, devGeom, y);
                        }
                        hipDeviceSynchronize();
                    }
                }
                // hipMemcpy(losses.data() + ep, loss1, sizeof(float), hipMemcpyDeviceToHost); // loss
            }
            hipMemcpy(voxel[cond].get(), devVoxel, sizeof(float) * lenV, hipMemcpyDeviceToHost);
        }

        hipFree(devProj);
        hipFree(devSino);
        hipFree(devVoxel);
        hipFree(devGeom);
        hipFree(devVoxelFactor);
        hipFree(devVoxelTmp);
    }
}

namespace XTT {
    void orthReconstruct(Volume<float> *sinogram, Volume<float> voxel[3], Volume<float> md[3], const Geometry &geom,
                         int iter1, int iter2, int batch, Rotate dir, Method method, float lambda) {
        std::cout << "starting reconstruct(orth)..." << std::endl;

        // int rotation = (dir == Rotate::CW) ? -1 : 1;
        int rotation = (dir == Rotate::CW) ? 1 : -1;

        int sizeV[3] = {voxel[0].x(), voxel[0].y(), voxel[0].z()};
        int sizeD[3] = {sinogram[0].x(), sinogram[0].y(), sinogram[0].z()};
        int nProj = sizeD[2];

        // hipMalloc
        float *devSino, *devProj, *devVoxel, *devVoxelFactor, *devVoxelTmp, *devDirection;
        const long lenV = sizeV[0] * sizeV[1] * sizeV[2];
        const long lenD = sizeD[0] * sizeD[1] * sizeD[2];

        hipMalloc(&devSino, sizeof(float) * lenD * NUM_PROJ_COND);
        hipMalloc(&devProj, sizeof(float) * lenD * NUM_PROJ_COND); // memory can be small to subsetSize
        hipMalloc(&devVoxel, sizeof(float) * lenV * NUM_BASIS_VECTOR);
        hipMalloc(&devVoxelFactor, sizeof(float) * sizeV[0] * sizeV[1] * NUM_BASIS_VECTOR);
        hipMalloc(&devVoxelTmp, sizeof(float) * sizeV[0] * sizeV[1] * NUM_BASIS_VECTOR);

        for (int i = 0; i < NUM_PROJ_COND; i++)
            hipMemcpy(&devSino[i * lenD], sinogram[i].get(), sizeof(float) * lenD, hipMemcpyHostToDevice);
        for (int i = 0; i < NUM_BASIS_VECTOR; i++)
            hipMemcpy(&devVoxel[i * lenV], voxel[i].get(), sizeof(float) * lenV, hipMemcpyHostToDevice);

        // store theta, phi on polar coordination to devDirection
        float *devCoef, *devCoefTmp;
        hipMalloc(&devCoef, sizeof(float) * lenV * 2);
        hipMalloc(&devCoefTmp, sizeof(float) * lenV * 2);
        Volume<float> coef[2];
        for (auto &co: coef)
            co = Volume<float>(NUM_VOXEL, NUM_VOXEL, NUM_VOXEL);

        // !!!!!!!!!!!!!!!!!!!!!!!!!!
        coef[0].forEach([](float value) -> float { return 0.0f; });
        coef[1].forEach([](float value) -> float { return 1.0f; });
        // coef[0].forEach([](float value) -> float { return 3.0 * M_PI / 4.0f; });
        // coef[1].forEach([](float value) -> float { return std::cos(M_PI / 4.0f); });
        hipMemcpy(&devCoef[0], coef[0].get(), sizeof(float) * lenV, hipMemcpyHostToDevice);
        hipMemcpy(&devCoef[lenV], coef[1].get(), sizeof(float) * lenV, hipMemcpyHostToDevice);

        Geometry *devGeom;
        hipMalloc(&devGeom, sizeof(Geometry));
        hipMemcpy(devGeom, &geom, sizeof(Geometry), hipMemcpyHostToDevice);

        // define blocksize
        dim3 blockV(BLOCK_SIZE, BLOCK_SIZE, 1);
        dim3 gridV((sizeV[0] + BLOCK_SIZE - 1) / BLOCK_SIZE, (sizeV[2] + BLOCK_SIZE - 1) / BLOCK_SIZE, 1);
        dim3 blockD(BLOCK_SIZE, BLOCK_SIZE, 1);
        dim3 gridD((sizeD[0] + BLOCK_SIZE - 1) / BLOCK_SIZE, (sizeD[1] + BLOCK_SIZE - 1) / BLOCK_SIZE, 1);

        hiprandState *devStates;
        int threadNum = BLOCK_SIZE * (int) ((sizeV[0] + BLOCK_SIZE - 1) / BLOCK_SIZE);
        hipMalloc((void **) (&devStates), threadNum * threadNum * threadNum * sizeof(hiprandState));
        setup_rand<<<gridV, blockV>>>(devStates, threadNum, 0);
        // forwardProj, divide, backwardProj proj
        int subsetSize = (nProj + batch - 1) / batch;
        std::vector<int> subsetOrder(batch);
        for (int i = 0; i < batch; i++) {
            subsetOrder[i] = i;
        }

        // progress bar
        progressbar pbar(iter1 * iter2 * batch * NUM_PROJ_COND * (subsetSize + sizeV[1]));

        // set scattering vector direction
        // setScatterDirecOn4D(2.0f * (float) M_PI * scatter_angle_xy / 360.0f, basisVector);
        std::random_device seed_gen;
        std::mt19937 engine(seed_gen());
        std::uniform_real_distribution<float> dist(0.f, 1.f);

        Volume<float> loss_map1 = Volume<float>(NUM_VOXEL, NUM_VOXEL, NUM_VOXEL);
        Volume<float> loss_map2 = Volume<float>(NUM_VOXEL, NUM_VOXEL, NUM_VOXEL);
        float *devLoss1;
        float *devLoss2;

        hipMalloc(&devLoss1, sizeof(float));
        hipMalloc(&devLoss2, sizeof(float) * lenV);

        std::vector<float> proj_loss(iter1 * iter2);
        std::vector<float> norm_loss(iter1);
        Volume<float> tmp[3];
        for (auto &e: tmp) {
            e = Volume<float>(NUM_VOXEL, NUM_VOXEL, NUM_VOXEL);
        }

        // main routine
        for (int ep1 = 0; ep1 < iter1; ep1++) {
            for (int i = 0; i < 3; i++) {
                voxel[i].forEach([](float value) -> float { return 0.0f; });
                hipMemcpy(&devVoxel[i * lenV], voxel[i].get(), sizeof(float) * lenV, hipMemcpyHostToDevice);
            }
            hipMemset(devLoss2, 0.0f, sizeof(float) * lenV);
            float judge = dist(engine);

            for (int ep2 = 0; ep2 < iter2; ep2++) {
                std::shuffle(subsetOrder.begin(), subsetOrder.end(), engine);
                hipMemset(devLoss1, 0.0f, sizeof(float));
                hipMemset(devProj, 0.0f, sizeof(float) * lenD * NUM_PROJ_COND);
                for (int &sub: subsetOrder) {
                    // forwardProj and ratio
                    for (int cond = 0; cond < NUM_PROJ_COND; cond++) {
                        for (int subOrder = 0; subOrder < subsetSize; subOrder++) {
                            int n = rotation * ((sub + batch * subOrder) % nProj);
                            // !!care!! judge from vecSod which plane we chose
                            pbar.update();

                            // forwardProj process
                            for (int y = 0; y < sizeV[1]; y++) {
                                // 回転行列に従って3方向散乱係数の順投影
                                forwardOrth<<<gridV, blockV>>>(&devProj[lenD * cond], devVoxel, devCoef,
                                                               cond, y, n, ep1, devGeom);
                                hipDeviceSynchronize();
                            }

                            // ratio process
                            if (method == Method::ART)
                                projSubtract<<<gridD, blockD>>>(&devProj[lenD * cond],
                                                                &devSino[lenD * cond], devGeom, n, devLoss1);
                            else
                                projRatio<<<gridD, blockD>>>(&devProj[lenD * cond], &devSino[lenD * cond],
                                                             devGeom, n, devLoss1);
                            hipDeviceSynchronize();
                        }
                    }

                    // backwardProj process
                    for (int y = 0; y < sizeV[1]; y++) {
                        hipMemset(devVoxelFactor, 0, sizeof(float) * sizeV[0] * sizeV[1] * NUM_BASIS_VECTOR);
                        hipMemset(devVoxelTmp, 0, sizeof(float) * sizeV[0] * sizeV[1] * NUM_BASIS_VECTOR);
                        for (int cond = 0; cond < NUM_PROJ_COND; cond++) {
                            pbar.update();
                            for (int subOrder = 0; subOrder < subsetSize; subOrder++) {
                                int n = rotation * ((sub + batch * subOrder) % nProj);

                                backwardOrth<<<gridV, blockV>>>(&devProj[lenD * cond], devCoef, devVoxelTmp,
                                                                devVoxelFactor, devGeom, cond, y, n, ep1);
                                hipDeviceSynchronize();
                            }
                        }
                        if (method == Method::ART) {
                            voxelPlus<<<gridV, blockV>>>(devVoxel, devVoxelTmp, lambda / (float) subsetSize,
                                                         devGeom, y);
                        } else {
                            voxelProduct<<<gridV, blockV>>>(devVoxel, devVoxelTmp, devVoxelFactor, devGeom, y);
                        }
                        hipDeviceSynchronize();
                    }
                }
                hipMemcpy(proj_loss.data() + ep1 * iter2 + ep2, devLoss1, sizeof(float),
                           hipMemcpyDeviceToHost); // loss
                // std::cout << proj_loss[ep2 * (ep1 + 1)] << std::endl;
                // ----- end iter1 ----- //
            }
            /*
            for (int i = 0; i < NUM_BASIS_VECTOR; i++) {
                std::string savefilePathCT =
                        "../volume_bin/cfrp_xyz7_mark/orth_" + std::to_string(ep1) + "_" + std::to_string(i + 1) + "_" +
                        // "../volume_bin/cfrp_xyz7/xtt" + std::to_string(i + 1) + "_" +
                        std::to_string(NUM_VOXEL) + "x" +
                        std::to_string(NUM_VOXEL) + "x" + std::to_string(NUM_VOXEL) + ".raw";
                voxel[i].save(savefilePathCT);
            }
             */

            // swap later
            for (int y = 0; y < sizeV[1]; y++) {
                voxelSqrt<<<gridV, blockV>>>(devVoxel, devGeom, y);
                hipDeviceSynchronize();
            }
            for (int i = 0; i < NUM_BASIS_VECTOR; i++)
                hipMemcpy(voxel[i].get(), &devVoxel[i * lenV], sizeof(float) * lenV, hipMemcpyDeviceToHost);

            for (int y = 0; y < sizeV[1]; y++) {
                // calcNormalVector<<<gridV, blockV>>>(devVoxel, devCoef, y, ep1, devGeom, devLoss2);
                calcNormalVectorThreeDirec<<<gridV, blockV>>>(devVoxel, devCoef, y, ep1, devGeom, devLoss2, devStates,
                                                              judge);
                hipDeviceSynchronize();
            }
            std::string xyz[] = {"x", "y", "z"};

            for (int i = 0; i < 2; i++) {
                hipMemcpy(coef[i].get(), &devCoef[i * lenV], sizeof(float) * lenV, hipMemcpyDeviceToHost);
            }
            convertNormVector(voxel, md, coef);
            for (int i = 0; i < 3; i++) {
                std::string savefilePathCT =
                        // "../volume_bin/cfrp_xyz7_mark/pca/main_direction_orth_art_5proj" + std::to_string(i + 1) + "_" +
                        "../volume_bin/cfrp_xyz7_13axis/sequence/pca/md_nofilt3_art" +
                        // "../volume_bin/simulation/sequence_13axis/pca/+x+y+z_filt_rand_all" +
                        std::to_string(ep1 + 1) + "_" + xyz[i] + "_" + std::to_string(NUM_VOXEL) + "x" +
                        std::to_string(NUM_VOXEL) + "x" + std::to_string(NUM_VOXEL) + ".raw";
                md[i].save(savefilePathCT);
            }

            for (int filt = 0; filt < 3; filt++) {
                for (int y = 1; y < sizeV[1] - 1; y++) {
                    meanFiltFiber<<<gridV, blockV>>>(devCoef, devCoefTmp, devVoxel, devGeom, y,
                                                     1.0f);
                    hipDeviceSynchronize();
                }
                hipMemcpy(devCoef, devCoefTmp, sizeof(float) * lenV * 2, hipMemcpyDeviceToDevice);
            }

            // calc main direction
            /*
            for (int z = 0; z < NUM_VOXEL; z++) {
#pragma parallel omp for
                for (int y = 0; y < NUM_VOXEL; y++) {
                    for (int x = 0; x < NUM_VOXEL; x++) {
                        calcEigenVector(voxel, md, tmp, x, y, z);
                    }
                }
            }

            for (int i = 0; i < 3; i++)
                hipMemcpy(&devDirection[i * lenV], md[i].get(), sizeof(float) * lenV, hipMemcpyHostToDevice);

            for (int y = 0; y < sizeV[1]; y++) {
                calcRotation<<<gridV, blockV>>>(devDirection, devCoef, y, devGeom, devLoss2);
                hipDeviceSynchronize();
            }
             */

            hipMemcpy(loss_map2.get(), devLoss2, sizeof(float) * lenV, hipMemcpyDeviceToHost);
            norm_loss[ep1] = loss_map2.mean();
            // ----- end iter2 -----

            for (int i = 0; i < NUM_PROJ_COND; i++)
                hipMemcpy(sinogram[i].get(), &devProj[i * lenD], sizeof(float) * lenD, hipMemcpyDeviceToHost);

            for (int i = 0; i < 2; i++) {
                hipMemcpy(coef[i].get(), &devCoef[i * lenV], sizeof(float) * lenV, hipMemcpyDeviceToHost);
            }

            convertNormVector(voxel, md, coef);
            // save direction volume
            for (int i = 0; i < 3; i++) {
                std::string savefilePathCT =
                        // "../volume_bin/cfrp_xyz7_mark/pca/main_direction_orth_art_5proj" + std::to_string(i + 1) + "_" +
                        "../volume_bin/cfrp_xyz7_13axis/sequence/pca/md_filt3_art" +
                        // "../volume_bin/simulation/sequence_13axis/pca/+x+y+z_filt_rand_all" +
                        std::to_string(ep1 + 1) + "_" + xyz[i] + "_" + std::to_string(NUM_VOXEL) + "x" +
                        std::to_string(NUM_VOXEL) + "x" + std::to_string(NUM_VOXEL) + ".raw";
                md[i].save(savefilePathCT);
            }

            // save ct volume
            for (int i = 0; i < NUM_BASIS_VECTOR; i++) {
                std::string savefilePathCT =
                        // "../volume_bin/simulation/sequence_13axis/+x+y+z_filt_rand_all" + std::to_string(ep1) +
                        // "../volume_bin/cfrp_xyz7_mark/sequence/direc_discrete_iter" + std::to_string(ep1) +
                        "../volume_bin/cfrp_xyz7_13axis/sequence/volume_filt3_art" + std::to_string(ep1 + 1) +
                        "_orth" + std::to_string(i + 1) + "_" + std::to_string(NUM_VOXEL) + "x" +
                        std::to_string(NUM_VOXEL) + "x" + std::to_string(NUM_VOXEL) + ".raw";
                voxel[i].save(savefilePathCT);
            }
        }
        /*
        for (int i = 0; i < NUM_PROJ_COND; i++)
            hipMemcpy(sinogram[i].get(), &devProj[i * lenD], sizeof(float) * lenD, hipMemcpyDeviceToHost);
        for (int i = 0; i < NUM_BASIS_VECTOR; i++)
            hipMemcpy(voxel[i].get(), &devVoxel[i * lenV], sizeof(float) * lenV, hipMemcpyDeviceToHost);

        Volume<float> coef[5];
        for (int i = 0; i < 5; i++) {
            coef[i] = Volume<float>(NUM_VOXEL, NUM_VOXEL, NUM_VOXEL);
            hipMemcpy(coef[i].get(), &devCoef[i * lenV], sizeof(float) * lenV, hipMemcpyDeviceToHost);
        }
        convertNormVector(voxel, md, coef);
        */

        /* loss
        Volume<float> loss_norm;
        loss_norm = Volume<float>(NUM_VOXEL, NUM_VOXEL, NUM_VOXEL);
        hipMemcpy(loss_norm.get(), devLoss2, sizeof(float) * lenV, hipMemcpyDeviceToHost);
        loss_norm.save("../volume_bin/cfrp_xyz7_mark/orth_loss.raw");
        */
        // need convert phi, theta to direction(size<-mu1 + mu2 / 2)

        hipFree(devProj);
        hipFree(devSino);
        hipFree(devVoxel);
        hipFree(devGeom);
        hipFree(devVoxelFactor);
        hipFree(devVoxelTmp);
        hipFree(devCoef);
        hipFree(devLoss1);
        hipFree(devLoss2);
        hipFree(devCoefTmp);
        hipFree(devStates);

        std::ofstream ofs1("../python/loss1.csv");
        std::ofstream ofs2("../python/loss2.csv");
        for (auto &e: proj_loss)
            ofs1 << e / static_cast<float>(NUM_DETECT_V * NUM_DETECT_U * NUM_PROJ * NUM_PROJ_COND) << ",";
        for (auto &e: norm_loss)
            ofs2 << e << ",";
    }

    void newReconstruct(Volume<float> *sinogram, Volume<float> *voxel, Volume<float> *md, const Geometry &geom,
                        int iter1, int iter2, int batch, Rotate dir, Method method, float lambda) {
        std::cout << "starting reconstruct(XTT)..." << std::endl;
        if (method == Method::MLEM) {
            for (int i = 0; i < NUM_BASIS_VECTOR; i++) {
                voxel[i].forEach([](float value) -> float { return 0.01; });
            }
        }

        // int rotation = (dir == Rotate::CW) ? -1 : 1;
        int rotation = (dir == Rotate::CW) ? 1 : -1;

        int sizeV[3] = {voxel[0].x(), voxel[0].y(), voxel[0].z()};
        int sizeD[3] = {sinogram[0].x(), sinogram[0].y(), sinogram[0].z()};
        int nProj = sizeD[2];

        // hipMalloc
        float *devSino, *devProj, *devVoxel, *hostVoxel, *devVoxelFactor, *devVoxelTmp;
        const long lenV = sizeV[0] * sizeV[1] * sizeV[2];
        const long lenD = sizeD[0] * sizeD[1] * sizeD[2];

        hipMalloc(&devSino, sizeof(float) * lenD * NUM_PROJ_COND);
        hipMalloc(&devProj, sizeof(float) * lenD * NUM_PROJ_COND); // memory can be small to subsetSize
        hipMalloc(&devVoxel, sizeof(float) * lenV * NUM_BASIS_VECTOR);
        hipMalloc(&hostVoxel, sizeof(float) * lenV * NUM_BASIS_VECTOR);
        hipMalloc(&devVoxelFactor, sizeof(float) * sizeV[0] * sizeV[1] * NUM_BASIS_VECTOR);
        hipMalloc(&devVoxelTmp, sizeof(float) * sizeV[0] * sizeV[1] * NUM_BASIS_VECTOR);

        // direction, phi, theta
        float *devDirection;
        hipMalloc(&devDirection, sizeof(float) * lenV * 2);
        hipMemset(devDirection, 0.0f, sizeof(float) * lenV * 2);
        for (int i = 0; i < 3; i++) {
            md[i] = Volume<float>(NUM_VOXEL, NUM_VOXEL, NUM_VOXEL);
        }

        for (int i = 0; i < NUM_PROJ_COND; i++)
            hipMemcpy(&devSino[i * lenD], sinogram[i].get(), sizeof(float) * lenD, hipMemcpyHostToDevice);
        for (int i = 0; i < NUM_BASIS_VECTOR; i++)
            hipMemcpy(&devVoxel[i * lenV], voxel[i].get(), sizeof(float) * lenV, hipMemcpyHostToDevice);

        Geometry *devGeom;
        hipMalloc(&devGeom, sizeof(Geometry));
        hipMemcpy(devGeom, &geom, sizeof(Geometry), hipMemcpyHostToDevice);

        // define blocksize
        dim3 blockV(BLOCK_SIZE, BLOCK_SIZE, 1);
        dim3 gridV((sizeV[0] + BLOCK_SIZE - 1) / BLOCK_SIZE, (sizeV[2] + BLOCK_SIZE - 1) / BLOCK_SIZE, 1);
        dim3 blockD(BLOCK_SIZE, BLOCK_SIZE, 1);
        dim3 gridD((sizeD[0] + BLOCK_SIZE - 1) / BLOCK_SIZE, (sizeD[1] + BLOCK_SIZE - 1) / BLOCK_SIZE, 1);

        // forwardProj, divide, backwardProj proj
        int subsetSize = (nProj + batch - 1) / batch;
        std::vector<int> subsetOrder(batch);
        for (int i = 0; i < batch; i++) {
            subsetOrder[i] = i;
        }

        std::vector<float> losses(iter1 * iter2);

        // progress bar
        progressbar pbar(iter1 * iter2 * batch * NUM_PROJ_COND * (subsetSize + sizeV[1]));

        // set scattering vector direction
        // setScatterDirecOn4D(2.0f * (float) M_PI * scatter_angle_xy / 360.0f, basisVector);

        // main routine
        for (int it2 = 0; it2 < iter1; it2++) {
            for (int ep = 0; ep < iter2; ep++) {
                std::mt19937_64 get_rand_mt; // fixed seed
                std::shuffle(subsetOrder.begin(), subsetOrder.end(), get_rand_mt);
                hipMemset(&d_loss_proj, 0.0f, sizeof(float));
                hipMemset(devProj, 0.0f, sizeof(float) * lenD * NUM_PROJ_COND);

                for (int &sub: subsetOrder) {
                    // forwardProj and ratio
                    for (int cond = 0; cond < NUM_PROJ_COND; cond++) {
                        for (int subOrder = 0; subOrder < subsetSize; subOrder++) {
                            int n = rotation * ((sub + batch * subOrder) % nProj);
                            // !!care!! judge from vecSod which plane we chose
                            pbar.update();

                            // forwardProj process
                            for (int y = 0; y < sizeV[1]; y++) {
                                // iterate basis vector in forwardProjXTT
                                forwardProjXTTbyFiber<<<gridV, blockV>>>(&devProj[lenD * cond], devVoxel, *devGeom,
                                                                         cond, y, n, devDirection);
                                hipDeviceSynchronize();
                            }

                            // ratio process
                            if (method == Method::ART) {
                                projSubtract<<<gridD, blockD>>>(&devProj[lenD * cond], &devSino[lenD * cond],
                                                                devGeom, n, nullptr);
                            } else {
                                projRatio<<<gridD, blockD>>>(&devProj[lenD * cond], &devSino[lenD * cond], devGeom, n,
                                                             nullptr);
                            }
                            hipDeviceSynchronize();
                        }
                    }

                    // backwardProj process
                    for (int y = 0; y < sizeV[1]; y++) {
                        hipMemset(devVoxelFactor, 0, sizeof(float) * sizeV[0] * sizeV[1] * NUM_BASIS_VECTOR);
                        hipMemset(devVoxelTmp, 0, sizeof(float) * sizeV[0] * sizeV[1] * NUM_BASIS_VECTOR);
                        for (int cond = 0; cond < NUM_PROJ_COND; cond++) {
                            pbar.update();
                            for (int subOrder = 0; subOrder < subsetSize; subOrder++) {
                                int n = rotation * ((sub + batch * subOrder) % nProj);
                                backwardProjXTTbyFiber<<<gridV, blockV>>>(&devProj[lenD * cond], devVoxelTmp,
                                                                          devVoxelFactor,
                                                                          *devGeom, cond, y, n, devDirection);
                                hipDeviceSynchronize();
                            }
                        }
                        if (method == Method::ART) {
                            voxelPlus<<<gridV, blockV>>>(devVoxel, devVoxelTmp, lambda / (float) subsetSize, devGeom,
                                                         y);
                        } else {
                            voxelProduct<<<gridV, blockV>>>(devVoxel, devVoxelTmp, devVoxelFactor, devGeom, y);
                        }
                        hipDeviceSynchronize();
                    }
                }

                d_loss_proj /= static_cast<float>(NUM_DETECT_V * NUM_DETECT_U * NUM_PROJ);
                hipMemcpy(losses.data() + ep, &d_loss_proj, sizeof(float), hipMemcpyDeviceToHost); // loss

                // record sqrt of voxel val to host memory
                for (int y = 0; y < sizeV[1]; y++) {
                    voxelSqrtFromSrc<<<gridV, blockV>>>(hostVoxel, devVoxel, devGeom, y); // host
                    hipDeviceSynchronize();
                }

                for (int i = 0; i < NUM_BASIS_VECTOR; i++) {
                    hipMemcpy(voxel[i].get(), &hostVoxel[i * lenV], sizeof(float) * lenV, hipMemcpyDeviceToHost);
                    hipDeviceSynchronize();
                }

                // calc main direction
                Volume<float> tmp[3];
                for (auto &e: tmp) {
                    e = Volume<float>(NUM_VOXEL, NUM_VOXEL, NUM_VOXEL);
                }

                for (int z = 0; z < NUM_VOXEL; z++) {
                    for (int y = 0; y < NUM_VOXEL; y++) {
                        for (int x = 0; x < NUM_VOXEL; x++) {
                            calcEigenVector(voxel, md, tmp, y, z, x);
                        }
                    }
                }
                for (int i = 0; i < 3; i++)
                    hipMemcpy(&devDirection[i * lenV], md[i].get(), sizeof(float) * lenV, hipMemcpyHostToDevice);
            }
            // copy md to devMD
            for (int i = 0; i < 3; i++)
                hipMemcpy(md[i].get(), &devDirection[i * lenV], sizeof(float) * lenV, hipMemcpyDeviceToHost);
        }

        for (int i = 0; i < NUM_PROJ_COND; i++)
            hipMemcpy(sinogram[i].get(), &devProj[i * lenD], sizeof(float) * lenD, hipMemcpyDeviceToHost);
        /*
        for (int i = 0; i < NUM_BASIS_VECTOR; i++)
            hipMemcpy(voxel[i].get(), &hostVoxel[i * lenV], sizeof(float) * lenV, hipMemcpyDeviceToHost);
*/
        hipFree(devProj);
        hipFree(devSino);
        hipFree(devVoxel);
        hipFree(devGeom);
        hipFree(devVoxelFactor);
        hipFree(devVoxelTmp);
        hipFree(hostVoxel);
        hipFree(devDirection);

        std::ofstream ofs("../python/loss.csv");
        for (auto &e: losses)
            ofs << e << ",";
    }

    void reconstruct(Volume<float> *sinogram, Volume<float> *voxel, Volume<float> *md, const Geometry &geom,
                     int epoch, int batch, Rotate dir, Method method, float lambda) {
        std::cout << "starting reconstruct(XTT)..." << std::endl;

        // int rotation = (dir == Rotate::CW) ? -1 : 1;
        int rotation = (dir == Rotate::CW) ? 1 : -1;

        int sizeV[3] = {voxel[0].x(), voxel[0].y(), voxel[0].z()};
        int sizeD[3] = {sinogram[0].x(), sinogram[0].y(), sinogram[0].z()};
        int nProj = sizeD[2];

        // hipMalloc
        float *devSino, *devProj, *devVoxel, *devVoxelFactor, *devVoxelTmp;
        const long lenV = sizeV[0] * sizeV[1] * sizeV[2];
        const long lenD = sizeD[0] * sizeD[1] * sizeD[2];
        const long lenP = sizeV[0] * sizeV[2];

        hipMalloc(&devSino, sizeof(float) * lenD * NUM_PROJ_COND);
        hipMalloc(&devProj, sizeof(float) * lenD * NUM_PROJ_COND); // memory can be small to subsetSize
        hipMalloc(&devVoxel, sizeof(float) * lenV * NUM_BASIS_VECTOR);
        hipMalloc(&devVoxelFactor, sizeof(float) * sizeV[0] * sizeV[1] * NUM_BASIS_VECTOR);
        hipMalloc(&devVoxelTmp, sizeof(float) * sizeV[0] * sizeV[1] * NUM_BASIS_VECTOR);

        float *loss1;
        hipMalloc(&loss1, sizeof(float));

        for (int i = 0; i < NUM_PROJ_COND; i++)
            hipMemcpy(&devSino[i * lenD], sinogram[i].get(), sizeof(float) * lenD, hipMemcpyHostToDevice);
        for (int i = 0; i < NUM_BASIS_VECTOR; i++)
            hipMemcpy(&devVoxel[i * lenV], voxel[i].get(), sizeof(float) * lenV, hipMemcpyHostToDevice);

        Geometry *devGeom;
        hipMalloc(&devGeom, sizeof(Geometry));
        hipMemcpy(devGeom, &geom, sizeof(Geometry), hipMemcpyHostToDevice);

        // define blocksize
        dim3 blockV(BLOCK_SIZE, BLOCK_SIZE, 1);
        dim3 gridV((sizeV[0] + BLOCK_SIZE - 1) / BLOCK_SIZE, (sizeV[2] + BLOCK_SIZE - 1) / BLOCK_SIZE, 1);
        dim3 blockD(BLOCK_SIZE, BLOCK_SIZE, 1);
        dim3 gridD((sizeD[0] + BLOCK_SIZE - 1) / BLOCK_SIZE, (sizeD[1] + BLOCK_SIZE - 1) / BLOCK_SIZE, 1);

        // forwardProj, divide, backwardProj proj
        int subsetSize = (nProj + batch - 1) / batch;
        std::vector<int> subsetOrder(batch);
        for (int i = 0; i < batch; i++) {
            subsetOrder[i] = i;
        }

        std::vector<float> losses(epoch);

        // progress bar
        progressbar pbar(epoch * batch * NUM_PROJ_COND * (subsetSize + sizeV[1]));

        // set scattering vector direction
        // setScatterDirecOn4D(2.0f * (float) M_PI * scatter_angle_xy / 360.0f, basisVector);

        // main routine
        for (int ep = 0; ep < epoch; ep++) {
            std::mt19937_64 get_rand_mt; // fixed seed
            std::shuffle(subsetOrder.begin(), subsetOrder.end(), get_rand_mt);
            hipMemset(loss1, 0.0f, sizeof(float));
            hipMemset(devProj, 0.0f, sizeof(float) * lenD * NUM_PROJ_COND);
            for (int &sub: subsetOrder) {
                // forwardProj and ratio
                for (int cond = 0; cond < NUM_PROJ_COND; cond++) {
                    for (int subOrder = 0; subOrder < subsetSize; subOrder++) {
                        int n = rotation * ((sub + batch * subOrder) % nProj);
                        // !!care!! judge from vecSod which plane we chose
                        pbar.update();

                        // forwardProj process
                        for (int y = 0; y < sizeV[1]; y++) {
                            // iterate basis vector in forwardProjXTT
                            forwardProjXTT<<<gridV, blockV>>>(&devProj[lenD * cond], devVoxel, devGeom, cond, y, n);
                            hipDeviceSynchronize();
                        }

                        // ratio process
                        if (method == Method::ART) {
                            projSubtract<<<gridD, blockD>>>(&devProj[lenD * cond], &devSino[lenD * cond], devGeom,
                                                            n, loss1);
                        } else {
                            projRatio<<<gridD, blockD>>>(&devProj[lenD * cond], &devSino[lenD * cond],
                                                         devGeom, n, loss1);
                        }
                        hipDeviceSynchronize();
                    }
                }

                // backwardProj process
                for (int y = 0; y < sizeV[1]; y++) {
                    hipMemset(devVoxelFactor, 0, sizeof(float) * sizeV[0] * sizeV[1] * NUM_BASIS_VECTOR);
                    hipMemset(devVoxelTmp, 0, sizeof(float) * sizeV[0] * sizeV[1] * NUM_BASIS_VECTOR);
                    for (int cond = 0; cond < NUM_PROJ_COND; cond++) {
                        pbar.update();
                        for (int subOrder = 0; subOrder < subsetSize; subOrder++) {
                            int n = rotation * ((sub + batch * subOrder) % nProj);
                            backwardProjXTT<<<gridV, blockV>>>(&devProj[lenD * cond], devVoxelTmp, devVoxelFactor,
                                                               devGeom, cond, y, n);
                            hipDeviceSynchronize();
                        }
                    }
                    for (int vec = 0; vec < NUM_BASIS_VECTOR; vec++) {
                        if (method == Method::ART) {
                            voxelPlus<<<gridV, blockV>>>(&devVoxel[lenV * vec], &devVoxelTmp[lenP * vec], lambda / (float) subsetSize, devGeom, y);
                        } else {
                            voxelProduct<<<gridV, blockV>>>(&devVoxel[lenV * vec], &devVoxelTmp[lenP * vec],
                                                            &devVoxelFactor[lenP * vec], devGeom, y);
                        }
                    }
                    hipDeviceSynchronize();
                }
            }

            hipMemcpy(losses.data() + ep, loss1, sizeof(float), hipMemcpyDeviceToHost); // loss
        }

        for (int y = 0; y < sizeV[1]; y++) {
            voxelSqrt<<<gridV, blockV>>>(devVoxel, devGeom, y);
            hipDeviceSynchronize();
        }

        for (int i = 0; i < NUM_PROJ_COND; i++)
            hipMemcpy(sinogram[i].get(), &devProj[i * lenD], sizeof(float) * lenD, hipMemcpyDeviceToHost);
        for (int i = 0; i < NUM_BASIS_VECTOR; i++)
            hipMemcpy(voxel[i].get(), &devVoxel[i * lenV], sizeof(float) * lenV, hipMemcpyDeviceToHost);

        std::cout << "\ncalculate main direction\n";
        Volume<float> tmp[3];
        for (auto &e: tmp) {
            e = Volume<float>(NUM_VOXEL, NUM_VOXEL, NUM_VOXEL);
        }
        // calc main direction
        for (int z = 0; z < NUM_VOXEL; z++) {
#pragma omp parallel for
            for (int y = 0; y < NUM_VOXEL; y++) {
                for (int x = 0; x < NUM_VOXEL; x++) {
                    calcEigenVector(voxel, md, tmp, x, y, z);
                }
            }
        }

        hipFree(devProj);
        hipFree(devSino);
        hipFree(devVoxel);
        hipFree(devGeom);
        hipFree(devVoxelFactor);
        hipFree(devVoxelTmp);

        std::ofstream ofs("../python/loss.csv");
        for (auto &e: losses)
            ofs << e / static_cast<float>(NUM_DETECT_V * NUM_DETECT_U * NUM_PROJ) << ",";
    }

    void
    fiberModelReconstruct(Volume<float> *sinogram, Volume<float> *voxel, const Geometry &geom, int epoch, int batch,
                          Rotate dir, Method method, float lambda) {
        std::cout << "starting reconstruct(XTT), use fiber model..." << std::endl;
        for (int i = 0; i < NUM_BASIS_VECTOR; i++) {
            voxel[i].forEach([](float value) -> float { return 0.01; });
        }

        int rotation = (dir == Rotate::CW) ? 1 : -1;

        int sizeV[3] = {voxel[0].x(), voxel[0].y(), voxel[0].z()};
        int sizeD[3] = {sinogram[0].x(), sinogram[0].y(), sinogram[0].z()};
        int nProj = sizeD[2];

        // hipMalloc
        float *devSino, *devProj, *devVoxel, *devVoxelFactor, *devVoxelTmp;
        const long lenV = sizeV[0] * sizeV[1] * sizeV[2];
        const long lenD = sizeD[0] * sizeD[1] * sizeD[2];

        hipMalloc(&devSino, sizeof(float) * lenD * NUM_PROJ_COND);
        hipMalloc(&devProj, sizeof(float) * lenD * NUM_PROJ_COND); // memory can be small to subsetSize
        hipMalloc(&devVoxel, sizeof(float) * lenV * NUM_BASIS_VECTOR);
        hipMalloc(&devVoxelFactor, sizeof(float) * sizeV[0] * sizeV[1] * NUM_BASIS_VECTOR);
        hipMalloc(&devVoxelTmp, sizeof(float) * sizeV[0] * sizeV[1] * NUM_BASIS_VECTOR);

        for (int i = 0; i < NUM_PROJ_COND; i++)
            hipMemcpy(&devSino[i * lenD], sinogram[i].get(), sizeof(float) * lenD, hipMemcpyHostToDevice);
        for (int i = 0; i < NUM_BASIS_VECTOR; i++)
            hipMemcpy(&devVoxel[i * lenV], voxel[i].get(), sizeof(float) * lenV, hipMemcpyHostToDevice);

        Geometry *devGeom;
        hipMalloc(&devGeom, sizeof(Geometry));
        hipMemcpy(devGeom, &geom, sizeof(Geometry), hipMemcpyHostToDevice);

        dim3 blockV(BLOCK_SIZE, BLOCK_SIZE, 1);
        dim3 gridV((sizeV[0] + BLOCK_SIZE - 1) / BLOCK_SIZE, (sizeV[2] + BLOCK_SIZE - 1) / BLOCK_SIZE, 1);
        dim3 blockD(BLOCK_SIZE, BLOCK_SIZE, 1);
        dim3 gridD((sizeD[0] + BLOCK_SIZE - 1) / BLOCK_SIZE, (sizeD[1] + BLOCK_SIZE - 1) / BLOCK_SIZE, 1);

        // forwardProj, divide, backwardProj proj
        int subsetSize = (nProj + batch - 1) / batch;
        std::vector<int> subsetOrder(batch);
        for (int i = 0; i < batch; i++) {
            subsetOrder[i] = i;
        }

        std::vector<float> losses(epoch);

        // progress bar
        progressbar pbar(epoch * batch * NUM_PROJ_COND * (subsetSize + sizeV[1]));

        // set scattering vector direction
        // setScatterDirecOn4D(2.0f * (float) M_PI * scatter_angle_xy / 360.0f, basisVector);

        // main routine
        for (int ep = 0; ep < epoch; ep++) {
            std::mt19937_64 get_rand_mt; // fixed seed
            std::shuffle(subsetOrder.begin(), subsetOrder.end(), get_rand_mt);
            hipMemset(&d_loss_proj, 0.0f, sizeof(float));
            hipMemset(devProj, 0.0f, sizeof(float) * lenD * NUM_PROJ_COND);
            for (int &sub: subsetOrder) {
                // forwardProj and ratio
                for (int cond = 0; cond < NUM_PROJ_COND; cond++) {
                    for (int subOrder = 0; subOrder < subsetSize; subOrder++) {
                        int n = rotation * ((sub + batch * subOrder) % nProj);
                        // !!care!! judge from vecSod which plane we chose
                        pbar.update();

                        // forwardProj process
                        for (int y = 0; y < sizeV[1]; y++) {
                            // iterate basis vector in forwardProjXTT
                            forwardProjFiber<<<gridV, blockV>>>(&devProj[lenD * cond], devVoxel, devGeom, cond, y, n);
                            hipDeviceSynchronize();
                        }

                        // ratio process
                        if (method == Method::ART) {
                            projSubtract<<<gridD, blockD>>>(&devProj[lenD * cond], &devSino[lenD * cond], devGeom,
                                                            n, nullptr);
                        } else {
                            projRatio<<<gridD, blockD>>>(&devProj[lenD * cond], &devSino[lenD * cond], devGeom, n,
                                                         nullptr);
                        }
                        hipDeviceSynchronize();
                    }
                }

                // backwardProj process
                for (int y = 0; y < sizeV[1]; y++) {
                    hipMemset(devVoxelFactor, 0, sizeof(float) * sizeV[0] * sizeV[1] * NUM_BASIS_VECTOR);
                    hipMemset(devVoxelTmp, 0, sizeof(float) * sizeV[0] * sizeV[1] * NUM_BASIS_VECTOR);
                    for (int cond = 0; cond < NUM_PROJ_COND; cond++) {
                        pbar.update();
                        for (int subOrder = 0; subOrder < subsetSize; subOrder++) {
                            int n = rotation * ((sub + batch * subOrder) % nProj);
                            backwardProjFiber<<<gridV, blockV>>>(&devProj[lenD * cond], devVoxel, devVoxelTmp,
                                                                 devVoxelFactor, devGeom, cond, y, n);
                            hipDeviceSynchronize();
                        }
                    }
                    if (method == Method::ART) {
                        voxelPlus<<<gridV, blockV>>>(devVoxel, devVoxelTmp, lambda / (float) subsetSize, devGeom,
                                                     y);
                    } else {
                        voxelProduct<<<gridV, blockV>>>(devVoxel, devVoxelTmp, devVoxelFactor, devGeom, y);
                    }
                    hipDeviceSynchronize();
                }
            }

            d_loss_proj /= static_cast<float>(NUM_DETECT_V * NUM_DETECT_U * NUM_PROJ);
            hipMemcpy(losses.data() + ep, &d_loss_proj, sizeof(float), hipMemcpyDeviceToHost); // loss
        }

        for (int i = 0; i < NUM_PROJ_COND; i++)
            hipMemcpy(sinogram[i].get(), &devProj[i * lenD], sizeof(float) * lenD, hipMemcpyDeviceToHost);
        for (int i = 0; i < NUM_BASIS_VECTOR; i++)
            hipMemcpy(voxel[i].get(), &devVoxel[i * lenV], sizeof(float) * lenV, hipMemcpyDeviceToHost);

        hipFree(devProj);
        hipFree(devSino);
        hipFree(devVoxel);
        hipFree(devGeom);
        hipFree(devVoxelFactor);
        hipFree(devVoxelTmp);

        std::ofstream ofs("../python/loss.csv");
        for (auto &e: losses)
            ofs << e << ",";
    }

    void
    orthTwiceReconstruct(Volume<float> *sinogram, Volume<float> voxel[3], Volume<float> md[3], const Geometry &geom,
                         int iter1, int iter2, int batch, Rotate dir, Method method, float lambda) {
        std::cout << "starting reconstruct(orth)..." << std::endl;

        // int rotation = (dir == Rotate::CW) ? -1 : 1;
        int rotation = (dir == Rotate::CW) ? 1 : -1;

        int sizeV[3] = {voxel[0].x(), voxel[0].y(), voxel[0].z()};
        int sizeD[3] = {sinogram[0].x(), sinogram[0].y(), sinogram[0].z()};
        int nProj = sizeD[2];

        // hipMalloc
        float *devSino, *devProj, *devVoxel, *devVoxelFactor, *devVoxelTmp, *devEstimate;
        const long lenV = sizeV[0] * sizeV[1] * sizeV[2];
        const long lenD = sizeD[0] * sizeD[1] * sizeD[2];
        const long lenP = sizeV[0] * sizeV[2];

        hipMalloc(&devSino, sizeof(float) * lenD * NUM_PROJ_COND);
        hipMalloc(&devProj, sizeof(float) * lenD * NUM_PROJ_COND); // memory can be small to subsetSize
        hipMalloc(&devVoxel, sizeof(float) * lenV * NUM_BASIS_VECTOR);
        hipMalloc(&devVoxelFactor, sizeof(float) * sizeV[0] * sizeV[1] * NUM_BASIS_VECTOR);
        hipMalloc(&devVoxelTmp, sizeof(float) * sizeV[0] * sizeV[1] * NUM_BASIS_VECTOR);
        hipMalloc(&devEstimate, sizeof(float) * lenV * 2);

        for (int i = 0; i < NUM_PROJ_COND; i++)
            hipMemcpy(&devSino[i * lenD], sinogram[i].get(), sizeof(float) * lenD, hipMemcpyHostToDevice);
        for (int i = 0; i < NUM_BASIS_VECTOR; i++)
            hipMemcpy(&devVoxel[i * lenV], voxel[i].get(), sizeof(float) * lenV, hipMemcpyHostToDevice);
        // store theta, phi on polar coordination to devDirection
        float *devCoef, *devCoefTmp;
        hipMalloc(&devCoef, sizeof(float) * lenV * 2);
        hipMalloc(&devCoefTmp, sizeof(float) * lenV * 2);
        Volume<float> coef[2];
        for (auto &co: coef)
            co = Volume<float>(NUM_VOXEL, NUM_VOXEL, NUM_VOXEL);

        // !!!!!!!!!!!!!!!!!!!!!!!!!!
        coef[0].forEach([](float value) -> float { return 0.0f; });
        coef[1].forEach([](float value) -> float { return 1.0f; });
        // coef[0].forEach([](float value) -> float { return 3.0 * M_PI / 4.0f; });
        // coef[1].forEach([](float value) -> float { return std::cos(M_PI / 4.0f); });
        hipMemcpy(&devCoef[0], coef[0].get(), sizeof(float) * lenV, hipMemcpyHostToDevice);
        hipMemcpy(&devCoef[lenV], coef[1].get(), sizeof(float) * lenV, hipMemcpyHostToDevice);

        Geometry *devGeom;
        hipMalloc(&devGeom, sizeof(Geometry));
        hipMemcpy(devGeom, &geom, sizeof(Geometry), hipMemcpyHostToDevice);

        // define blocksize
        dim3 blockV(BLOCK_SIZE, BLOCK_SIZE, 1);
        dim3 gridV((sizeV[0] + BLOCK_SIZE - 1) / BLOCK_SIZE, (sizeV[2] + BLOCK_SIZE - 1) / BLOCK_SIZE, 1);
        dim3 blockD(BLOCK_SIZE, BLOCK_SIZE, 1);
        dim3 gridD((sizeD[0] + BLOCK_SIZE - 1) / BLOCK_SIZE, (sizeD[1] + BLOCK_SIZE - 1) / BLOCK_SIZE, 1);

        hiprandState *devStates;
        int threadNum = BLOCK_SIZE * (int) ((sizeV[0] + BLOCK_SIZE - 1) / BLOCK_SIZE);
        hipMalloc(&devStates, threadNum * threadNum * threadNum * sizeof(hiprandState));
        setup_rand<<<gridV, blockV>>>(devStates, threadNum, 0);
        // forwardProj, divide, backwardProj proj
        int subsetSize = (nProj + batch - 1) / batch;
        std::vector<int> subsetOrder(batch);
        for (int i = 0; i < batch; i++) {
            subsetOrder[i] = i;
        }

        // progress bar
        progressbar pbar((iter1 + 5) * iter2 * batch * NUM_PROJ_COND * (subsetSize + sizeV[1]));

        std::random_device seed_gen;
        std::mt19937 engine(seed_gen());
        std::uniform_real_distribution<float> dist(0.f, 1.f);

        Volume<float> loss_map1 = Volume<float>(NUM_VOXEL, NUM_VOXEL, NUM_VOXEL);
        Volume<float> loss_map2 = Volume<float>(NUM_VOXEL, NUM_VOXEL, NUM_VOXEL);
        float *devLoss1;
        float *devLoss2;

        hipMalloc(&devLoss1, sizeof(float));
        hipMalloc(&devLoss2, sizeof(float) * lenV);

        std::vector<float> proj_loss(iter1 * iter2);
        std::vector<float> norm_loss(iter1);
        Volume<float> tmp[3];
        for (auto &e: tmp) {
            e = Volume<float>(NUM_VOXEL, NUM_VOXEL, NUM_VOXEL);
        }

        // main routine
        // 5 kai de zyubun
        for (int ep1 = 0; ep1 < 5; ep1++) {
            for (int i = 0; i < 3; i++) {
                hipMemcpy(&devVoxel[i * lenV], voxel[i].get(), sizeof(float) * lenV, hipMemcpyHostToDevice);
            }
            coef[0].forEach([](float value) -> float { return 0.0f; });
            coef[1].forEach([](float value) -> float { return 1.0f; });
            hipMemcpy(&devCoef[0], coef[0].get(), sizeof(float) * lenV, hipMemcpyHostToDevice);
            hipMemcpy(&devCoef[lenV], coef[1].get(), sizeof(float) * lenV, hipMemcpyHostToDevice);
            float judge = 0.0f;
            if (ep1 != 0) {
                judge = 1.0f - (float) (ep1 - 1) * 0.3f;
                for (int y = 0; y < sizeV[1]; y++) {
                    calcNormalVectorThreeDirec<<<gridV, blockV>>>(devVoxel, devCoef, y, ep1, devGeom, devLoss2,
                                                                  devStates, judge);
                    hipDeviceSynchronize();
                }
            }

            for (int ep2 = 0; ep2 < iter2; ep2++) {
                std::shuffle(subsetOrder.begin(), subsetOrder.end(), engine);
                hipMemset(devProj, 0.0f, sizeof(float) * lenD * NUM_PROJ_COND);
                for (int &sub: subsetOrder) {
                    // forwardProj and ratio
                    for (int cond = 0; cond < NUM_PROJ_COND; cond++) {
                        for (int subOrder = 0; subOrder < subsetSize; subOrder++) {
                            int n = rotation * ((sub + batch * subOrder) % nProj);
                            // !!care!! judge from vecSod which plane we chose
                            pbar.update();

                            // forwardProj process
                            for (int y = 0; y < sizeV[1]; y++) {
                                // 回転行列に従って3方向散乱係数の順投影
                                forwardOrth<<<gridV, blockV>>>(&devProj[lenD * cond], devVoxel, devCoef,
                                                               cond, y, n, ep1, devGeom);
                                hipDeviceSynchronize();
                            }
                            // ratio process
                            if (method == Method::ART)
                                projSubtract<<<gridD, blockD>>>(&devProj[lenD * cond],
                                                                &devSino[lenD * cond], devGeom, n, devLoss1);
                            else
                                projRatio<<<gridD, blockD>>>(&devProj[lenD * cond], &devSino[lenD * cond],
                                                             devGeom, n, devLoss1);
                            hipDeviceSynchronize();
                        }
                    }

                    // backwardProj process
                    for (int y = 0; y < sizeV[1]; y++) {
                        hipMemset(devVoxelFactor, 0, sizeof(float) * sizeV[0] * sizeV[1] * NUM_BASIS_VECTOR);
                        hipMemset(devVoxelTmp, 0, sizeof(float) * sizeV[0] * sizeV[1] * NUM_BASIS_VECTOR);
                        for (int cond = 0; cond < NUM_PROJ_COND; cond++) {
                            pbar.update();
                            for (int subOrder = 0; subOrder < subsetSize; subOrder++) {
                                int n = rotation * ((sub + batch * subOrder) % nProj);

                                backwardOrth<<<gridV, blockV>>>(&devProj[lenD * cond], devCoef, devVoxelTmp,
                                                                devVoxelFactor, devGeom, cond, y, n, ep1);
                                hipDeviceSynchronize();
                            }
                        }
                        for (int vec = 0; vec < NUM_BASIS_VECTOR; vec++) {
                            if (method == Method::ART) {
                                voxelPlus<<<gridV, blockV>>>(&devVoxel[lenV * vec], &devVoxelTmp[lenP * vec],
                                                             lambda / (float) subsetSize, devGeom, y);
                            } else {
                                voxelProduct<<<gridV, blockV>>>(&devVoxel[lenV * vec], &devVoxelTmp[lenP * vec],
                                                                &devVoxelFactor[lenP * vec], devGeom, y);
                            }
                        }
                        hipDeviceSynchronize();
                    }
                }
                // ----- end iter1 ----- //
            }

            // swap later
            for (int y = 0; y < sizeV[1]; y++) {
                voxelSqrt<<<gridV, blockV>>>(devVoxel, devGeom, y);
                hipDeviceSynchronize();
            }
            if (ep1 == 0) {
                for (int i = 0; i < NUM_BASIS_VECTOR; i++)
                    hipMemcpy(voxel[i].get(), &devVoxel[i * lenV], sizeof(float) * lenV, hipMemcpyDeviceToHost);
            }
            if (ep1 != 0) {
                for (int y = 0; y < sizeV[1]; y++) {
                    calcNormalVectorThreeDirecSaveEst<<<gridV, blockV>>>(devVoxel, devCoef, y, devGeom, devLoss2,
                                                                         devEstimate, ep1 - 1);
                    hipDeviceSynchronize();
                }
            }
            std::string xyz[] = {"x", "y", "z"};

            for (int i = 0; i < NUM_PROJ_COND; i++)
                hipMemcpy(sinogram[i].get(), &devProj[i * lenD], sizeof(float) * lenD, hipMemcpyDeviceToHost);
            for (int i = 0; i < 2; i++) {
                hipMemcpy(coef[i].get(), &devCoef[i * lenV], sizeof(float) * lenV, hipMemcpyDeviceToHost);
            }
            convertNormVector(voxel, md, coef);
            /*
            Volume<float> est = Volume<float>(NUM_VOXEL, NUM_VOXEL, NUM_VOXEL);
            hipMemcpy(est.get(), &devEstimate[lenV], sizeof(float) * lenV, hipMemcpyDeviceToHost);
            std::string savefilePathCT =
                    "../volume_bin/cfrp_xyz7_13axis/sequence/est" + std::to_string(ep1) + "_" + std::to_string(NUM_VOXEL) + "x" +
                    std::to_string(NUM_VOXEL) + "x" + std::to_string(NUM_VOXEL) + ".raw";
            est.save(savefilePathCT);
             */
            // ----- end iter2 -----
        }

        if (iter1 != 0) {
            coef[0].forEach([](float value) -> float { return 0.0f; });
            coef[1].forEach([](float value) -> float { return 1.0f; });
            hipMemcpy(&devCoef[0], coef[0].get(), sizeof(float) * lenV, hipMemcpyHostToDevice);
            hipMemcpy(&devCoef[lenV], coef[1].get(), sizeof(float) * lenV, hipMemcpyHostToDevice);
        }

        for (int ep1 = 0; ep1 < iter1; ep1++) {
            for (int i = 0; i < 3; i++) {
                voxel[i].forEach([](float value) -> float { return 0.1f; });
                hipMemcpy(&devVoxel[i * lenV], voxel[i].get(), sizeof(float) * lenV, hipMemcpyHostToDevice);
            }
            hipMemset(devLoss2, 0.0f, sizeof(float) * lenV);
            float judge = dist(engine);

            for (int ep2 = 0; ep2 < iter2; ep2++) {
                std::shuffle(subsetOrder.begin(), subsetOrder.end(), engine);
                hipMemset(devLoss1, 0.0f, sizeof(float));
                hipMemset(devProj, 0.0f, sizeof(float) * lenD * NUM_PROJ_COND);
                for (int &sub: subsetOrder) {
                    // forwardProj and ratio
                    for (int cond = 0; cond < NUM_PROJ_COND; cond++) {
                        for (int subOrder = 0; subOrder < subsetSize; subOrder++) {
                            int n = rotation * ((sub + batch * subOrder) % nProj);
                            // !!care!! judge from vecSod which plane we chose
                            pbar.update();

                            // forwardProj process
                            for (int y = 0; y < sizeV[1]; y++) {
                                // 回転行列に従って3方向散乱係数の順投影
                                forwardOrth<<<gridV, blockV>>>(&devProj[lenD * cond], devVoxel, devCoef,
                                                               cond, y, n, ep1, devGeom);
                                hipDeviceSynchronize();
                            }

                            // ratio process
                            if (method == Method::ART)
                                projSubtract<<<gridD, blockD>>>(&devProj[lenD * cond],
                                                                &devSino[lenD * cond], devGeom, n, devLoss1);
                            else
                                projRatio<<<gridD, blockD>>>(&devProj[lenD * cond], &devSino[lenD * cond],
                                                             devGeom, n, devLoss1);
                            hipDeviceSynchronize();

                        }
                    }

                    // backwardProj process
                    for (int y = 0; y < sizeV[1]; y++) {
                        hipMemset(devVoxelFactor, 0, sizeof(float) * sizeV[0] * sizeV[1] * NUM_BASIS_VECTOR);
                        hipMemset(devVoxelTmp, 0, sizeof(float) * sizeV[0] * sizeV[1] * NUM_BASIS_VECTOR);
                        for (int cond = 0; cond < NUM_PROJ_COND; cond++) {
                            pbar.update();
                            for (int subOrder = 0; subOrder < subsetSize; subOrder++) {
                                int n = rotation * ((sub + batch * subOrder) % nProj);

                                backwardOrth<<<gridV, blockV>>>(&devProj[lenD * cond], devCoef, devVoxelTmp,
                                                                devVoxelFactor, devGeom, cond, y, n, ep1);
                                hipDeviceSynchronize();
                            }
                        }
                        for (int vec = 0; vec < NUM_BASIS_VECTOR; vec++) {
                            if (method == Method::ART) {
                                voxelPlus<<<gridV, blockV>>>(&devVoxel[lenV * vec], &devVoxelTmp[lenP * vec],
                                                             lambda / (float) subsetSize, devGeom, y);
                            } else {
                                voxelProduct<<<gridV, blockV>>>(&devVoxel[lenV * vec], &devVoxelTmp[lenP * vec],
                                                                &devVoxelFactor[lenP * vec], devGeom, y);
                            }
                        }
                        hipDeviceSynchronize();
                        hipDeviceSynchronize();
                    }
                }
                hipMemcpy(proj_loss.data() + ep1 * iter2 + ep2, devLoss1, sizeof(float),
                           hipMemcpyDeviceToHost); // loss
                // std::cout << proj_loss[ep2 * (ep1 + 1)] << std::endl;
                // ----- end iter1 ----- //
            }
            // swap later
            for (int y = 0; y < sizeV[1]; y++) {
                voxelSqrt<<<gridV, blockV>>>(devVoxel, devGeom, y);
                hipDeviceSynchronize();
            }
            for (int i = 0; i < NUM_BASIS_VECTOR; i++)
                hipMemcpy(voxel[i].get(), &devVoxel[i * lenV], sizeof(float) * lenV, hipMemcpyDeviceToHost);
            for (int y = 0; y < sizeV[1]; y++) {
                calcNormalVectorThreeDirecWithEst<<<gridV, blockV>>>(devVoxel, devCoef, y, devGeom,
                                                                     devLoss2, devEstimate);
                hipDeviceSynchronize();
            }
            for (int i = 0; i < 2; i++)
                hipMemcpy(coef[i].get(), &devCoef[i * lenV], sizeof(float) * lenV, hipMemcpyDeviceToHost);

            std::string xyz[] = {"x", "y", "z"};
            // filtering
            // quadlicFormFilterCPU(voxel, coef, 0.01);

            for (int i = 0; i < 2; i++)
                hipMemcpy(&devCoef[i * lenV], coef[i].get(), sizeof(float) * lenV, hipMemcpyHostToDevice);

            /*
            for (int filt = 0; filt < 2; filt++) {
                for (int y = 1; y < sizeV[1] - 1; y++) {
                    meanFiltFiber<<<gridV, blockV>>>(devCoef, devCoefTmp, devVoxel, devGeom, y, 1.0f);
                    hipDeviceSynchronize();
                }
                hipMemcpy(devCoef, devCoefTmp, sizeof(float) * lenV * 2, hipMemcpyDeviceToDevice);
            }*/

            hipMemcpy(loss_map2.get(), devLoss2, sizeof(float) * lenV, hipMemcpyDeviceToHost);
            norm_loss[ep1] = loss_map2.mean();
            // ----- end iter2 -----

            for (int i = 0; i < NUM_PROJ_COND; i++)
                hipMemcpy(sinogram[i].get(), &devProj[i * lenD], sizeof(float) * lenD, hipMemcpyDeviceToHost);
            for (int i = 0; i < 2; i++) {
                hipMemcpy(coef[i].get(), &devCoef[i * lenV], sizeof(float) * lenV, hipMemcpyDeviceToHost);
            }

            convertNormVector(voxel, md, coef);
            // save direction volume
            for (int i = 0; i < 2; i++) {
                std::string savefilePathCT =
                        "../volume_bin/cfrp_xyz7_13axis/sequence/coef_tvmin0.01" +
                        std::to_string(ep1 + 1) + "_" + xyz[i] + "_" + std::to_string(NUM_VOXEL) + "x" +
                        std::to_string(NUM_VOXEL) + "x" + std::to_string(NUM_VOXEL) + ".raw";
                coef[i].save(savefilePathCT);
            }
            for (int i = 0; i < 3; i++) {
                std::string savefilePathCT =
                        "../volume_bin/cfrp_xyz7_13axis/sequence/pca/md_tvmin0.01" +
                        std::to_string(ep1 + 1) + "_" + xyz[i] + "_" + std::to_string(NUM_VOXEL) + "x" +
                        std::to_string(NUM_VOXEL) + "x" + std::to_string(NUM_VOXEL) + ".raw";
                md[i].save(savefilePathCT);
            }
            // save ct volume
            for (int i = 0; i < 3; i++) {
                hipMemcpy(voxel[i].get(), &devVoxel[i * lenV], sizeof(float) * lenV, hipMemcpyDeviceToHost);
                std::string savefilePathCT =
                        "../volume_bin/cfrp_xyz7_13axis/sequence/volume_tvmin0.01" + std::to_string(ep1 + 1) +
                        "_orth" + std::to_string(i + 1) + "_" + std::to_string(NUM_VOXEL) + "x" +
                        std::to_string(NUM_VOXEL) + "x" + std::to_string(NUM_VOXEL) + ".raw";
                voxel[i].save(savefilePathCT);
            }
        }

        hipFree(devProj);
        hipFree(devSino);
        hipFree(devVoxel);
        hipFree(devGeom);
        hipFree(devVoxelFactor);
        hipFree(devVoxelTmp);
        hipFree(devCoef);
        hipFree(devLoss1);
        hipFree(devLoss2);
        hipFree(devCoefTmp);
        hipFree(devStates);
        hipFree(devEstimate);

        std::ofstream ofs1("../python/loss1.csv");
        std::ofstream ofs2("../python/loss2.csv");
        for (auto &e: proj_loss)
            ofs1 << e / static_cast<float>(NUM_DETECT_V * NUM_DETECT_U * NUM_PROJ * NUM_PROJ_COND) << ",";
        for (auto &e: norm_loss)
            ofs2 << e << ",";
    }
}

namespace FDK {
    void reconstruct(Volume<float> *sinogram, Volume<float> *voxel, const Geometry &geom, Rotate dir) {
        std::cout << "starting reconstruct(FDK)..." << std::endl;
        for (int i = 0; i < NUM_BASIS_VECTOR; i++) {
            voxel[i].forEach([](float value) -> float { return 0.0; });
        }

        int rotation = (dir == Rotate::CW) ? 1 : -1;

        int sizeV[3] = {voxel[0].x(), voxel[0].y(), voxel[0].z()};
        int sizeD[3] = {sinogram[0].x(), sinogram[0].y(), sinogram[0].z()};
        int nProj = sizeD[2];

        // hipMalloc
        float *devSino, *devSinoFilt, *devVoxel, *weight, *filt;
        const long lenV = sizeV[0] * sizeV[1] * sizeV[2];
        const long lenD = sizeD[0] * sizeD[1] * sizeD[2];

        hipMalloc(&devSino, sizeof(float) * lenD * NUM_PROJ_COND);
        hipMalloc(&devSinoFilt, sizeof(float) * lenD * NUM_PROJ_COND); // memory can be small to subsetSize
        hipMalloc(&devVoxel, sizeof(float) * lenV * NUM_BASIS_VECTOR);
        hipMalloc(&weight, sizeof(float) * sizeD[0] * sizeD[1]);
        hipMallocManaged(&filt, sizeof(float) * geom.detect);

        for (int i = 0; i < NUM_PROJ_COND; i++)
            hipMemcpy(&devSino[i * lenD], sinogram[i].get(), sizeof(float) * lenD, hipMemcpyHostToDevice);

        Geometry *devGeom;
        hipMalloc(&devGeom, sizeof(Geometry));
        hipMemcpy(devGeom, &geom, sizeof(Geometry), hipMemcpyHostToDevice);

        // define blocksize
        dim3 blockV(BLOCK_SIZE, BLOCK_SIZE, 1);
        dim3 gridV((sizeV[0] + BLOCK_SIZE - 1) / BLOCK_SIZE, (sizeV[2] + BLOCK_SIZE - 1) / BLOCK_SIZE, 1);
        dim3 blockD(BLOCK_SIZE, BLOCK_SIZE, 1);
        dim3 gridD((sizeD[0] + BLOCK_SIZE - 1) / BLOCK_SIZE, (sizeD[1] + BLOCK_SIZE - 1) / BLOCK_SIZE, 1);

        // progress bar

        progressbar pbar(nProj);
        calcWeight<<<gridD, blockD>>>(weight, devGeom);
        hipDeviceSynchronize();
        // make Shepp-Logan fliter

        float d = geom.detSize * (geom.sod / geom.sdd);
        // float d = geom.detSize * (geom.sod / geom.sdd);
        for (int v = 0; v < geom.detect; v++) {
            filt[v] = 1.0f / (float) (M_PI * M_PI * d * (1.0f - 4.0f * (float) (v * v)));
        }

        for (int cond = 0; cond < NUM_PROJ_COND; cond++) {
            for (int n = 0; n < nProj; n++) {
                // convolution
                // hogeTmpWakaran<<<gridD, blockD>>>();
                projConv<<<gridD, blockD>>>(&devSinoFilt[lenD * cond], &devSino[lenD * cond], devGeom, n, filt,
                                            weight);
                hipDeviceSynchronize();
                for (int y = 0; y < geom.voxel; y++) {
                    filteredBackProj<<<gridV, blockV>>>(devSinoFilt, devVoxel, devGeom, cond, y, rotation * n);
                }
            }
        }

        for (int i = 0; i < NUM_PROJ_COND; i++)
            hipMemcpy(sinogram[i].get(), &devSinoFilt[i * lenD], sizeof(float) * lenD, hipMemcpyDeviceToHost);
        for (int i = 0; i < NUM_BASIS_VECTOR; i++)
            hipMemcpy(voxel[i].get(), &devVoxel[i * lenV], sizeof(float) * lenV, hipMemcpyDeviceToHost);

        hipFree(devSinoFilt);
        hipFree(devSino);
        hipFree(devVoxel);
        hipFree(devGeom);
        hipFree(filt);
        hipFree(weight);
    }
}

void forwardProjOnly(Volume<float> *sinogram, Volume<float> *voxel, const Geometry &geom, Rotate dir) {
    std::cout << "starting forward projection..." << std::endl;

    int rotation = (dir == Rotate::CW) ? 1 : -1;

    int sizeV[3] = {voxel[0].x(), voxel[0].y(), voxel[0].z()};
    int sizeD[3] = {sinogram[0].x(), sinogram[0].y(), sinogram[0].z()};
    int nProj = sizeD[2];

    // hipMalloc
    float *devProj, *devVoxel;
    const long lenV = sizeV[0] * sizeV[1] * sizeV[2];
    const long lenD = sizeD[0] * sizeD[1] * sizeD[2];

    hipMalloc(&devProj, sizeof(float) * lenD * NUM_PROJ_COND); // memory can be small to subsetSize
    hipMalloc(&devVoxel, sizeof(float) * lenV * NUM_BASIS_VECTOR);

    for (int i = 0; i < NUM_BASIS_VECTOR; i++)
        hipMemcpy(&devVoxel[i * lenV], voxel[i].get(), sizeof(float) * lenV, hipMemcpyHostToDevice);

    Geometry *devGeom;
    hipMalloc(&devGeom, sizeof(Geometry));
    hipMemcpy(devGeom, &geom, sizeof(Geometry), hipMemcpyHostToDevice);

    // define blocksize
    dim3 blockV(BLOCK_SIZE, BLOCK_SIZE, 1);
    dim3 gridV((sizeV[0] + BLOCK_SIZE - 1) / BLOCK_SIZE, (sizeV[2] + BLOCK_SIZE - 1) / BLOCK_SIZE, 1);

    // forwardProj, divide, backwardProj proj
    // progress bar
    progressbar pbar(NUM_PROJ * NUM_PROJ_COND);

    // set scattering vector direction
    // setScatterDirecOn4D(2.0f * (float) M_PI * scatter_angle_xy / 360.0f, basisVector);

    // main routine
    hipMemset(devProj, 0.0f, sizeof(float) * lenD * NUM_PROJ_COND);
    // forwardProj and ratio
    for (int cond = 0; cond < NUM_PROJ_COND; cond++) {
        for (int n = 0; n < NUM_PROJ; n++) {
            // !!care!! judge from vecSod which plane we chose
            pbar.update();
            // forwardProj process
            for (int y = 0; y < sizeV[1]; y++) {
                forwardProj<<<gridV, blockV>>>(&devProj[lenD * cond], devVoxel, devGeom, cond, y, n * rotation);
                hipDeviceSynchronize();
            }
        }
    }

    for (int i = 0; i < NUM_PROJ_COND; i++)
        hipMemcpy(sinogram[i].get(), &devProj[i * lenD], sizeof(float) * lenD, hipMemcpyDeviceToHost);
    for (int i = 0; i < NUM_BASIS_VECTOR; i++)
        hipMemcpy(voxel[i].get(), &devVoxel[i * lenV], sizeof(float) * lenV, hipMemcpyDeviceToHost);

    hipFree(devProj);
    hipFree(devVoxel);
    hipFree(devGeom);
}

void
forwardProjFiber(Volume<float> *sinogram, Volume<float> *voxel, Volume<float> *md, Rotate dir, const Geometry &geom) {

    std::cout << "starting forward projection(orth)..." << std::endl;

    int rotation = (dir == Rotate::CW) ? 1 : -1;

    int sizeV[3] = {voxel[0].x(), voxel[0].y(), voxel[0].z()};
    int sizeD[3] = {sinogram[0].x(), sinogram[0].y(), sinogram[0].z()};
    int nProj = sizeD[2];

    float mu_strong = 1.0f;
    float mu_weak = 0.f;

    for (int i = 0; i < 3; i++) {
        voxel[i].forEach([](float value) -> float { return 0.0f; });
    }

    // hipMalloc
    float *devProj, *devVoxel, *devCoef, *devLoss;
    const long lenV = sizeV[0] * sizeV[1] * sizeV[2];
    const long lenD = sizeD[0] * sizeD[1] * sizeD[2];

    hipMalloc(&devProj, sizeof(float) * lenD * NUM_PROJ_COND); // memory can be small to subsetSize
    hipMalloc(&devVoxel, sizeof(float) * lenV * NUM_BASIS_VECTOR);
    hipMalloc(&devCoef, sizeof(float) * lenV * 2);
    hipMalloc(&devLoss, sizeof(float) * lenV);
    hipMemset(devCoef, 0.0f, sizeof(float) * lenV * 2);

    Geometry *devGeom;
    hipMalloc(&devGeom, sizeof(Geometry));
    hipMemcpy(devGeom, &geom, sizeof(Geometry), hipMemcpyHostToDevice);

    // define blocksize
    dim3 blockV(BLOCK_SIZE, BLOCK_SIZE, 1);
    dim3 gridV((sizeV[0] + BLOCK_SIZE - 1) / BLOCK_SIZE, (sizeV[2] + BLOCK_SIZE - 1) / BLOCK_SIZE, 1);
    dim3 blockD(BLOCK_SIZE, BLOCK_SIZE, 1);
    dim3 gridD((sizeD[0] + BLOCK_SIZE - 1) / BLOCK_SIZE, (sizeD[1] + BLOCK_SIZE - 1) / BLOCK_SIZE, 1);

    // set scattering vector direction
    // setScatterDirecOn4D(2.0f * (float) M_PI * scatter_angle_xy / 360.0f, basisVector);

    progressbar pbar(NUM_PROJ * NUM_PROJ_COND);

    // change devCoef if you want to rotate fiber direction
    Volume<float> coef[2];
    for (auto &e: coef)
        e = Volume<float>(NUM_VOXEL, NUM_VOXEL, NUM_VOXEL);

    /*
    for (int x = NUM_VOXEL * 1 / 5; x < NUM_VOXEL * 4 / 5; x++) {
        for (int y = NUM_VOXEL * 2 / 5; y < NUM_VOXEL * 3 / 5; y++) {
            for (int z = NUM_VOXEL * 2 / 5; z < NUM_VOXEL * 3 / 5; z++) {
                if (x < NUM_VOXEL * 2 / 5) {
                    // r->g
                    float theta = (M_PI / 2.0) * (1.0f - (x - 1 * (float) NUM_VOXEL / 5) / ((float) NUM_VOXEL / 5));
                    coef[0](x, y, z) = std::cos(theta);
                    coef[1](x, y, z) = std::sin(theta);
                    coef[2](x, y, z) = 0.0f;
                    coef[3](x, y, z) = std::cos(M_PI / 2.0f);
                    coef[4](x, y, z) = std::sin(M_PI / 2.0f);
                } else if ( x < NUM_VOXEL * 3 / 5) {
                    // g->b
                    float theta = (M_PI /2.0) * (1.0f - (x - 2 * (float) NUM_VOXEL / 5) / ((float) NUM_VOXEL / 5));
                    coef[0](x, y, z) = 1.0f;
                    coef[1](x, y, z) = 0.0f;
                    coef[2](x, y, z) = 0.0f;
                    coef[3](x, y, z) = std::cos(theta);
                    coef[4](x, y, z) = std::sin(theta);
                } else {
                    // b->r
                    float theta = (M_PI /2.0) * ((x - 3 * (float) NUM_VOXEL / 5) / ((float) NUM_VOXEL / 5));
                    coef[0](x, y, z) = 0.0f;
                    coef[1](x, y, z) = 1.0f;
                    coef[2](x, y, z) = 0.0f;
                    coef[3](x, y, z) = std::cos(theta);
                    coef[4](x, y, z) = std::sin(theta);
                }
            }
        }
    }
    */
    for (int x = NUM_VOXEL * 1 / 5; x < NUM_VOXEL * 4 / 5; x++) {
        for (int y = NUM_VOXEL * 2 / 5; y < NUM_VOXEL * 3 / 5; y++) {
            for (int z = NUM_VOXEL * 2 / 5; z < NUM_VOXEL * 3 / 5; z++) {
                if (x < NUM_VOXEL * 2 / 5) {
                    // r->g
                    /*
                    coef[0](x, y, z) = -M_PI / 4.0;
                    coef[1](x, y, z) = std::cos(M_PI / 4.0f);

                    voxel[0](x, y, z) = mu_weak;
                    voxel[1](x, y, z) = mu_strong;
                    voxel[2](x, y, z) = mu_strong;
                     */
                } else if (x < NUM_VOXEL * 3 / 5) {
                    // g->b
                    float theta = (1.0 * M_PI / 4.0);
                    coef[0](x, y, z) = theta + M_PI / 2.0f;
                    coef[1](x, y, z) = 0.57735026f;

                    voxel[0](x, y, z) = mu_weak;
                    voxel[1](x, y, z) = mu_strong;
                    voxel[2](x, y, z) = mu_strong;

                } else {
                    // b->r
                    /*
                    coef[0](x, y, z) = -3 * M_PI / 4.0;
                    coef[1](x, y, z) = std::cos(M_PI / 4.0f);

                    voxel[0](x, y, z) = mu_weak;
                    voxel[1](x, y, z) = mu_strong;
                    voxel[2](x, y, z) = mu_strong;
                     */
                }
            }
        }
    }
    // -M_PI / 4.0f
    /*
    coef[0].forEach([](float dummy) -> float {return std::cos(0.0f);});
    coef[1].forEach([](float dummy) -> float {return std::sin(0.0f);});
    coef[2].forEach([](float dummy) -> float {return 0.0f;});
    coef[3].forEach([](float dummy) -> float {return std::cos(M_PI / 2.0f);});
    coef[4].forEach([](float dummy) -> float {return std::sin(M_PI / 2.0f);});
     */

    for (int i = 0; i < NUM_BASIS_VECTOR; i++)
        hipMemcpy(&devVoxel[i * lenV], voxel[i].get(), sizeof(float) * lenV, hipMemcpyHostToDevice);
    for (int i = 0; i < 2; i++) {
        hipMemcpy(&devCoef[i * lenV], coef[i].get(), sizeof(float) * lenV, hipMemcpyHostToDevice);
    }
    for (int cond = 0; cond < NUM_PROJ_COND; cond++) {
        for (int n = 0; n < NUM_PROJ; n++) {
            // !!care!! judge from vecSod which plane we chose
            pbar.update();
            // forwardProj process
            for (int y = 0; y < sizeV[1]; y++) {
                forwardOrth<<<gridV, blockV>>>(&devProj[lenD * cond], devVoxel, devCoef,
                                               cond, y, n, 0, devGeom);
                hipDeviceSynchronize();
            }
        }
    }
    convertNormVector(voxel, md, coef);

    for (int i = 0; i < NUM_PROJ_COND; i++)
        hipMemcpy(sinogram[i].get(), &devProj[i * lenD], sizeof(float) * lenD, hipMemcpyDeviceToHost);

    hipFree(devProj);
    hipFree(devVoxel);
    hipFree(devGeom);
    hipFree(devCoef);
}

void compareXYZTensorVolume(Volume<float> *voxel, const Geometry &geom) {
    for (int i = 0; i < geom.voxel; i++) {
        for (int j = 0; j < geom.voxel; j++) {
            for (int k = 0; k < geom.voxel; k++) {
                float min = voxel[0](i, j, k);
                int idx = 0;
                for (int n = 1; n < NUM_BASIS_VECTOR; n++) {
                    if (min > voxel[n](i, j, k)) {
                        min = voxel[n](i, j, k);
                        idx = n;
                    }
                }
                for (int n = 0; n < NUM_BASIS_VECTOR; n++) {
                    if (n != idx) {
                        voxel[n](i, j, k) = 0.0f;
                    }
                }
            }
        }
    }
}

/*
__host__ void
reconstructDebugHost(Volume<float> &sinogram, Volume<float> &voxel, const Geometry &geom, const int epoch,
                     const int batch, bool dir) {

    printf("pass");
    CudaVolume<float> sino(sinogram);
    CudaVolume<float> vox(voxel);

    int sizeV[3] = {voxel.x(), voxel.y(), voxel.z()};
    int sizeD[3] = {sinogram.x(), sinogram.y(), sinogram.z()};
    int nProj = sizeD[2];


    // forward, divide, backwardProj proj
    int subsetSize = (nProj + batch - 1) / batch;
    std::vector<int> subsetOrder(batch);
    for (int i = 0; i < batch; i++) {
        subsetOrder[i] = i;
    }

    std::mt19937_64 get_rand_mt; // fixed seed
    std::shuffle(subsetOrder.begin(), subsetOrder.end(), get_rand_mt);

    // main routine
    for (int ep = 0; ep < epoch; ep++) {
        // forward
        for (int n = 15; n < nProj; n++) {

            // forwardProj
            for (int x = 0; x < sizeV[0]; x++) {
                for (int y = 0; y < sizeV[1]; y++) {
                    for (int z = 0; z < sizeV[2]; z++) {
                        int coord[4] = {x, y, z, n};
                        forwardXTTonDevice(coord, sino, &vox, geom);
                    }
                }
            }
        }
    }
}
 */
