#include "hip/hip_runtime.h"
//
// Created by tomokimori on 22/07/20.
//
#include <Geometry.h>
#include <ir.cuh>
#include <random>
#include <Params.h>
#include <cmath>

__global__ void
forwardProjXTTbyFiber(float *devProj, float *devVoxel, Geometry &geom, int cond,
                      int y, int p, float *devDirection) {
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int z = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= geom.voxel || z >= geom.voxel) return;

    const int coord[4] = {x, y, z, p};
    int sizeV[3] = {geom.voxel, geom.voxel, geom.voxel};
    int sizeD[3] = {geom.detect, geom.detect, geom.nProj};

    float u = 0.0f, v = 0.0f;
    Vector3f B(0.0f, 0.0f, 0.0f), G(0.0f, 0.0f, 0.0f);
    rayCasting(u, v, B, G, cond, coord, geom);

    Vector3f F(devDirection[coord[0] + sizeV[0] * coord[1] + sizeV[0] * sizeV[1] * coord[2] +
                            0 * (sizeV[0] * sizeV[1] * sizeV[2])],
               devDirection[coord[0] + sizeV[0] * coord[1] + sizeV[0] * sizeV[1] * coord[2] +
                            1 * (sizeV[0] * sizeV[1] * sizeV[2])],
               devDirection[coord[0] + sizeV[0] * coord[1] + sizeV[0] * sizeV[1] * coord[2] +
                            2 * (sizeV[0] * sizeV[1] * sizeV[2])]);
    if (F.norm2() < 1e-10)
        F = Vector3f(1.0f, 1.0f, 1.0f);
    F.normalize();

    if (!(0.55f < u && u < (float) sizeD[0] - 0.55f && 0.55f < v && v < (float) sizeD[1] - 0.55f) ||
        abs(F * B) > fdThresh)
        return;

    const int n = abs(coord[3]);

    float u_tmp = u - 0.5f, v_tmp = v - 0.5f;
    int intU = floor(u_tmp), intV = floor(v_tmp);
    float c1 = (1.0f - (u_tmp - (float) intU)) * (v_tmp - (float) intV),
            c2 = (u_tmp - (float) intU) * (v_tmp - (float) intV),
            c3 = (u_tmp - (float) intU) * (1.0f - (v_tmp - (float) intV)),
            c4 = (1.0f - (u_tmp - (float) intU)) * (1.0f - (v_tmp - (float) intV));

    const float ratio = (geom.voxSize * geom.voxSize) /
                        (geom.detSize * (geom.sod / geom.sdd) * geom.detSize * (geom.sod / geom.sdd));

    float proj = 0.0f;
    for (int i = 0; i < NUM_BASIS_VECTOR; i++) {
        // add scattering coefficient (read paper)
        // B->beam direction unit vector (src2voxel)
        // S->scattering base vector
        // G->grating sensivity vector

        const int idxVoxel =
                coord[0] + sizeV[0] * coord[1] + sizeV[0] * sizeV[1] * coord[2] + i * (sizeV[0] * sizeV[1] * sizeV[2]);
        Vector3f S(basisVector[3 * i + 0], basisVector[3 * i + 1], basisVector[3 * i + 2]);
        float vkm = B.cross(S).norm2() * abs(S * G);
        // float vkm = abs(S * G);

        proj += vkm * vkm * geom.voxSize * ratio * devVoxel[idxVoxel];
    }
    atomicAdd(&devProj[intU + sizeD[0] * (intV + 1) + sizeD[0] * sizeD[1] * n], c1 * proj);
    atomicAdd(&devProj[(intU + 1) + sizeD[0] * (intV + 1) + sizeD[0] * sizeD[1] * n], c2 * proj);
    atomicAdd(&devProj[(intU + 1) + sizeD[0] * intV + sizeD[0] * sizeD[1] * n], c3 * proj);
    atomicAdd(&devProj[intU + sizeD[0] * intV + sizeD[0] * sizeD[1] * n], c4 * proj);
}

__global__ void
backwardProjXTTbyFiber(float *devProj, float *devVoxelTmp, float *devVoxelFactor, Geometry &geom, int cond,
                       int y, int p, float *devDirection) {
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int z = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= geom.voxel || z >= geom.voxel) return;

    const int coord[4] = {x, y, z, p};
    int sizeV[3] = {geom.voxel, geom.voxel, geom.voxel};
    int sizeD[3] = {geom.detect, geom.detect, geom.nProj};

    float u = 0.0f, v = 0.0f;
    Vector3f B(0.0f, 0.0f, 0.0f), G(0.0f, 0.0f, 0.0f);
    rayCasting(u, v, B, G, cond, coord, geom);

    Vector3f F(devDirection[coord[0] + sizeV[0] * coord[1] + sizeV[0] * sizeV[1] * coord[2] +
                            0 * (sizeV[0] * sizeV[1] * sizeV[2])],
               devDirection[coord[0] + sizeV[0] * coord[1] + sizeV[0] * sizeV[1] * coord[2] +
                            1 * (sizeV[0] * sizeV[1] * sizeV[2])],
               devDirection[coord[0] + sizeV[0] * coord[1] + sizeV[0] * sizeV[1] * coord[2] +
                            2 * (sizeV[0] * sizeV[1] * sizeV[2])]);
    if (F.norm2() < 1e-10)
        F = Vector3f(1.0f, 1.0f, 1.0f);
    F.normalize();

    if (!(0.55f < u && u < (float) sizeD[0] - 0.55f && 0.55f < v && v < (float) sizeD[1] - 0.55f) ||
        (abs(F * B) > fdThresh))
        return;

    const int n = abs(coord[3]);
    float u_tmp = u - 0.5f, v_tmp = v - 0.5f;
    int intU = floor(u_tmp), intV = floor(v_tmp);
    float c1 = (1.0f - (u_tmp - (float) intU)) * (v_tmp - (float) intV), c2 =
            (u_tmp - (float) intU) * (v_tmp - (float) intV),
            c3 = (u_tmp - (float) intU) * (1.0f - (v_tmp - (float) intV)), c4 =
            (1.0f - (u_tmp - (float) intU)) * (1.0f - (v_tmp - (float) intV));

    for (int i = 0; i < NUM_BASIS_VECTOR; i++) {
        // calculate immutable geometry
        // add scattering coefficient (read paper)
        // B->beam direction unit vector (src2voxel)
        // S->scattering base vector
        // G->grating sensivity vector
        // v_km = (|B_m x S_k|<S_k*G>)^2
        Vector3f S(basisVector[3 * i + 0], basisVector[3 * i + 1], basisVector[3 * i + 2]);
        float vkm = B.cross(S).norm2() * abs(S * G);
        //float vkm = abs(S * G);

        const int idxVoxel = coord[0] + sizeV[0] * coord[2] + i * (sizeV[0] * sizeV[1]);
        const float backForward = vkm * vkm * c1 * devProj[intU + sizeD[0] * (intV + 1) + sizeD[0] * sizeD[1] * n] +
                                  vkm * vkm * c2 *
                                  devProj[(intU + 1) + sizeD[0] * (intV + 1) + sizeD[0] * sizeD[1] * n] +
                                  vkm * vkm * c3 * devProj[(intU + 1) + sizeD[0] * intV + sizeD[0] * sizeD[1] * n] +
                                  vkm * vkm * c4 * devProj[intU + sizeD[0] * intV + sizeD[0] * sizeD[1] * n];

        devVoxelFactor[idxVoxel] += (vkm * vkm);
        devVoxelTmp[idxVoxel] += backForward;
    }
}

__global__ void
forwardProjXTT(float *devProj, float *devVoxel, Geometry *geom, int cond,
               int y, int n) {
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int z = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= geom->voxel || z >= geom->voxel) return;

    const int coord[4] = {x, y, z, n};
    forwardXTTonDevice(coord, devProj, devVoxel, *geom, cond);
}

__global__ void
backwardProjXTT(float *devProj, float *devVoxelTmp, float *devVoxelFactor, Geometry *geom, int cond,
                int y, int n) {
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int z = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= geom->voxel || z >= geom->voxel) return;

    const int coord[4] = {x, y, z, n};
    backwardXTTonDevice(coord, devProj, devVoxelTmp, devVoxelFactor, *geom, cond);
}

__global__ void
forwardOrth(float *devProj, const float *devVoxel, const float *coefficient, int cond, int y, int n, int it,
            Geometry *geom) {
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int z = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= geom->voxel || z >= geom->voxel) return;

    const int coord[4] = {x, y, z, n};
    int sizeV[3] = {geom->voxel, geom->voxel, geom->voxel};
    int sizeD[3] = {geom->detect, geom->detect, geom->nProj};

    float u = 0.0f, v = 0.0f;
    Vector3f B(0.0f, 0.0f, 0.0f), G(0.0f, 0.0f, 0.0f);
    rayCasting(u, v, B, G, cond, coord, *geom);

    if (!(0.55f < u && u < (float) sizeD[0] - 0.55f && 0.55f < v && v < (float) sizeD[1] - 0.55f))
        return;

    float u_tmp = u - 0.5f, v_tmp = v - 0.5f;
    int intU = floor(u_tmp), intV = floor(v_tmp);
    float c1 = (1.0f - (u_tmp - (float) intU)) * (v_tmp - (float) intV),
            c2 = (u_tmp - (float) intU) * (v_tmp - (float) intV),
            c3 = (u_tmp - (float) intU) * (1.0f - (v_tmp - (float) intV)),
            c4 = (1.0f - (u_tmp - (float) intU)) * (1.0f - (v_tmp - (float) intV));

    const float ratio = (geom->voxSize * geom->voxSize) /
                        (geom->detSize * (geom->sod / geom->sdd) * geom->detSize * (geom->sod / geom->sdd));

    const float coef[5] = {
            coefficient[coord[0] + sizeV[0] * coord[1] + sizeV[0] * sizeV[1] * coord[2] +
                        0 * (sizeV[0] * sizeV[1] * sizeV[2])], // n_x
            coefficient[coord[0] + sizeV[0] * coord[1] + sizeV[0] * sizeV[1] * coord[2] +
                        1 * (sizeV[0] * sizeV[1] * sizeV[2])], // n_y
            coefficient[coord[0] + sizeV[0] * coord[1] + sizeV[0] * sizeV[1] * coord[2] +
                        2 * (sizeV[0] * sizeV[1] * sizeV[2])], // n_z
            coefficient[coord[0] + sizeV[0] * coord[1] + sizeV[0] * sizeV[1] * coord[2] +
                        3 * (sizeV[0] * sizeV[1] * sizeV[2])], // cos
            coefficient[coord[0] + sizeV[0] * coord[1] + sizeV[0] * sizeV[1] * coord[2] +
                        4 * (sizeV[0] * sizeV[1] * sizeV[2])], // sin
    };

    /*
    Matrix3f R(cos(phi) * cos(theta), -sin(phi), cos(phi) * sin(theta),
       sin(phi) * cos(theta), cos(phi), sin(phi) * sin(theta),
       -sin(theta), 0, cos(theta));
    */
    Matrix3f R = rodriguesRotation(coef[0], coef[1], coef[2], coef[3], coef[4]);

    float proj = 0.0f;

    for (int i = 0; i < 3; i++) {
        // add scattering coefficient (read paper)
        // B->beam direction unit vector (src2voxel)
        // S->scattering base vector
        // G->grating sensivity vector
        // Vector3f S(basisVector[3 * i + 0], basisVector[3 * i + 1], basisVector[3 * i + 2]);

        // float vkm = abs(S * G);
        const int idxVoxel = coord[0] + sizeV[0] * coord[1] + sizeV[0] * sizeV[1] * coord[2] +
                            i * (sizeV[0] * sizeV[1] * sizeV[2]);

        Vector3f S(0.0f, 0.0f, 0.0f);
        S[i] = 1.0f;
        S = R * S;

        float vkm = B.cross(S).norm2() * abs(S * G);
        proj += vkm * vkm * geom->voxSize * ratio * devVoxel[idxVoxel];
        // printf("%d: %lf, %lf\n", i+1, vkm, proj);
    }

    // printf("angle: (%lf, %lf), back: %lf\n", phi, theta, backward);
    /*
    if (isnan(proj)) {
        printf("proj: (%lf), coef: (%lf, %lf, %lf, %lf)\n",
               proj, coef[0], coef[1], coef[2], coef[3]);
    }*/

    atomicAdd(&devProj[intU + sizeD[0] * (intV + 1) + sizeD[0] * sizeD[1] * n], c1 * proj);
    atomicAdd(&devProj[(intU + 1) + sizeD[0] * (intV + 1) + sizeD[0] * sizeD[1] * n], c2 * proj);
    atomicAdd(&devProj[(intU + 1) + sizeD[0] * intV + sizeD[0] * sizeD[1] * n], c3 * proj);
    atomicAdd(&devProj[intU + sizeD[0] * intV + sizeD[0] * sizeD[1] * n], c4 * proj);
}

__global__ void
backwardOrth(const float *devProj, const float *coefficient, float *devVoxelTmp, float *devVoxelFactor,
             const Geometry *geom, int cond, int y, int n, int it) {
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int z = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= geom->voxel || z >= geom->voxel) return;

    const int coord[4] = {x, y, z, n};

    float u = 0.0f, v = 0.0f;
    Vector3f B(0.0f, 0.0f, 0.0f), G(0.0f, 0.0f, 0.0f);
    rayCasting(u, v, B, G, cond, coord, *geom);

    int sizeV[3] = {geom->voxel, geom->voxel, geom->voxel};
    int sizeD[3] = {geom->detect, geom->detect, geom->nProj};
    if (!(0.55f < u && u < (float) sizeD[0] - 0.55f && 0.55f < v && v < (float) sizeD[1] - 0.55f))
        return;

    float u_tmp = u - 0.5f, v_tmp = v - 0.5f;
    int intU = floor(u_tmp), intV = floor(v_tmp);
    float c1 = (1.0f - (u_tmp - (float) intU)) * (v_tmp - (float) intV),
            c2 = (u_tmp - (float) intU) * (v_tmp - (float) intV),
            c3 = (u_tmp - (float) intU) * (1.0f - (v_tmp - (float) intV)),
            c4 = (1.0f - (u_tmp - (float) intU)) * (1.0f - (v_tmp - (float) intV));

    const float coef[5] = {
            coefficient[coord[0] + sizeV[0] * coord[1] + sizeV[0] * sizeV[1] * coord[2] +
                        0 * (sizeV[0] * sizeV[1] * sizeV[2])], // cos(phi)
            coefficient[coord[0] + sizeV[0] * coord[1] + sizeV[0] * sizeV[1] * coord[2] +
                        1 * (sizeV[0] * sizeV[1] * sizeV[2])], // sin(phi)
            coefficient[coord[0] + sizeV[0] * coord[1] + sizeV[0] * sizeV[1] * coord[2] +
                        2 * (sizeV[0] * sizeV[1] * sizeV[2])], // cos(theta)
            coefficient[coord[0] + sizeV[0] * coord[1] + sizeV[0] * sizeV[1] * coord[2] +
                        3 * (sizeV[0] * sizeV[1] * sizeV[2])], // sin(theta)
            coefficient[coord[0] + sizeV[0] * coord[1] + sizeV[0] * sizeV[1] * coord[2] +
                        4 * (sizeV[0] * sizeV[1] * sizeV[2])]
    };


    Matrix3f R = rodriguesRotation(coef[0], coef[1], coef[2], coef[3], coef[4]);

    for (int i = 0; i < 3; i++) {
        // calculate immutable geometry
        // add scattering coefficient (read paper)
        // B->beam direction unit vector (src2voxel)
        // S->scattering base vector
        // G->grating sensivity vector
        // v_km = (|B_m x S_k|<S_k*G>)^2

        Vector3f S(0.0f, 0.0f, 0.0f);
        S[i] = 1.0f;
        S = R * S;

        float vkm = B.cross(S).norm2() * abs(S * G);
        //float vkm = abs(S * G);

        const int idxVoxel = coord[0] + sizeV[0] * coord[2] + i * (sizeV[0] * sizeV[1]);
        const float backward = vkm * vkm * c1 * devProj[intU + sizeD[0] * (intV + 1) + sizeD[0] * sizeD[1] * n] +
                               vkm * vkm * c2 * devProj[(intU + 1) + sizeD[0] * (intV + 1) + sizeD[0] * sizeD[1] * n] +
                               vkm * vkm * c3 * devProj[(intU + 1) + sizeD[0] * intV + sizeD[0] * sizeD[1] * n] +
                               vkm * vkm * c4 * devProj[intU + sizeD[0] * intV + sizeD[0] * sizeD[1] * n];

        devVoxelFactor[idxVoxel] += (vkm * vkm);
        devVoxelTmp[idxVoxel] += backward;

    }
}

__both__ Matrix3f rodriguesRotation(float x, float y, float z, float cos, float sin) {
    // x, y, zを軸選択、軸と直交となる平面内での回転量で決定できる。not yet
    // ideally, onlt store theta, phi
    float eps = 1e-8f;
    if (std::sqrt(x * x + y * y + z * z) < eps) {
        Matrix3f R(1.0f, 0.0f, 0.0f,
                   0.0f, 1.0f, 0.0f,
                   0.0f, 0.0f, 1.0f);
        return R;
    }

    const float n_x = x / std::sqrt(x * x + y * y + z * z);
    const float n_y = y / std::sqrt(x * x + y * y + z * z);
    const float n_z = z / std::sqrt(x * x + y * y + z * z);

    Matrix3f rot1(n_x * n_x, n_x * n_y, n_x * n_z,
                  n_x * n_y, n_y * n_y, n_y * n_z,
                  n_x * n_z, n_y * n_z, n_z * n_z);

    Matrix3f rot2(cos, -n_z * sin, n_y * sin,
                  n_z * sin, cos, -n_x * sin,
                  -n_y * sin, n_x * sin, cos);

    Matrix3f rot = ((1.0f - cos) * rot1 + rot2);

    return rot;
}

__global__ void
calcNormalVector(const float *devVoxel, float *coefficient, int y, int it, const Geometry *geom, float *norm_loss) {
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int z = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= geom->voxel || z >= geom->voxel) return;

    int coord[3] = {x, y, z};
    int sizeV[3] = {geom->voxel, geom->voxel, geom->voxel};
    int sizeD[3] = {geom->detect, geom->detect, geom->nProj};

    const float coef[5] = {
            coefficient[coord[0] + sizeV[0] * coord[1] + sizeV[0] * sizeV[1] * coord[2] +
                        0 * (sizeV[0] * sizeV[1] * sizeV[2])], // ax_x
            coefficient[coord[0] + sizeV[0] * coord[1] + sizeV[0] * sizeV[1] * coord[2] +
                        1 * (sizeV[0] * sizeV[1] * sizeV[2])], // ax_y
            coefficient[coord[0] + sizeV[0] * coord[1] + sizeV[0] * sizeV[1] * coord[2] +
                        2 * (sizeV[0] * sizeV[1] * sizeV[2])], // ax_z
            coefficient[coord[0] + sizeV[0] * coord[1] + sizeV[0] * sizeV[1] * coord[2] +
                        3 * (sizeV[0] * sizeV[1] * sizeV[2])], // theta
            coefficient[coord[0] + sizeV[0] * coord[1] + sizeV[0] * sizeV[1] * coord[2] +
                        4 * (sizeV[0] * sizeV[1] * sizeV[2])]
    };

    Matrix3f R = rodriguesRotation(coef[0], coef[1], coef[2], coef[3], coef[4]);

    const float mu[3] =
            {devVoxel[coord[0] + sizeV[0] * coord[1] + sizeV[0] * sizeV[1] * coord[2] +
                      0 * (sizeV[0] * sizeV[1] * sizeV[2])],
             devVoxel[coord[0] + sizeV[0] * coord[1] + sizeV[0] * sizeV[1] * coord[2] +
                      1 * (sizeV[0] * sizeV[1] * sizeV[2])],
             devVoxel[coord[0] + sizeV[0] * coord[1] + sizeV[0] * sizeV[1] * coord[2] +
                      2 * (sizeV[0] * sizeV[1] * sizeV[2])]};

    Vector3f zx(mu[0], 0.0f, -mu[2]);
    Vector3f zy(0.0f, mu[1], -mu[2]);

    zx = R * zx;
    zy = R * zy;

    Vector3f norm = zx.cross(zy);
    norm.normalize();

    Vector3f base = {0.0f, 0.0f, 1.0f};
    /*
    if (mu[0] >= mu[1] && mu[0] >= mu[2]) {
        base = {1.0f, 0.0f, 0.0f};
    } else if (mu[1] >= mu[0] && mu[1] >= mu[2]) {
        base = {0.0f, 1.0f, 0.0f};
    } else {
        base = {0.0f, 0.0f, 1.0f};
    }*/
    Vector3f norm_diff = (R * base).cross(norm);
    Vector3f rotAxis = base.cross(norm);
    // printf("loss: %lf", norm_diff.norm2());
    float cos = base * norm;
    float sin = rotAxis.norm2();
    float diff = norm_diff.norm2();
    // printf("%lf, ", diff);
    norm_loss[x + sizeV[0] * y + sizeV[0] * sizeV[1] * z] = diff;

    if (cos > 1.0f) {
        cos = 1.0f;
        sin = 0.0f;
    } else if (cos < -1.0f) {
        cos = -1.0f;
        sin = 0.0f;
    }
    if (sin > 1.0f) {
        sin = 1.0f;
        cos = 0.0f;
    } else if (sin < -1.0f) {
        sin = -1.0f;
        cos = 0.0f;
    }

    /*
    if (isnan(theta))
        printf("norm: (%lf), cos(theta): (%lf)\n", rotAxis.norm2(), base * norm);
    */
    atan2(rotAxis[0], rotAxis[1]);
    coefficient[coord[0] + sizeV[0] * coord[1] + sizeV[0] * sizeV[1] * coord[2] +
                0 * (sizeV[0] * sizeV[1] * sizeV[2])] = rotAxis[0];
    coefficient[coord[0] + sizeV[0] * coord[1] + sizeV[0] * sizeV[1] * coord[2] +
                1 * (sizeV[0] * sizeV[1] * sizeV[2])] = rotAxis[1];
    coefficient[coord[0] + sizeV[0] * coord[1] + sizeV[0] * sizeV[1] * coord[2] +
                2 * (sizeV[0] * sizeV[1] * sizeV[2])] = rotAxis[2];
    coefficient[coord[0] + sizeV[0] * coord[1] + sizeV[0] * sizeV[1] * coord[2] +
                3 * (sizeV[0] * sizeV[1] * sizeV[2])] = cos;
    coefficient[coord[0] + sizeV[0] * coord[1] + sizeV[0] * sizeV[1] * coord[2] +
                4 * (sizeV[0] * sizeV[1] * sizeV[2])] = sin;
}

void convertNormVector(const Volume<float> *voxel, Volume<float> *md, const Volume<float> *coefficient) {
    for (int x = 0; x < NUM_VOXEL; x++) {
        for (int y = 0; y < NUM_VOXEL; y++) {
            for (int z = 0; z < NUM_VOXEL; z++) {
                float mu = (voxel[0](x, y, z) + voxel[1](x, y, z) + voxel[2](x, y, z)) / 3.0f;
                // printf("phi: %lf, theta: %lf\n", angle[0](x, y, z), angle[1](x, y, z));

                const float v[3] =
                        {voxel[0](x, y, z), voxel[1](x, y, z), voxel[2](x, y, z)};

                Vector3f zx(v[0], 0.0f, -v[2]);
                Vector3f zy(0.0f, v[1], -v[2]);

                const float coef[5] = {
                        coefficient[0](x, y, z), coefficient[1](x, y, z),
                        coefficient[2](x, y, z), coefficient[3](x, y, z), coefficient[4](x, y, z)};

                Matrix3f R = rodriguesRotation(coef[0], coef[1], coef[2], coef[3], coef[4]);

                zx = R * zx;
                zy = R * zy;

                Vector3f norm = zx.cross(zy);
                norm.normalize();

                /*
                printf("R:\n[%lf, %lf, %lf]\n[%lf, %lf, %lf],\n[%lf, %lf, %lf]\n",
                       R[0], R[1], R[2], R[3], R[4], R[5], R[6], R[7], R[8]);
                printf("base: [%lf, %lf, %lf]\n", base[0], base[1], base[2]);
                */

                for (int i = 0; i < 3; i++) {
                    md[i](x, y, z) = mu * norm[i];
                }
            }
        }
    }
}

__global__ void
forwardProj(float *devProj, float *devVoxel, Geometry *geom, int cond, int y, int n) {
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int z = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= geom->voxel || z >= geom->voxel) return;

    const int coord[4] = {x, y, z, n};
    forwardonDevice(coord, devProj, devVoxel, *geom, cond);
}

__global__ void
backwardProj(float *devProj, float *devVoxelTmp, float *devVoxelFactor, Geometry *geom, int cond,
             int y, int n) {
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int z = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= geom->voxel || z >= geom->voxel) return;

    const int coord[4] = {x, y, z, n};
    backwardonDevice(coord, devProj, devVoxelTmp, devVoxelFactor, *geom, cond);
}

__global__ void projRatio(float *devProj, const float *devSino, const Geometry *geom, int n, float *loss) {
    const int u = blockIdx.x * blockDim.x + threadIdx.x;
    const int v = blockIdx.y * blockDim.y + threadIdx.y;
    if (u >= geom->detect || v >= geom->detect) return;

    float threshold = 2.0f;
    const int idx = u + geom->detect * v + geom->detect * geom->detect * abs(n);
    atomicAdd(loss, abs(devSino[idx] - devProj[idx]));
    // printf("%lf\n", *loss);
    // const float div = devSino[idx] / devProj[idx];
    if (devProj[idx] != 0.0f) {
        // devProj[idx] = devSino[idx] / (devProj[idx] + 0.1f * (1.0f - exp(-abs(1.0f - devSino[idx] / devProj[idx]))));
        devProj[idx] = devSino[idx] / devProj[idx];
    }

    if (devProj[idx] > threshold) {
        devProj[idx] = threshold;
    }
}

__global__ void
voxelProduct(float *devVoxel, const float *devVoxelTmp, const float *devVoxelFactor, const Geometry *geom, int y) {
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int z = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= geom->voxel || z >= geom->voxel) return;

    for (int i = 0; i < NUM_BASIS_VECTOR; i++) {
        const int idxVoxel =
                x + geom->voxel * y + geom->voxel * geom->voxel * z + (geom->voxel * geom->voxel * geom->voxel) * i;
        const int idxOnPlane = x + geom->voxel * z + geom->voxel * geom->voxel * i;
        devVoxel[idxVoxel] = (devVoxelFactor[idxOnPlane] == 0.0f) ? 1e-8f : devVoxel[idxVoxel] *
                                                                            devVoxelTmp[idxOnPlane] /
                                                                            devVoxelFactor[idxOnPlane];

        if (isnan(devVoxel[idxVoxel])) {
            printf("voxel: %lf, tmp: %lf, fact: %lf\n", devVoxel[idxVoxel], devVoxelTmp[idxOnPlane],
                   devVoxelFactor[idxOnPlane]);
        }
    }
}

__global__ void projSubtract(float *devProj, const float *devSino, const Geometry *geom, int n, float *loss) {
    const int u = blockIdx.x * blockDim.x + threadIdx.x;
    const int v = blockIdx.y * blockDim.y + threadIdx.y;
    if (u >= geom->detect || v >= geom->detect) return;

    const int idx = u + geom->detect * v + geom->detect * geom->detect * abs(n);
    atomicAdd(loss, abs(devSino[idx] - devProj[idx]));
    // const float div = devSino[idx] / devProj[idx];
    devProj[idx] = devSino[idx] - devProj[idx];
    // a = b / c;
}

__global__ void projCompare(float *devCompare, const float *devSino, const float *devProj, const Geometry *geom, int n) {
    const int u = blockIdx.x * blockDim.x + threadIdx.x;
    const int v = blockIdx.y * blockDim.y + threadIdx.y;
    if (u >= geom->detect || v >= geom->detect) return;

    const int idx = u + geom->detect * v + geom->detect * geom->detect * abs(n);
    // const float div = devSino[idx] / devProj[idx];
    // devCompare[idx] = devSino[idx] - devProj[idx];
    devCompare[idx] = devSino[idx] / devProj[idx];
    // a = b / c;
}


__global__ void
voxelPlus(float *devVoxel, const float *devVoxelTmp, float alpha, const Geometry *geom, int y) {
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int z = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= geom->voxel || z >= geom->voxel) return;

    for (int i = 0; i < NUM_BASIS_VECTOR; i++) {
        const int idxVoxel =
                x + geom->voxel * y + geom->voxel * geom->voxel * z + (geom->voxel * geom->voxel * geom->voxel) * i;
        const int idxOnPlane = x + geom->voxel * z + geom->voxel * geom->voxel * i;
        devVoxel[idxVoxel] = devVoxel[idxVoxel] + alpha * devVoxelTmp[idxOnPlane];
    }
}

__global__ void voxelSqrtFromSrc(float *hostVoxel, const float *devVoxel, const Geometry *geom, int y) {
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int z = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= geom->voxel || z >= geom->voxel) return;

    for (int i = 0; i < NUM_BASIS_VECTOR; i++) {
        const int idxVoxel =
                x + geom->voxel * y + geom->voxel * geom->voxel * z + (geom->voxel * geom->voxel * geom->voxel) * i;
        hostVoxel[idxVoxel] = sqrt(abs(devVoxel[idxVoxel]));
    }
}

__global__ void voxelSqrt(float *devVoxel, const Geometry *geom, int y) {
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int z = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= geom->voxel || z >= geom->voxel) return;

    for (int i = 0; i < NUM_BASIS_VECTOR; i++) {
        const int idxVoxel =
                x + geom->voxel * y + geom->voxel * geom->voxel * z + (geom->voxel * geom->voxel * geom->voxel) * i;

        devVoxel[idxVoxel] = (devVoxel[idxVoxel] < 0.0f) ? 0.0f : sqrt(devVoxel[idxVoxel]);
    }
}

__device__ void
forwardonDevice(const int coord[4], float *devProj, const float *devVoxel,
                const Geometry &geom, int cond) {

    int sizeV[3] = {geom.voxel, geom.voxel, geom.voxel};
    int sizeD[3] = {geom.detect, geom.detect, geom.nProj};

    float u, v;
    Vector3f B, G;
    rayCasting(u, v, B, G, cond, coord, geom);

    if (!(0.55f < u && u < (float) sizeD[0] - 0.55f && 0.55f < v && v < (float) sizeD[1] - 0.55f))
        return;

    float u_tmp = u - 0.5f, v_tmp = v - 0.5f;
    int intU = floor(u_tmp), intV = floor(v_tmp);
    float c1 = (1.0f - (u_tmp - (float) intU)) * (v_tmp - (float) intV),
            c2 = (u_tmp - (float) intU) * (v_tmp - (float) intV),
            c3 = (u_tmp - (float) intU) * (1.0f - (v_tmp - (float) intV)),
            c4 = (1.0f - (u_tmp - (float) intU)) * (1.0f - (v_tmp - (float) intV));

    const int n = abs(coord[3]);

    const float ratio = (geom.voxSize * geom.voxSize) /
                        (geom.detSize * (geom.sod / geom.sdd) * geom.detSize * (geom.sod / geom.sdd));
    const int idxVoxel =
            coord[0] + sizeV[0] * coord[1] + sizeV[0] * sizeV[1] * coord[2] +
            cond * (sizeV[0] * sizeV[1] * sizeV[2]);
    atomicAdd(&devProj[intU + sizeD[0] * (intV + 1) + sizeD[0] * sizeD[1] * n],
              c1 * geom.voxSize * ratio * devVoxel[idxVoxel]);
    atomicAdd(&devProj[(intU + 1) + sizeD[0] * (intV + 1) + sizeD[0] * sizeD[1] * n],
              c2 * geom.voxSize * ratio * devVoxel[idxVoxel]);
    atomicAdd(&devProj[(intU + 1) + sizeD[0] * intV + sizeD[0] * sizeD[1] * n],
              c3 * geom.voxSize * ratio * devVoxel[idxVoxel]);
    atomicAdd(&devProj[intU + sizeD[0] * intV + sizeD[0] * sizeD[1] * n],
              c4 * geom.voxSize * ratio * devVoxel[idxVoxel]);
}

__device__ void
backwardonDevice(const int coord[4], const float *devProj, float *devVoxelTmp, float *devVoxelFactor,
                 const Geometry &geom, int cond) {

    int sizeV[3] = {geom.voxel, geom.voxel, geom.voxel};
    int sizeD[3] = {geom.detect, geom.detect, geom.nProj};

    float u, v;
    Vector3f B, G;
    rayCasting(u, v, B, G, cond, coord, geom);

    if (!(0.55f < u && u < (float) sizeD[0] - 0.55f && 0.55f < v && v < (float) sizeD[1] - 0.55f))
        return;

    const int n = abs(coord[3]);

    float u_tmp = u - 0.5f, v_tmp = v - 0.5f;
    int intU = floor(u_tmp), intV = floor(v_tmp);
    float c1 = (1.0f - (u_tmp - (float) intU)) * (v_tmp - (float) intV), c2 =
            (u_tmp - (float) intU) * (v_tmp - (float) intV),
            c3 = (u_tmp - (float) intU) * (1.0f - (v_tmp - (float) intV)), c4 =
            (1.0f - (u_tmp - (float) intU)) * (1.0f - (v_tmp - (float) intV));

    const int idxVoxel = coord[0] + sizeV[0] * coord[2] + cond * (sizeV[0] * sizeV[1]);
    const float numBack = c1 * devProj[intU + sizeD[0] * (intV + 1) + sizeD[0] * sizeD[1] * n] +
                          c2 * devProj[(intU + 1) + sizeD[0] * (intV + 1) + sizeD[0] * sizeD[1] * n] +
                          c3 * devProj[(intU + 1) + sizeD[0] * intV + sizeD[0] * sizeD[1] * n] +
                          c4 * devProj[intU + sizeD[0] * intV + sizeD[0] * sizeD[1] * n];

    devVoxelFactor[idxVoxel] += 1.0f;
    devVoxelTmp[idxVoxel] += numBack;
}

__device__ void
forwardXTTonDevice(const int coord[4], float *devProj, const float *devVoxel,
                   const Geometry &geom, int cond) {

    int sizeV[3] = {geom.voxel, geom.voxel, geom.voxel};
    int sizeD[3] = {geom.detect, geom.detect, geom.nProj};

    float u = 0.0f, v = 0.0f;
    Vector3f B(0.0f, 0.0f, 0.0f), G(0.0f, 0.0f, 0.0f);
    rayCasting(u, v, B, G, cond, coord, geom);

    if (!(0.55f < u && u < (float) sizeD[0] - 0.55f && 0.55f < v && v < (float) sizeD[1] - 0.55f))
        return;

    const int n = abs(coord[3]);

    float u_tmp = u - 0.5f, v_tmp = v - 0.5f;
    int intU = floor(u_tmp), intV = floor(v_tmp);
    float c1 = (1.0f - (u_tmp - (float) intU)) * (v_tmp - (float) intV),
            c2 = (u_tmp - (float) intU) * (v_tmp - (float) intV),
            c3 = (u_tmp - (float) intU) * (1.0f - (v_tmp - (float) intV)),
            c4 = (1.0f - (u_tmp - (float) intU)) * (1.0f - (v_tmp - (float) intV));

    const float ratio = (geom.voxSize * geom.voxSize) /
                        (geom.detSize * (geom.sod / geom.sdd) * geom.detSize * (geom.sod / geom.sdd));

    float proj = 0.0f;
    for (int i = 0; i < NUM_BASIS_VECTOR; i++) {
        // add scattering coefficient (read paper)
        // B->beam direction unit vector (src2voxel)
        // S->scattering base vector
        // G->grating sensivity vector
        Vector3f S(basisVector[3 * i + 0], basisVector[3 * i + 1], basisVector[3 * i + 2]);
        float vkm = B.cross(S).norm2() * abs(S * G);
        // float vkm = abs(S * G);
        const int idxVoxel =
                coord[0] + sizeV[0] * coord[1] + sizeV[0] * sizeV[1] * coord[2] +
                i * (sizeV[0] * sizeV[1] * sizeV[2]);
        proj += vkm * vkm * geom.voxSize * ratio * devVoxel[idxVoxel];
        // printf("%d: %lf\n", i+1, vkm);
        // printf("sinogram: %lf\n", devSino[intU + sizeD[0] * intV + sizeD[0] * sizeD[1] * n]);
    }
    atomicAdd(&devProj[intU + sizeD[0] * (intV + 1) + sizeD[0] * sizeD[1] * n], c1 * proj);
    atomicAdd(&devProj[(intU + 1) + sizeD[0] * (intV + 1) + sizeD[0] * sizeD[1] * n], c2 * proj);
    atomicAdd(&devProj[(intU + 1) + sizeD[0] * intV + sizeD[0] * sizeD[1] * n], c3 * proj);
    atomicAdd(&devProj[intU + sizeD[0] * intV + sizeD[0] * sizeD[1] * n], c4 * proj);

}

// change to class
__device__ void
backwardXTTonDevice(const int coord[4], const float *devProj, float *devVoxelTmp, float *devVoxelFactor,
                    const Geometry &geom, int cond) {

    int sizeV[3] = {geom.voxel, geom.voxel, geom.voxel};
    int sizeD[3] = {geom.detect, geom.detect, geom.nProj};

    float u = 0.0f, v = 0.0f;
    Vector3f B(0.0f, 0.0f, 0.0f), G(0.0f, 0.0f, 0.0f);
    rayCasting(u, v, B, G, cond, coord, geom);

    if (!(0.55f < u && u < (float) sizeD[0] - 0.55f && 0.55f < v && v < (float) sizeD[1] - 0.55f))
        return;

    const int n = abs(coord[3]);
    float u_tmp = u - 0.5f, v_tmp = v - 0.5f;
    int intU = floor(u_tmp), intV = floor(v_tmp);
    float c1 = (1.0f - (u_tmp - (float) intU)) * (v_tmp - (float) intV), c2 =
            (u_tmp - (float) intU) * (v_tmp - (float) intV),
            c3 = (u_tmp - (float) intU) * (1.0f - (v_tmp - (float) intV)), c4 =
            (1.0f - (u_tmp - (float) intU)) * (1.0f - (v_tmp - (float) intV));

    for (int i = 0; i < NUM_BASIS_VECTOR; i++) {
        // calculate immutable geometry
        // add scattering coefficient (read paper)
        // B->beam direction unit vector (src2voxel)
        // S->scattering base vector
        // G->grating sensivity vector
        // v_km = (|B_m x S_k|<S_k*G>)^2
        Vector3f S(basisVector[3 * i + 0], basisVector[3 * i + 1], basisVector[3 * i + 2]);
        float vkm = B.cross(S).norm2() * abs(S * G);
        //float vkm = abs(S * G);

        const int idxVoxel = coord[0] + sizeV[0] * coord[2] + i * (sizeV[0] * sizeV[1]);
        const float backForward = vkm * vkm * c1 * devProj[intU + sizeD[0] * (intV + 1) + sizeD[0] * sizeD[1] * n] +
                                  vkm * vkm * c2 *
                                  devProj[(intU + 1) + sizeD[0] * (intV + 1) + sizeD[0] * sizeD[1] * n] +
                                  vkm * vkm * c3 * devProj[(intU + 1) + sizeD[0] * intV + sizeD[0] * sizeD[1] * n] +
                                  vkm * vkm * c4 * devProj[intU + sizeD[0] * intV + sizeD[0] * sizeD[1] * n];

        devVoxelFactor[idxVoxel] += (vkm * vkm);
        devVoxelTmp[idxVoxel] += backForward;
    }
}

__device__ void
rayCasting(float &u, float &v, Vector3f &B, Vector3f &G, int cond, const int coord[4],
           const Geometry &geom) {

    const int n = coord[3];
    int sizeV[3] = {geom.voxel, geom.voxel, geom.voxel};
    int sizeD[3] = {geom.detect, geom.detect, geom.nProj};

    const float theta = 2.0f * (float) M_PI * (float) n / (float) sizeD[2];
    Vector3f offset(INIT_OFFSET[3 * cond + 0], INIT_OFFSET[3 * cond + 1], INIT_OFFSET[3 * cond + 2]);

    // need to modify
    // need multiply Rotate matrix (axis and rotation geom) to vecSod
    Matrix3f Rotate(cosf(theta), -sinf(theta), 0.0f, sinf(theta), cosf(theta), 0.0f, 0.0f, 0.0f, 1.0f);

    Matrix3f condR(elemR[9 * cond + 0], elemR[9 * cond + 1], elemR[9 * cond + 2],
                   elemR[9 * cond + 3], elemR[9 * cond + 4], elemR[9 * cond + 5],
                   elemR[9 * cond + 6], elemR[9 * cond + 7], elemR[9 * cond + 8]);
    Vector3f t(elemT[3 * cond + 0], elemT[3 * cond + 1], elemT[3 * cond + 2]);

    Rotate = condR * Rotate; // no need
    offset = Rotate * offset;
    Vector3f vecSod(0.0f, geom.sod, 0.0f);
    Vector3f base1(1.0f, 0.0f, 0.0f);
    Vector3f base2(0.0f, 0.0f, -1.0f);

    vecSod = Rotate * vecSod;

    Vector3f vecVoxel(
            (2.0f * (float) coord[0] - (float) sizeV[0] + 1.0f) * 0.5f * geom.voxSize - offset[0] - t[0], // -R * offset
            (2.0f * (float) coord[1] - (float) sizeV[1] + 1.0f) * 0.5f * geom.voxSize - offset[1] - t[1],
            (2.0f * (float) coord[2] - (float) sizeV[2] + 1.0f) * 0.5f * geom.voxSize - offset[2] - t[2]);

    // Source to voxel center
    Vector3f src2cent(-vecSod[0], -vecSod[1], -vecSod[2]);
    // Source to voxel
    Vector3f src2voxel(vecVoxel[0] + src2cent[0],
                       vecVoxel[1] + src2cent[1],
                       vecVoxel[2] + src2cent[2]);

    // src2voxel and plane that have vecSod norm vector
    // p = s + t*d (vector p is on the plane, s is vecSod, d is src2voxel)
    const float coeff = -(vecSod * vecSod) / (vecSod * src2voxel); // -(n * s) / (n * v)
    Vector3f p = vecSod + coeff * src2voxel;

    u = (p * (Rotate * base1)) * (geom.sdd / geom.sod) / geom.detSize + 0.5f * (float) (sizeD[0]);
    v = (p * (Rotate * base2)) * (geom.sdd / geom.sod) / geom.detSize + 0.5f * (float) (sizeD[1]);

    B = src2voxel;
    B.normalize();
    G = Rotate * Vector3f(0.0f, 0.0f, 1.0f);
}



