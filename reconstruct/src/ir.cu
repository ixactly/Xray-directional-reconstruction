#include "hip/hip_runtime.h"
//
// Created by tomokimori on 22/07/20.
//
#include <geometry.h>
#include <ir.cuh>
#include <random>
#include <params.h>
#include <cmath>


__global__ void
forwardProjXTTbyFiber(float *devProj, float *devVoxel, Geometry &geom, int cond,
                      int y, int p, float *devDirection) {
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int z = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= geom.voxel || z >= geom.voxel) return;

    const int coord[4] = {x, y, z, p};
    int sizeV[3] = {geom.voxel, geom.voxel, geom.voxel};
    int sizeD[3] = {geom.detect, geom.detect, geom.nProj};

    float u = 0.0f, v = 0.0f;
    Vector3f B(0.0f, 0.0f, 0.0f), G(0.0f, 0.0f, 0.0f);
    rayCasting(u, v, B, G, cond, coord, geom);

    Vector3f F(devDirection[coord[0] + sizeV[0] * coord[1] + sizeV[0] * sizeV[1] * coord[2] +
                            0 * (sizeV[0] * sizeV[1] * sizeV[2])],
               devDirection[coord[0] + sizeV[0] * coord[1] + sizeV[0] * sizeV[1] * coord[2] +
                            1 * (sizeV[0] * sizeV[1] * sizeV[2])],
               devDirection[coord[0] + sizeV[0] * coord[1] + sizeV[0] * sizeV[1] * coord[2] +
                            2 * (sizeV[0] * sizeV[1] * sizeV[2])]);
    if (F.norm2() < 1e-10)
        F = Vector3f(1.0f, 1.0f, 1.0f);
    F.normalize();

    if (!(0.55f < u && u < (float) sizeD[0] - 0.55f && 0.55f < v && v < (float) sizeD[1] - 0.55f) ||
        abs(F * B) > fdThresh)
        return;

    const int n = abs(coord[3]);

    float u_tmp = u - 0.5f, v_tmp = v - 0.5f;
    int intU = floor(u_tmp), intV = floor(v_tmp);
    float c1 = (1.0f - (u_tmp - (float) intU)) * (v_tmp - (float) intV),
            c2 = (u_tmp - (float) intU) * (v_tmp - (float) intV),
            c3 = (u_tmp - (float) intU) * (1.0f - (v_tmp - (float) intV)),
            c4 = (1.0f - (u_tmp - (float) intU)) * (1.0f - (v_tmp - (float) intV));

    const float ratio = (geom.voxSize * geom.voxSize) /
                        (geom.detSize * (geom.sod / geom.sdd) * geom.detSize * (geom.sod / geom.sdd));

    float proj = 0.0f;
    for (int i = 0; i < NUM_BASIS_VECTOR; i++) {
        // add scattering coefficient (read paper)
        // B->beam direction unit vector (src2voxel)
        // S->scattering base vector
        // G->grating sensivity vector

        const int idxVoxel =
                coord[0] + sizeV[0] * coord[1] + sizeV[0] * sizeV[1] * coord[2] + i * (sizeV[0] * sizeV[1] * sizeV[2]);
        Vector3f S(basisVector[3 * i + 0], basisVector[3 * i + 1], basisVector[3 * i + 2]);
        float vkm = B.cross(S).norm2() * abs(S * G);
        // float vkm = abs(S * G);

        proj += vkm * vkm * geom.voxSize * ratio * devVoxel[idxVoxel];
    }
    atomicAdd(&devProj[intU + sizeD[0] * (intV + 1) + sizeD[0] * sizeD[1] * n], c1 * proj);
    atomicAdd(&devProj[(intU + 1) + sizeD[0] * (intV + 1) + sizeD[0] * sizeD[1] * n], c2 * proj);
    atomicAdd(&devProj[(intU + 1) + sizeD[0] * intV + sizeD[0] * sizeD[1] * n], c3 * proj);
    atomicAdd(&devProj[intU + sizeD[0] * intV + sizeD[0] * sizeD[1] * n], c4 * proj);
}

__global__ void
backwardProjXTTbyFiber(float *devProj, float *devVoxelTmp, float *devVoxelFactor, Geometry &geom, int cond,
                       int y, int p, float *devDirection) {
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int z = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= geom.voxel || z >= geom.voxel) return;

    const int coord[4] = {x, y, z, p};
    int sizeV[3] = {geom.voxel, geom.voxel, geom.voxel};
    int sizeD[3] = {geom.detect, geom.detect, geom.nProj};

    float u = 0.0f, v = 0.0f;
    Vector3f B(0.0f, 0.0f, 0.0f), G(0.0f, 0.0f, 0.0f);
    rayCasting(u, v, B, G, cond, coord, geom);

    Vector3f F(devDirection[coord[0] + sizeV[0] * coord[1] + sizeV[0] * sizeV[1] * coord[2] +
                            0 * (sizeV[0] * sizeV[1] * sizeV[2])],
               devDirection[coord[0] + sizeV[0] * coord[1] + sizeV[0] * sizeV[1] * coord[2] +
                            1 * (sizeV[0] * sizeV[1] * sizeV[2])],
               devDirection[coord[0] + sizeV[0] * coord[1] + sizeV[0] * sizeV[1] * coord[2] +
                            2 * (sizeV[0] * sizeV[1] * sizeV[2])]);
    if (F.norm2() < 1e-10)
        F = Vector3f(1.0f, 1.0f, 1.0f);
    F.normalize();

    if (!(0.55f < u && u < (float) sizeD[0] - 0.55f && 0.55f < v && v < (float) sizeD[1] - 0.55f) ||
        (abs(F * B) > fdThresh))
        return;

    const int n = abs(coord[3]);
    float u_tmp = u - 0.5f, v_tmp = v - 0.5f;
    int intU = floor(u_tmp), intV = floor(v_tmp);
    float c1 = (1.0f - (u_tmp - (float) intU)) * (v_tmp - (float) intV), c2 =
            (u_tmp - (float) intU) * (v_tmp - (float) intV),
            c3 = (u_tmp - (float) intU) * (1.0f - (v_tmp - (float) intV)), c4 =
            (1.0f - (u_tmp - (float) intU)) * (1.0f - (v_tmp - (float) intV));

    for (int i = 0; i < NUM_BASIS_VECTOR; i++) {
        // calculate immutable geometry
        // add scattering coefficient (read paper)
        // B->beam direction unit vector (src2voxel)
        // S->scattering base vector
        // G->grating sensivity vector
        // v_km = (|B_m x S_k|<S_k*G>)^2
        Vector3f S(basisVector[3 * i + 0], basisVector[3 * i + 1], basisVector[3 * i + 2]);
        float vkm = B.cross(S).norm2() * abs(S * G);
        // float vkm = abs(S * G);

        const int idxVoxel = coord[0] + sizeV[0] * coord[2] + i * (sizeV[0] * sizeV[1]);
        const float backForward = vkm * vkm * c1 * devProj[intU + sizeD[0] * (intV + 1) + sizeD[0] * sizeD[1] * n] +
                                  vkm * vkm * c2 *
                                  devProj[(intU + 1) + sizeD[0] * (intV + 1) + sizeD[0] * sizeD[1] * n] +
                                  vkm * vkm * c3 * devProj[(intU + 1) + sizeD[0] * intV + sizeD[0] * sizeD[1] * n] +
                                  vkm * vkm * c4 * devProj[intU + sizeD[0] * intV + sizeD[0] * sizeD[1] * n];

        devVoxelFactor[idxVoxel] += (vkm * vkm);
        devVoxelTmp[idxVoxel] += backForward;
    }
}

__global__ void
forwardProjXTT(float *devProj, float *devVoxel, Geometry *geom, int cond,
               int y, int n) {
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int z = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= geom->voxel || z >= geom->voxel) return;

    const int coord[4] = {x, y, z, n};
    forwardXTTonDevice(coord, devProj, devVoxel, *geom, cond);
}

__global__ void
backwardProjXTT(float *devProj, float *devVoxelTmp, float *devVoxelFactor, Geometry *geom, int cond,
                int y, int n) {
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int z = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= geom->voxel || z >= geom->voxel) return;

    const int coord[4] = {x, y, z, n};
    backwardXTTonDevice(coord, devProj, devVoxelTmp, devVoxelFactor, *geom, cond);
}

__global__ void
forwardOrth(float *devProj, const float *devVoxel, const float *coefficient, int cond, int y, int n, int it,
            Geometry *geom) {
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int z = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= geom->voxel || z >= geom->voxel) return;

    const int coord[4] = {x, y, z, n};
    int sizeV[3] = {geom->voxel, geom->voxel, geom->voxel};
    int sizeD[3] = {geom->detect, geom->detect, geom->nProj};

    float u = 0.0f, v = 0.0f;
    Vector3f B(0.0f, 0.0f, 0.0f), G(0.0f, 0.0f, 0.0f);
    rayCasting(u, v, B, G, cond, coord, *geom);

    if (!(0.55f < u && u < (float) sizeD[0] - 0.55f && 0.55f < v && v < (float) sizeD[1] - 0.55f))
        return;

    float u_tmp = u - 0.5f, v_tmp = v - 0.5f;
    int intU = floor(u_tmp), intV = floor(v_tmp);
    float c1 = (1.0f - (u_tmp - (float) intU)) * (v_tmp - (float) intV),
            c2 = (u_tmp - (float) intU) * (v_tmp - (float) intV),
            c3 = (u_tmp - (float) intU) * (1.0f - (v_tmp - (float) intV)),
            c4 = (1.0f - (u_tmp - (float) intU)) * (1.0f - (v_tmp - (float) intV));

    const float ratio = (geom->voxSize * geom->voxSize) /
                        (geom->detSize * (geom->sod / geom->sdd) * geom->detSize * (geom->sod / geom->sdd));
    const float phi_c = coefficient[coord[0] + sizeV[0] * coord[1] + sizeV[0] * sizeV[1] * coord[2] +
                                    0 * (sizeV[0] * sizeV[1] * sizeV[2])];
    const float cos_c = coefficient[coord[0] + sizeV[0] * coord[1] + sizeV[0] * sizeV[1] * coord[2] +
                                    1 * (sizeV[0] * sizeV[1] * sizeV[2])];
    const float coef[5] = {cos(phi_c), sin(phi_c), 0, cos_c, sqrt(1 - cos_c * cos_c)};
    /*
    Matrix3f R(cos(phi) * cos(theta), -sin(phi), cos(phi) * sin(theta),
       sin(phi) * cos(theta), cos(phi), sin(phi) * sin(theta),
       -sin(theta), 0, cos(theta));
    */
    Matrix3f R = rodriguesRotation(coef[0], coef[1], coef[2], coef[3], coef[4]);

    float proj = 0.0f;

    // bool out = (x == 50 && y == 50 && z == 50 && (n == 0 || n == 45));
    /*
    if (out)
        printf("B: (%lf, %lf, %lf), G: (%lf, %lf, %lf)\n", B(0), B(1), B(2), G(0), G(1), G(2));
        */

    for (int i = 0; i < NUM_BASIS_VECTOR; i++) {
        // add scattering coefficient (read paper)
        // B->beam direction unit vector (src2voxel)
        // S->scattering base vector
        // G->grating sensivity vector
        // Vector3f S(basisVector[3 * i + 0], basisVector[3 * i + 1], basisVector[3 * i + 2]);

        // float vkm = abs(S * G);
        const int idxVoxel = coord[0] + sizeV[0] * coord[1] + sizeV[0] * sizeV[1] * coord[2] +
                             i * (sizeV[0] * sizeV[1] * sizeV[2]);

        Vector3f S(basisVector[3 * i + 0], basisVector[3 * i + 1], basisVector[3 * i + 2]);
        S = R * S;

        float vkm = B.cross(S).norm2() * abs(S * G);
        // float vkm = abs(S * G);
        proj += vkm * vkm * geom->voxSize * ratio * devVoxel[idxVoxel];
    }

    atomicAdd(&devProj[intU + sizeD[0] * (intV + 1) + sizeD[0] * sizeD[1] * n], c1 * proj);
    atomicAdd(&devProj[(intU + 1) + sizeD[0] * (intV + 1) + sizeD[0] * sizeD[1] * n], c2 * proj);
    atomicAdd(&devProj[(intU + 1) + sizeD[0] * intV + sizeD[0] * sizeD[1] * n], c3 * proj);
    atomicAdd(&devProj[intU + sizeD[0] * intV + sizeD[0] * sizeD[1] * n], c4 * proj);
}

__global__ void
backwardOrth(const float *devProj, const float *coefficient, float *devVoxelTmp, float *devVoxelFactor,
             const Geometry *geom, int cond, int y, int n, int it) {
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int z = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= geom->voxel || z >= geom->voxel) return;

    const int coord[4] = {x, y, z, n};

    float u = 0.0f, v = 0.0f;
    Vector3f B(0.0f, 0.0f, 0.0f), G(0.0f, 0.0f, 0.0f);
    rayCasting(u, v, B, G, cond, coord, *geom);

    int sizeV[3] = {geom->voxel, geom->voxel, geom->voxel};
    int sizeD[3] = {geom->detect, geom->detect, geom->nProj};
    if (!(0.55f < u && u < (float) sizeD[0] - 0.55f && 0.55f < v && v < (float) sizeD[1] - 0.55f))
        return;

    float u_tmp = u - 0.5f, v_tmp = v - 0.5f;
    int intU = floor(u_tmp), intV = floor(v_tmp);
    float c1 = (1.0f - (u_tmp - (float) intU)) * (v_tmp - (float) intV),
            c2 = (u_tmp - (float) intU) * (v_tmp - (float) intV),
            c3 = (u_tmp - (float) intU) * (1.0f - (v_tmp - (float) intV)),
            c4 = (1.0f - (u_tmp - (float) intU)) * (1.0f - (v_tmp - (float) intV));

    const float phi_c = coefficient[coord[0] + sizeV[0] * coord[1] + sizeV[0] * sizeV[1] * coord[2] +
                                    0 * (sizeV[0] * sizeV[1] * sizeV[2])];
    const float cos_c = coefficient[coord[0] + sizeV[0] * coord[1] + sizeV[0] * sizeV[1] * coord[2] +
                                    1 * (sizeV[0] * sizeV[1] * sizeV[2])];
    const float coef[5] = {cos(phi_c), sin(phi_c), 0, cos_c, sqrt(1 - cos_c * cos_c)};

    Matrix3f R = rodriguesRotation(coef[0], coef[1], coef[2], coef[3], coef[4]);

    for (int i = 0; i < NUM_BASIS_VECTOR; i++) {
        // calculate immutable geometry
        // add scattering coefficient (read paper)
        // B->beam direction unit vector (src2voxel)
        // S->scattering base vector
        // G->grating sensivity vector
        // v_km = (|B_m x S_k|<S_k*G>)^2

        Vector3f S(basisVector[3 * i + 0], basisVector[3 * i + 1], basisVector[3 * i + 2]);
        S = R * S;

        float vkm = B.cross(S).norm2() * abs(S * G);
        //float vkm = abs(S * G);

        const int idxVoxel = coord[0] + sizeV[0] * coord[2] + i * (sizeV[0] * sizeV[1]);
        const float backward = vkm * vkm * c1 * devProj[intU + sizeD[0] * (intV + 1) + sizeD[0] * sizeD[1] * n] +
                               vkm * vkm * c2 * devProj[(intU + 1) + sizeD[0] * (intV + 1) + sizeD[0] * sizeD[1] * n] +
                               vkm * vkm * c3 * devProj[(intU + 1) + sizeD[0] * intV + sizeD[0] * sizeD[1] * n] +
                               vkm * vkm * c4 * devProj[intU + sizeD[0] * intV + sizeD[0] * sizeD[1] * n];

        devVoxelFactor[idxVoxel] += (vkm * vkm);
        devVoxelTmp[idxVoxel] += backward;

    }
}

__both__ Matrix3f rodriguesRotation(float x, float y, float z, float cos, float sin) {
    // x, y, zを軸選択、軸と直交となる平面内での回転量で決定できる。not yet
    // ideally, onlt store theta, phi
    float eps = 1e-8f;
    if (std::sqrt(x * x + y * y + z * z) < eps) {
        Matrix3f R(1.0f, 0.0f, 0.0f,
                   0.0f, 1.0f, 0.0f,
                   0.0f, 0.0f, 1.0f);
        return R;
    }

    const float n_x = x / std::sqrt(x * x + y * y + z * z);
    const float n_y = y / std::sqrt(x * x + y * y + z * z);
    const float n_z = z / std::sqrt(x * x + y * y + z * z);

    Matrix3f rot1(n_x * n_x, n_x * n_y, n_x * n_z,
                  n_x * n_y, n_y * n_y, n_y * n_z,
                  n_x * n_z, n_y * n_z, n_z * n_z);

    Matrix3f rot2(cos, -n_z * sin, n_y * sin,
                  n_z * sin, cos, -n_x * sin,
                  -n_y * sin, n_x * sin, cos);

    Matrix3f rot = ((1.0f - cos) * rot1 + rot2);
    return rot;
}

__global__ void
calcNormalVectorThreeDirec(float *devVoxel, float *devCoef, int y, int it, const Geometry *geom, float *norm_loss,
                           hiprandState *curandStates, float judge) {
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int z = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= geom->voxel || z >= geom->voxel) return;

    int coord[3] = {x, y, z};
    int sizeV[3] = {geom->voxel, geom->voxel, geom->voxel};
    int sizeD[3] = {geom->detect, geom->detect, geom->nProj};

    const float phi_c = devCoef[coord[0] + sizeV[0] * coord[1] + sizeV[0] * sizeV[1] * coord[2] +
                                0 * (sizeV[0] * sizeV[1] * sizeV[2])];
    const float cos_c = devCoef[coord[0] + sizeV[0] * coord[1] + sizeV[0] * sizeV[1] * coord[2] +
                                1 * (sizeV[0] * sizeV[1] * sizeV[2])];

    const float coef[5] = {cos(phi_c), sin(phi_c), 0, cos_c, sqrt(1.0f - cos_c * cos_c)};

    const float mu[3] =
            {devVoxel[coord[0] + sizeV[0] * coord[1] + sizeV[0] * sizeV[1] * coord[2] +
                      0 * (sizeV[0] * sizeV[1] * sizeV[2])],
             devVoxel[coord[0] + sizeV[0] * coord[1] + sizeV[0] * sizeV[1] * coord[2] +
                      1 * (sizeV[0] * sizeV[1] * sizeV[2])],
             devVoxel[coord[0] + sizeV[0] * coord[1] + sizeV[0] * sizeV[1] * coord[2] +
                      2 * (sizeV[0] * sizeV[1] * sizeV[2])]};

    // float rand_rotate = hiprand_uniform(&curandStates[z * sizeV[0] + x]);
    float rand_rotate = judge;
    // float rand_rotate = judge;
    // printf("rand: %lf\n", judge);

    float mu1 = mu[1], mu2 = mu[2];

    if (rand_rotate > .75f) {
        mu1 = mu[1];
        mu2 = mu[2];
    } else if (rand_rotate > .50f) {
        mu1 = -mu[1];
        mu2 = mu[2];
    } else if (rand_rotate > .25f) {
        mu1 = -mu[1];
        mu2 = -mu[2];
    } else {
        mu1 = mu[1];
        mu2 = -mu[2];
    }

    Vector3f vec1(mu[0] * basisVector[3 * 0 + 0] - mu1 * basisVector[3 * 1 + 0],
                  mu[0] * basisVector[3 * 0 + 1] - mu1 * basisVector[3 * 1 + 1],
                  mu[0] * basisVector[3 * 0 + 2] - mu1 * basisVector[3 * 1 + 2]);
    Vector3f vec2(mu[0] * basisVector[3 * 0 + 0] - mu2 * basisVector[3 * 2 + 0],
                  mu[0] * basisVector[3 * 0 + 1] - mu2 * basisVector[3 * 2 + 1],
                  mu[0] * basisVector[3 * 0 + 2] - mu2 * basisVector[3 * 2 + 2]);

    Vector3f norm = vec1.cross(vec2);
    norm.normalize();
    // Vector3f normal = (1.0f / (mu[0] + eps)) * S1 + (1.0f / (mu[1] + eps)) * S2 + (1.0f / (mu[2] + eps)) * S3;
    /*
    bool out = (y == 50 && z == 50);
    if (out) {
        printf("x: %d, n1: %lf, n2: %lf, n3: %lf\n", x, normal[0], normal[1], normal[2]);
        printf("normalized x: %d, n1: %lf, n2: %lf, n3: %lf\n", x, norm[0], norm[1], norm[2]);
    }
    */

    Matrix3f R = rodriguesRotation(coef[0], coef[1], coef[2], coef[3], coef[4]);
    norm = R * norm;

    Vector3f base(0.f, 0.f, 1.f);

    float dump = 0.0f;
    norm = norm + dump * base;
    if (norm[2] < 0.0f) {
        norm[0] = -norm[0];
        norm[1] = -norm[1];
        norm[2] = -norm[2];
    }

    norm.normalize();

    Vector3f norm_diff = (R * base).cross(norm);
    Vector3f rotAxis = base.cross(norm); // atan2(rotAxis[0], rotAxis[1])  -> phi_xy // mazui?
    // printf("loss: %lf", norm_diff.norm2());
    float cos = base * norm;
    float sin = rotAxis.norm2();
    float diff = norm_diff.norm2();

    // printf("%lf, ", diff);
    norm_loss[x + sizeV[0] * y + sizeV[0] * sizeV[1] * z] = diff;
    /*
    if (out)
    printf("x: %d, cos: %lf, sin: %lf\n", x, cos, sin);
    */
    /*
    if (isnan(theta))
        printf("norm: (%lf), cos(theta): (%lf)\n", rotAxis.norm2(), base * norm);
    */

    devCoef[coord[0] + sizeV[0] * coord[1] + sizeV[0] * sizeV[1] * coord[2] +
            0 * (sizeV[0] * sizeV[1] * sizeV[2])] = atan2(rotAxis[1], rotAxis[0]);
    devCoef[coord[0] + sizeV[0] * coord[1] + sizeV[0] * sizeV[1] * coord[2] +
            1 * (sizeV[0] * sizeV[1] * sizeV[2])] = cos;
}

__global__ void
calcNormalVectorThreeDirecWithEst(float *devVoxel, float *devCoef, int y, const Geometry *geom,
                                  float *norm_loss, const float *devEstimate) {
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int z = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= geom->voxel || z >= geom->voxel) return;

    int coord[3] = {x, y, z};
    int sizeV[3] = {geom->voxel, geom->voxel, geom->voxel};
    int sizeD[3] = {geom->detect, geom->detect, geom->nProj};

    const float phi_c = devCoef[coord[0] + sizeV[0] * coord[1] + sizeV[0] * sizeV[1] * coord[2] +
                                0 * (sizeV[0] * sizeV[1] * sizeV[2])];
    const float cos_c = devCoef[coord[0] + sizeV[0] * coord[1] + sizeV[0] * sizeV[1] * coord[2] +
                                1 * (sizeV[0] * sizeV[1] * sizeV[2])];

    const float coef[5] = {cos(phi_c), sin(phi_c), 0, cos_c, sqrt(1.0f - cos_c * cos_c)};

    const float mu[3] =
            {devVoxel[coord[0] + sizeV[0] * coord[1] + sizeV[0] * sizeV[1] * coord[2] +
                      0 * (sizeV[0] * sizeV[1] * sizeV[2])],
             devVoxel[coord[0] + sizeV[0] * coord[1] + sizeV[0] * sizeV[1] * coord[2] +
                      1 * (sizeV[0] * sizeV[1] * sizeV[2])],
             devVoxel[coord[0] + sizeV[0] * coord[1] + sizeV[0] * sizeV[1] * coord[2] +
                      2 * (sizeV[0] * sizeV[1] * sizeV[2])]};

    // float rand_rotate = hiprand_uniform(&curandStates[z * sizeV[0] + x]);
    float rand_rotate = devEstimate[coord[0] + sizeV[0] * coord[1] + sizeV[0] * sizeV[1] * coord[2] +
                                    1 * (sizeV[0] * sizeV[1] * sizeV[2])];
    // float rand_rotate = judge;
    // printf("rand: %lf\n", judge);

    float mu1 = mu[1], mu2 = mu[2];

    if (rand_rotate < 0.5f) {
        mu1 = mu[1];
        mu2 = mu[2];
    } else if (rand_rotate < 1.50f) {
        mu1 = -mu[1];
        mu2 = mu[2];
    } else if (rand_rotate < 2.50f) {
        mu1 = -mu[1];
        mu2 = -mu[2];
    } else {
        mu1 = mu[1];
        mu2 = -mu[2];
    }

    Vector3f vec1(mu[0] * basisVector[3 * 0 + 0] - mu1 * basisVector[3 * 1 + 0],
                  mu[0] * basisVector[3 * 0 + 1] - mu1 * basisVector[3 * 1 + 1],
                  mu[0] * basisVector[3 * 0 + 2] - mu1 * basisVector[3 * 1 + 2]);
    Vector3f vec2(mu[0] * basisVector[3 * 0 + 0] - mu2 * basisVector[3 * 2 + 0],
                  mu[0] * basisVector[3 * 0 + 1] - mu2 * basisVector[3 * 2 + 1],
                  mu[0] * basisVector[3 * 0 + 2] - mu2 * basisVector[3 * 2 + 2]);

    Vector3f norm = vec1.cross(vec2);
    norm.normalize();
    // Vector3f normal = (1.0f / (mu[0] + eps)) * S1 + (1.0f / (mu[1] + eps)) * S2 + (1.0f / (mu[2] + eps)) * S3;
    /*
    bool out = (y == 50 && z == 50);
    if (out) {
        printf("x: %d, n1: %lf, n2: %lf, n3: %lf\n", x, normal[0], normal[1], normal[2]);
        printf("normalized x: %d, n1: %lf, n2: %lf, n3: %lf\n", x, norm[0], norm[1], norm[2]);
    }
    */

    Matrix3f R = rodriguesRotation(coef[0], coef[1], coef[2], coef[3], coef[4]);
    norm = R * norm;

    Vector3f base(0.f, 0.f, 1.f);

    float dump = 0.0f;
    norm = norm + dump * base;
    if (norm[2] < 0.0f) {
        norm[0] = -norm[0];
        norm[1] = -norm[1];
        norm[2] = -norm[2];
    }

    norm.normalize();

    Vector3f norm_diff = (R * base).cross(norm);
    Vector3f rotAxis = base.cross(norm); // atan2(rotAxis[0], rotAxis[1])  -> phi_xy // mazui?
    // printf("loss: %lf", norm_diff.norm2());
    float cos = base * norm;
    float sin = rotAxis.norm2();
    float diff = norm_diff.norm2();

    // printf("%lf, ", diff);
    norm_loss[x + sizeV[0] * y + sizeV[0] * sizeV[1] * z] = diff;
    /*
    if (out)
    printf("x: %d, cos: %lf, sin: %lf\n", x, cos, sin);
    */
    /*
    if (isnan(theta))
        printf("norm: (%lf), cos(theta): (%lf)\n", rotAxis.norm2(), base * norm);
    */

    devCoef[coord[0] + sizeV[0] * coord[1] + sizeV[0] * sizeV[1] * coord[2] +
            0 * (sizeV[0] * sizeV[1] * sizeV[2])] = atan2(rotAxis[1], rotAxis[0]);
    devCoef[coord[0] + sizeV[0] * coord[1] + sizeV[0] * sizeV[1] * coord[2] +
            1 * (sizeV[0] * sizeV[1] * sizeV[2])] = cos;
}

__global__ void
calcNormalVectorThreeDirecSaveEst(float *devVoxel, float *devCoef, int y, const Geometry *geom, float *norm_loss,
                                  float *devEstimate, int iter) {
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int z = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= geom->voxel || z >= geom->voxel) return;

    int coord[3] = {x, y, z};
    int sizeV[3] = {geom->voxel, geom->voxel, geom->voxel};
    int sizeD[3] = {geom->detect, geom->detect, geom->nProj};

    const float phi_c = devCoef[coord[0] + sizeV[0] * coord[1] + sizeV[0] * sizeV[1] * coord[2] +
                                0 * (sizeV[0] * sizeV[1] * sizeV[2])];
    const float cos_c = devCoef[coord[0] + sizeV[0] * coord[1] + sizeV[0] * sizeV[1] * coord[2] +
                                1 * (sizeV[0] * sizeV[1] * sizeV[2])];

    const float coef[5] = {cos(phi_c), sin(phi_c), 0, cos_c, sqrt(1.0f - cos_c * cos_c)};

    const float mu[3] =
            {devVoxel[coord[0] + sizeV[0] * coord[1] + sizeV[0] * sizeV[1] * coord[2] +
                      0 * (sizeV[0] * sizeV[1] * sizeV[2])],
             devVoxel[coord[0] + sizeV[0] * coord[1] + sizeV[0] * sizeV[1] * coord[2] +
                      1 * (sizeV[0] * sizeV[1] * sizeV[2])],
             devVoxel[coord[0] + sizeV[0] * coord[1] + sizeV[0] * sizeV[1] * coord[2] +
                      2 * (sizeV[0] * sizeV[1] * sizeV[2])]};

    float mu1 = mu[1], mu2 = mu[2];
    Vector3f vec1(mu[0] * basisVector[3 * 0 + 0] - mu1 * basisVector[3 * 1 + 0],
                  mu[0] * basisVector[3 * 0 + 1] - mu1 * basisVector[3 * 1 + 1],
                  mu[0] * basisVector[3 * 0 + 2] - mu1 * basisVector[3 * 1 + 2]);
    Vector3f vec2(mu[0] * basisVector[3 * 0 + 0] - mu2 * basisVector[3 * 2 + 0],
                  mu[0] * basisVector[3 * 0 + 1] - mu2 * basisVector[3 * 2 + 1],
                  mu[0] * basisVector[3 * 0 + 2] - mu2 * basisVector[3 * 2 + 2]);

    Vector3f norm = vec1.cross(vec2);
    norm.normalize();

    Vector3f base(0.f, 0.f, 1.f);
    float dump = 0.0f;
    norm = norm + dump * base;
    if (norm[2] < 0.0f) {
        norm[0] = -norm[0];
        norm[1] = -norm[1];
        norm[2] = -norm[2];
    }
    norm.normalize();

    float cos = base * norm;
    float est = devEstimate[coord[0] + sizeV[0] * coord[1] + sizeV[0] * sizeV[1] * coord[2] +
                            0 * (sizeV[0] * sizeV[1] * sizeV[2])];
    if (cos > est) {
        devEstimate[coord[0] + sizeV[0] * coord[1] + sizeV[0] * sizeV[1] * coord[2] +
                    0 * (sizeV[0] * sizeV[1] * sizeV[2])] = cos;
        devEstimate[coord[0] + sizeV[0] * coord[1] + sizeV[0] * sizeV[1] * coord[2] +
                    1 * (sizeV[0] * sizeV[1] * sizeV[2])] = (float) iter;
    }
}


__global__ void
calcNormalVector(const float *devVoxel, float *coefficient, int y, int it, const Geometry *geom, float *norm_loss) {
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int z = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= geom->voxel || z >= geom->voxel) return;

    int coord[3] = {x, y, z};
    int sizeV[3] = {geom->voxel, geom->voxel, geom->voxel};
    int sizeD[3] = {geom->detect, geom->detect, geom->nProj};

    const float phi_c = coefficient[coord[0] + sizeV[0] * coord[1] + sizeV[0] * sizeV[1] * coord[2] +
                                    0 * (sizeV[0] * sizeV[1] * sizeV[2])];
    const float cos_c = coefficient[coord[0] + sizeV[0] * coord[1] + sizeV[0] * sizeV[1] * coord[2] +
                                    1 * (sizeV[0] * sizeV[1] * sizeV[2])];

    const float coef[5] = {cos(phi_c), sin(phi_c), 0, cos_c, sqrt(1.0f - cos_c * cos_c)};

    const float mu[4] =
            {devVoxel[coord[0] + sizeV[0] * coord[1] + sizeV[0] * sizeV[1] * coord[2] +
                      0 * (sizeV[0] * sizeV[1] * sizeV[2])],
             devVoxel[coord[0] + sizeV[0] * coord[1] + sizeV[0] * sizeV[1] * coord[2] +
                      1 * (sizeV[0] * sizeV[1] * sizeV[2])],
             devVoxel[coord[0] + sizeV[0] * coord[1] + sizeV[0] * sizeV[1] * coord[2] +
                      2 * (sizeV[0] * sizeV[1] * sizeV[2])],
             devVoxel[coord[0] + sizeV[0] * coord[1] + sizeV[0] * sizeV[1] * coord[2] +
                      3 * (sizeV[0] * sizeV[1] * sizeV[2])]};

    /*
    bool big_than_eps = true;
    for (int i = 0; i < NUM_BASIS_VECTOR; i++) {
        const float eps = 1e-6;
        if (mu[i] < eps) {
            norm = 1.0f * Vector3f(basisVector[3 * i + 0], basisVector[3 * i + 1], basisVector[3 * i + 2]);
            big_than_eps = false;
            break;
        }
    }
    if (big_than_eps)
        norm = (1 / mu[0]) * Vector3f(basisVector[3 * 0 + 0], basisVector[3 * 0 + 1], basisVector[3 * 0 + 2]) +
               (1 / mu[1]) * Vector3f(basisVector[3 * 1 + 0], basisVector[3 * 1 + 1], basisVector[3 * 1 + 2]) +
               (1 / mu[2]) * Vector3f(basisVector[3 * 2 + 0], basisVector[3 * 2 + 1], basisVector[3 * 2 + 2]) +
               (1 / mu[3]) * Vector3f(basisVector[3 * 3 + 0], basisVector[3 * 3 + 1], basisVector[3 * 3 + 2]);
    */
    float eps = 1e-2;

    Vector3f S1(basisVector[3 * 0 + 0], basisVector[3 * 0 + 1], basisVector[3 * 0 + 2]);
    Vector3f S2(basisVector[3 * 1 + 0], basisVector[3 * 1 + 1], basisVector[3 * 1 + 2]);
    Vector3f S3(basisVector[3 * 2 + 0], basisVector[3 * 2 + 1], basisVector[3 * 2 + 2]);
    Vector3f S4(basisVector[3 * 3 + 0], basisVector[3 * 3 + 1], basisVector[3 * 3 + 2]);

    /*
   Vector3f S1(1.0, 0.0, 0.0);
   Vector3f S2(0.0, 1.0, 0.0);
   Vector3f S3(0.0, 0.0, 1.0);
     */

    Vector3f norm = (1.0f / (mu[0] + eps)) * S1 + (1.0f / (mu[1] + eps)) * S2 + (1.0f / (mu[2] + eps)) * S3
                    + (1.0f / (mu[3] + eps)) * S4;
    // (1.0f / (mu[3] + eps)) * Vector3f(basisVector[3 * 3 + 0], basisVector[3 * 3 + 1], basisVector[3 * 3 + 2]);
    // Vector3f norm = (mu[0]) * S1 + (mu[1]) * S2 + (mu[2]) * S3;

    norm.normalize();
    // Vector3f normal = (1.0f / (mu[0] + eps)) * S1 + (1.0f / (mu[1] + eps)) * S2 + (1.0f / (mu[2] + eps)) * S3;
    /*
    bool out = (y == 50 && z == 50);
    if (out) {
        printf("x: %d, n1: %lf, n2: %lf, n3: %lf\n", x, normal[0], normal[1], normal[2]);
        printf("normalized x: %d, n1: %lf, n2: %lf, n3: %lf\n", x, norm[0], norm[1], norm[2]);
    }*/

    Matrix3f R = rodriguesRotation(coef[0], coef[1], coef[2], coef[3], coef[4]);
    norm = R * norm;

    Vector3f base(basisVector[0], basisVector[1], basisVector[2]);

    // printf("loss: %lf", norm_diff.norm2());

    float dump = 0.0f;
    norm = norm + dump * base;
    norm.normalize();
    if (norm[2] < 0.0f) {
        norm[0] = -norm[0];
        norm[1] = -norm[1];
        norm[2] = -norm[2];
    }

    float cos = base * norm;
    Vector3f norm_diff = (R * base).cross(norm);
    Vector3f rotAxis = base.cross(norm); // atan2(rotAxis[0], rotAxis[1])  -> phi_xy // mazui?
    float sin = rotAxis.norm2();
    float diff = norm_diff.norm2();

    // printf("%lf, ", diff);
    norm_loss[x + sizeV[0] * y + sizeV[0] * sizeV[1] * z] = diff;
    /*
    if (out)
    printf("x: %d, cos: %lf, sin: %lf\n", x, cos, sin);
    */
    /*
    if (isnan(theta))
        printf("norm: (%lf), cos(theta): (%lf)\n", rotAxis.norm2(), base * norm);
    */

    coefficient[coord[0] + sizeV[0] * coord[1] + sizeV[0] * sizeV[1] * coord[2] +
                0 * (sizeV[0] * sizeV[1] * sizeV[2])] = atan2(rotAxis[1], rotAxis[0]);
    coefficient[coord[0] + sizeV[0] * coord[1] + sizeV[0] * sizeV[1] * coord[2] +
                1 * (sizeV[0] * sizeV[1] * sizeV[2])] = cos;
}

__global__ void
meanFiltFiber(const float *devCoefSrc, float *devCoefDst, const float *devVoxel,
              const Geometry *geom, int y, float coef) {
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int z = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= geom->voxel - 1 || x < 1 || z >= geom->voxel - 1 || z < 1) return;

    int coord[3] = {x, y, z};
    int sizeV[3] = {geom->voxel, geom->voxel, geom->voxel};
    int sizeD[3] = {geom->detect, geom->detect, geom->nProj};
    Vector3f norm[27];
    int cnt = 0;
    for (int i = -1; i < 2; i++) {
        for (int j = -1; j < 2; j++) {
            for (int k = -1; k < 2; k++) {
                float cos_theta = devCoefSrc[coord[0] - i + sizeV[0] * (coord[1] - j) +
                                             sizeV[0] * sizeV[1] * (coord[2] - k) +
                                             1 * (sizeV[0] * sizeV[1] * sizeV[2])];
                float sin_theta = sqrt(1.f - cos_theta * cos_theta);
                float phi = -M_PI / 2.0f + devCoefSrc[coord[0] - i + sizeV[0] * (coord[1] - j) +
                                                      sizeV[0] * sizeV[1] * (coord[2] - k) +
                                                      0 * (sizeV[0] * sizeV[1] * sizeV[2])];
                float mu = /*devVoxel[coord[0] - i + sizeV[0] * (coord[1] - j) +
                                    sizeV[0] * sizeV[1] * (coord[2] - k) +
                                    0 * (sizeV[0] * sizeV[1] * sizeV[2])] +*/
                        devVoxel[coord[0] - i + sizeV[0] * (coord[1] - j) +
                                 sizeV[0] * sizeV[1] * (coord[2] - k) +
                                 1 * (sizeV[0] * sizeV[1] * sizeV[2])] +
                        devVoxel[coord[0] - i + sizeV[0] * (coord[1] - j) +
                                 sizeV[0] * sizeV[1] * (coord[2] - k) +
                                 2 * (sizeV[0] * sizeV[1] * sizeV[2])];
                // norm[cnt] = mu * Vector3f(sin_theta * cos(phi), sin_theta * sin(phi), cos_theta);
                norm[cnt] = mu * Vector3f(sin_theta * cos(phi), sin_theta * sin(phi), cos_theta);
                cnt++;
            }
        }
    }

    Vector3f norm_cent = norm[13];
    for (int i = 0; i < 13; i++) {
        if (norm[i] * norm[13] > 0.707106f) {
            norm_cent = norm_cent + coef * norm[i];
        } else if (norm[i] * norm[13] < -0.707106f) {
            norm_cent = norm_cent - coef * norm[i];
        }
        if (norm[26 - i] * norm[13] > 0.707106f) {
            norm_cent = norm_cent + coef * norm[26 - i];
        } else if (norm[26 - i] * norm[13] < -0.707106f) {
            norm_cent = norm_cent - coef * norm[26 - i];
        }
    }

    norm_cent.normalize(1e-8);
    if (norm_cent[2] < 0.0f) {
        norm_cent[0] = -norm_cent[0];
        norm_cent[1] = -norm_cent[1];
        norm_cent[2] = -norm_cent[2];
    }

    Vector3f base(basisVector[0], basisVector[1], basisVector[2]);
    Vector3f rotAxis = base.cross(norm_cent);
    float cos = base * norm_cent;
    float sin = rotAxis.norm2();

    devCoefDst[coord[0] + sizeV[0] * coord[1] + sizeV[0] * sizeV[1] * coord[2] +
               0 * (sizeV[0] * sizeV[1] * sizeV[2])] = atan2(rotAxis[1], rotAxis[0]);
    devCoefDst[coord[0] + sizeV[0] * coord[1] + sizeV[0] * sizeV[1] * coord[2] +
               1 * (sizeV[0] * sizeV[1] * sizeV[2])] = cos;
}

__global__ void
calcRotation(const float *md, float *coefficient, int y, const Geometry *geom, float *norm_loss) {
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int z = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= geom->voxel || z >= geom->voxel) return;

    int coord[3] = {x, y, z};
    int sizeV[3] = {geom->voxel, geom->voxel, geom->voxel};
    int sizeD[3] = {geom->detect, geom->detect, geom->nProj};

    const float coef[5] = {
            coefficient[coord[0] + sizeV[0] * coord[1] + sizeV[0] * sizeV[1] * coord[2] +
                        0 * (sizeV[0] * sizeV[1] * sizeV[2])], // ax_x
            coefficient[coord[0] + sizeV[0] * coord[1] + sizeV[0] * sizeV[1] * coord[2] +
                        1 * (sizeV[0] * sizeV[1] * sizeV[2])], // ax_y
            coefficient[coord[0] + sizeV[0] * coord[1] + sizeV[0] * sizeV[1] * coord[2] +
                        2 * (sizeV[0] * sizeV[1] * sizeV[2])], // ax_z
            coefficient[coord[0] + sizeV[0] * coord[1] + sizeV[0] * sizeV[1] * coord[2] +
                        3 * (sizeV[0] * sizeV[1] * sizeV[2])], // theta
            coefficient[coord[0] + sizeV[0] * coord[1] + sizeV[0] * sizeV[1] * coord[2] +
                        4 * (sizeV[0] * sizeV[1] * sizeV[2])]
    };

    Vector3f norm(md[coord[0] + sizeV[0] * coord[1] + sizeV[0] * sizeV[1] * coord[2] +
                     0 * (sizeV[0] * sizeV[1] * sizeV[2])],
                  md[coord[0] + sizeV[0] * coord[1] + sizeV[0] * sizeV[1] * coord[2] +
                     1 * (sizeV[0] * sizeV[1] * sizeV[2])],
                  md[coord[0] + sizeV[0] * coord[1] + sizeV[0] * sizeV[1] * coord[2] +
                     2 * (sizeV[0] * sizeV[1] * sizeV[2])]);

    norm.normalize();

    Matrix3f R = rodriguesRotation(coef[0], coef[1], coef[2], coef[3], coef[4]);
    norm = R * norm;

    Vector3f base(basisVector[0], basisVector[1], basisVector[2]);
    Vector3f norm_diff = base.cross(norm);
    Vector3f rotAxis = base.cross(norm); // atan2(rotAxis[0], rotAxis[1])  -> phi_xy
    // printf("loss: %lf", norm_diff.norm2());
    float cos = base * norm;
    float sin = rotAxis.norm2();
    float diff = norm_diff.norm2();

    // printf("%lf, ", diff);
    norm_loss[x + sizeV[0] * y + sizeV[0] * sizeV[1] * z] = diff;

    if (cos > 1.0f) {
        cos = 1.0f;
        sin = 0.0f;
    } else if (cos < -1.0f) {
        cos = -1.0f;
        sin = 0.0f;
    }
    if (sin > 1.0f) {
        sin = 1.0f;
        cos = 0.0f;
    } else if (sin < -1.0f) {
        sin = -1.0f;
        cos = 0.0f;
    }

    coefficient[coord[0] + sizeV[0] * coord[1] + sizeV[0] * sizeV[1] * coord[2] +
                0 * (sizeV[0] * sizeV[1] * sizeV[2])] = rotAxis[0];
    coefficient[coord[0] + sizeV[0] * coord[1] + sizeV[0] * sizeV[1] * coord[2] +
                1 * (sizeV[0] * sizeV[1] * sizeV[2])] = rotAxis[1];
    coefficient[coord[0] + sizeV[0] * coord[1] + sizeV[0] * sizeV[1] * coord[2] +
                2 * (sizeV[0] * sizeV[1] * sizeV[2])] = rotAxis[2];
    coefficient[coord[0] + sizeV[0] * coord[1] + sizeV[0] * sizeV[1] * coord[2] +
                3 * (sizeV[0] * sizeV[1] * sizeV[2])] = cos;
    coefficient[coord[0] + sizeV[0] * coord[1] + sizeV[0] * sizeV[1] * coord[2] +
                4 * (sizeV[0] * sizeV[1] * sizeV[2])] = sin;
}

void convertNormVector(const Volume<float> *voxel, Volume<float> *md, const Volume<float> *coefficient) {
    for (int x = 0; x < NUM_VOXEL; x++) {
        for (int y = 0; y < NUM_VOXEL; y++) {
            for (int z = 0; z < NUM_VOXEL; z++) {
                float mu = (voxel[0](x, y, z) + voxel[1](x, y, z) + voxel[2](x, y, z)) / 3.0f;
                // printf("phi: %lf, theta: %lf\n", angle[0](x, y, z), angle[1](x, y, z));

                const float v[3] =
                        {voxel[0](x, y, z), voxel[1](x, y, z), voxel[2](x, y, z)};

                const float phi_c = coefficient[0](x, y, z);
                const float cos_c = coefficient[1](x, y, z);
                const float coef[5] = {std::cos(phi_c), std::sin(phi_c), 0.0f, cos_c, std::sqrt(1.0f - cos_c * cos_c)};
                Matrix3f R = rodriguesRotation(coef[0], coef[1], coef[2], coef[3], coef[4]);

                Vector3f norm = R * Vector3f(basisVector[0], basisVector[1], basisVector[2]);

                /*
                printf("R:\n[%lf, %lf, %lf]\n[%lf, %lf, %lf],\n[%lf, %lf, %lf]\n",
                       R[0], R[1], R[2], R[3], R[4], R[5], R[6], R[7], R[8]);
                printf("base: [%lf, %lf, %lf]\n", base[0], base[1], base[2]);
                */
                float sign = (norm[2] >= 0) ? 1.0 : -1.0;

                for (int i = 0; i < 3; i++) {
                    md[i](x, y, z) = mu * norm[i];
                }
            }
        }
    }
}

__global__ void
forwardProj(float *devProj, float *devVoxel, Geometry *geom, int cond, int y, int n) {
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int z = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= geom->voxel || z >= geom->voxel) return;

    const int coord[4] = {x, y, z, n};
    forwardonDevice(coord, devProj, devVoxel, *geom, cond);
}

__global__ void
backwardProj(float *devProj, float *devVoxelTmp, float *devVoxelFactor, Geometry *geom, int cond,
             int y, int n) {
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int z = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= geom->voxel || z >= geom->voxel) return;

    const int coord[4] = {x, y, z, n};
    backwardonDevice(coord, devProj, devVoxelTmp, devVoxelFactor, *geom, cond);
}

__global__ void projRatio(float *devProj, const float *devSino, const Geometry *geom, int n, float *loss) {
    const int u = blockIdx.x * blockDim.x + threadIdx.x;
    const int v = blockIdx.y * blockDim.y + threadIdx.y;
    if (u >= geom->detect || v >= geom->detect) return;

    float threshold = 2.0f;
    const int idx = u + geom->detect * v + geom->detect * geom->detect * abs(n);
    atomicAdd(loss, abs(devSino[idx] - devProj[idx]));
    // printf("%lf\n", *loss);
    // const float div = devSino[idx] / devProj[idx];
    if (devProj[idx] != 0.0f) {
        // devProj[idx] = devSino[idx] / (devProj[idx] + 0.1f * (1.0f - exp(-abs(1.0f - devSino[idx] / devProj[idx]))));
        devProj[idx] = devSino[idx] / devProj[idx];
    }

    if (devProj[idx] > threshold) {
        devProj[idx] = threshold;
    }
}

__global__ void
voxelProduct(float *devVoxel, const float *devVoxelTmp, const float *devVoxelFactor, const Geometry *geom, int y) {
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int z = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= geom->voxel || z >= geom->voxel) return;

    for (int i = 0; i < NUM_BASIS_VECTOR; i++) {
        const int idxVoxel =
                x + geom->voxel * y + geom->voxel * geom->voxel * z + (geom->voxel * geom->voxel * geom->voxel) * i;
        const int idxOnPlane = x + geom->voxel * z + geom->voxel * geom->voxel * i;

        devVoxel[idxVoxel] = (devVoxelFactor[idxOnPlane] == 0.0f) ? 0.0f :
                             devVoxel[idxVoxel] * devVoxelTmp[idxOnPlane] / devVoxelFactor[idxOnPlane];

        if (isnan(devVoxel[idxVoxel])) {
            printf("voxel: %lf, tmp: %lf, fact: %lf\n", devVoxel[idxVoxel], devVoxelTmp[idxOnPlane],
                   devVoxelFactor[idxOnPlane]);
        }
    }
}

__global__ void projSubtract(float *devProj, const float *devSino, const Geometry *geom, int n, float *loss) {
    const int u = blockIdx.x * blockDim.x + threadIdx.x;
    const int v = blockIdx.y * blockDim.y + threadIdx.y;
    if (u >= geom->detect || v >= geom->detect) return;

    const int idx = u + geom->detect * v + geom->detect * geom->detect * abs(n);
    atomicAdd(loss, abs(devSino[idx] - devProj[idx]));
    // const float div = devSino[idx] / devProj[idx];
    devProj[idx] = devSino[idx] - devProj[idx];
    // a = b / c;
}

__global__ void
projCompare(float *devCompare, const float *devSino, const float *devProj, const Geometry *geom, int n) {
    const int u = blockIdx.x * blockDim.x + threadIdx.x;
    const int v = blockIdx.y * blockDim.y + threadIdx.y;
    if (u >= geom->detect || v >= geom->detect) return;

    const int idx = u + geom->detect * v + geom->detect * geom->detect * abs(n);
    // const float div = devSino[idx] / devProj[idx];
    // devCompare[idx] = devSino[idx] - devProj[idx];
    devCompare[idx] = devSino[idx] / devProj[idx];
    // a = b / c;
}


__global__ void
voxelPlus(float *devVoxel, const float *devVoxelTmp, float alpha, const Geometry *geom, int y) {
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int z = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= geom->voxel || z >= geom->voxel) return;

    for (int i = 0; i < NUM_BASIS_VECTOR; i++) {
        const int idxVoxel =
                x + geom->voxel * y + geom->voxel * geom->voxel * z + (geom->voxel * geom->voxel * geom->voxel) * i;
        const int idxOnPlane = x + geom->voxel * z + geom->voxel * geom->voxel * i;
        devVoxel[idxVoxel] = devVoxel[idxVoxel] + alpha * devVoxelTmp[idxOnPlane];
    }
}

__global__ void voxelSqrtFromSrc(float *hostVoxel, const float *devVoxel, const Geometry *geom, int y) {
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int z = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= geom->voxel || z >= geom->voxel) return;

    for (int i = 0; i < NUM_BASIS_VECTOR; i++) {
        const int idxVoxel =
                x + geom->voxel * y + geom->voxel * geom->voxel * z + (geom->voxel * geom->voxel * geom->voxel) * i;
        hostVoxel[idxVoxel] = sqrt(abs(devVoxel[idxVoxel]));
    }
}

__global__ void voxelSqrt(float *devVoxel, const Geometry *geom, int y) {
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int z = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= geom->voxel || z >= geom->voxel) return;

    for (int i = 0; i < NUM_BASIS_VECTOR; i++) {
        const int idxVoxel =
                x + geom->voxel * y + geom->voxel * geom->voxel * z + (geom->voxel * geom->voxel * geom->voxel) * i;

        devVoxel[idxVoxel] = (devVoxel[idxVoxel] < 0.0f) ? sqrt(-devVoxel[idxVoxel]) : sqrt(devVoxel[idxVoxel]);
    }
}

__device__ void
forwardonDevice(const int coord[4], float *devProj, const float *devVoxel,
                const Geometry &geom, int cond) {

    int sizeV[3] = {geom.voxel, geom.voxel, geom.voxel};
    int sizeD[3] = {geom.detect, geom.detect, geom.nProj};

    float u, v;
    Vector3f B, G;
    rayCasting(u, v, B, G, cond, coord, geom);

    if (!(0.55f < u && u < (float) sizeD[0] - 0.55f && 0.55f < v && v < (float) sizeD[1] - 0.55f))
        return;

    float u_tmp = u - 0.5f, v_tmp = v - 0.5f;
    int intU = floor(u_tmp), intV = floor(v_tmp);
    float c1 = (1.0f - (u_tmp - (float) intU)) * (v_tmp - (float) intV),
            c2 = (u_tmp - (float) intU) * (v_tmp - (float) intV),
            c3 = (u_tmp - (float) intU) * (1.0f - (v_tmp - (float) intV)),
            c4 = (1.0f - (u_tmp - (float) intU)) * (1.0f - (v_tmp - (float) intV));

    const int n = abs(coord[3]);

    const float ratio = (geom.voxSize * geom.voxSize) /
                        (geom.detSize * (geom.sod / geom.sdd) * geom.detSize * (geom.sod / geom.sdd));
    const int idxVoxel =
            coord[0] + sizeV[0] * coord[1] + sizeV[0] * sizeV[1] * coord[2] +
            cond * (sizeV[0] * sizeV[1] * sizeV[2]);
    atomicAdd(&devProj[intU + sizeD[0] * (intV + 1) + sizeD[0] * sizeD[1] * n],
              c1 * geom.voxSize * ratio * devVoxel[idxVoxel]);
    atomicAdd(&devProj[(intU + 1) + sizeD[0] * (intV + 1) + sizeD[0] * sizeD[1] * n],
              c2 * geom.voxSize * ratio * devVoxel[idxVoxel]);
    atomicAdd(&devProj[(intU + 1) + sizeD[0] * intV + sizeD[0] * sizeD[1] * n],
              c3 * geom.voxSize * ratio * devVoxel[idxVoxel]);
    atomicAdd(&devProj[intU + sizeD[0] * intV + sizeD[0] * sizeD[1] * n],
              c4 * geom.voxSize * ratio * devVoxel[idxVoxel]);
}

__device__ void
backwardonDevice(const int coord[4], const float *devProj, float *devVoxelTmp, float *devVoxelFactor,
                 const Geometry &geom, int cond) {

    int sizeV[3] = {geom.voxel, geom.voxel, geom.voxel};
    int sizeD[3] = {geom.detect, geom.detect, geom.nProj};

    float u, v;
    Vector3f B, G;
    rayCasting(u, v, B, G, cond, coord, geom);

    if (!(0.55f < u && u < (float) sizeD[0] - 0.55f && 0.55f < v && v < (float) sizeD[1] - 0.55f))
        return;

    const int n = abs(coord[3]);

    float u_tmp = u - 0.5f, v_tmp = v - 0.5f;
    int intU = floor(u_tmp), intV = floor(v_tmp);
    float c1 = (1.0f - (u_tmp - (float) intU)) * (v_tmp - (float) intV), c2 =
            (u_tmp - (float) intU) * (v_tmp - (float) intV),
            c3 = (u_tmp - (float) intU) * (1.0f - (v_tmp - (float) intV)), c4 =
            (1.0f - (u_tmp - (float) intU)) * (1.0f - (v_tmp - (float) intV));

    const int idxVoxel = coord[0] + sizeV[0] * coord[2] + cond * (sizeV[0] * sizeV[1]);
    const float numBack = c1 * devProj[intU + sizeD[0] * (intV + 1) + sizeD[0] * sizeD[1] * n] +
                          c2 * devProj[(intU + 1) + sizeD[0] * (intV + 1) + sizeD[0] * sizeD[1] * n] +
                          c3 * devProj[(intU + 1) + sizeD[0] * intV + sizeD[0] * sizeD[1] * n] +
                          c4 * devProj[intU + sizeD[0] * intV + sizeD[0] * sizeD[1] * n];

    devVoxelFactor[idxVoxel] += 1.0f;
    devVoxelTmp[idxVoxel] += numBack;
}

__device__ void
forwardXTTonDevice(const int coord[4], float *devProj, const float *devVoxel,
                   const Geometry &geom, int cond) {

    int sizeV[3] = {geom.voxel, geom.voxel, geom.voxel};
    int sizeD[3] = {geom.detect, geom.detect, geom.nProj};

    float u = 0.0f, v = 0.0f;
    Vector3f B(0.0f, 0.0f, 0.0f), G(0.0f, 0.0f, 0.0f);
    rayCasting(u, v, B, G, cond, coord, geom);

    if (!(0.55f < u && u < (float) sizeD[0] - 0.55f && 0.55f < v && v < (float) sizeD[1] - 0.55f))
        return;

    const int n = abs(coord[3]);

    float u_tmp = u - 0.5f, v_tmp = v - 0.5f;
    int intU = floor(u_tmp), intV = floor(v_tmp);
    float c1 = (1.0f - (u_tmp - (float) intU)) * (v_tmp - (float) intV),
            c2 = (u_tmp - (float) intU) * (v_tmp - (float) intV),
            c3 = (u_tmp - (float) intU) * (1.0f - (v_tmp - (float) intV)),
            c4 = (1.0f - (u_tmp - (float) intU)) * (1.0f - (v_tmp - (float) intV));

    const float ratio = (geom.voxSize * geom.voxSize) /
                        (geom.detSize * (geom.sod / geom.sdd) * geom.detSize * (geom.sod / geom.sdd));

    float proj = 0.0f;
    for (int i = 0; i < NUM_BASIS_VECTOR; i++) {
        // add scattering coefficient (read paper)
        // B->beam direction unit vector (src2voxel)
        // S->scattering base vector
        // G->grating sensivity vector
        Vector3f S(basisVector[3 * i + 0], basisVector[3 * i + 1], basisVector[3 * i + 2]);
        float vkm = B.cross(S).norm2() * abs(S * G);
        // float vkm = abs(S * G);
        const int idxVoxel =
                coord[0] + sizeV[0] * coord[1] + sizeV[0] * sizeV[1] * coord[2] +
                i * (sizeV[0] * sizeV[1] * sizeV[2]);
        proj += vkm * vkm * geom.voxSize * ratio * devVoxel[idxVoxel];
        // printf("%d: %lf\n", i+1, vkm);
        // printf("sinogram: %lf\n", devSino[intU + sizeD[0] * intV + sizeD[0] * sizeD[1] * n]);
    }
    atomicAdd(&devProj[intU + sizeD[0] * (intV + 1) + sizeD[0] * sizeD[1] * n], c1 * proj);
    atomicAdd(&devProj[(intU + 1) + sizeD[0] * (intV + 1) + sizeD[0] * sizeD[1] * n], c2 * proj);
    atomicAdd(&devProj[(intU + 1) + sizeD[0] * intV + sizeD[0] * sizeD[1] * n], c3 * proj);
    atomicAdd(&devProj[intU + sizeD[0] * intV + sizeD[0] * sizeD[1] * n], c4 * proj);

}

// change to class
__device__ void
backwardXTTonDevice(const int coord[4], const float *devProj, float *devVoxelTmp, float *devVoxelFactor,
                    const Geometry &geom, int cond) {

    int sizeV[3] = {geom.voxel, geom.voxel, geom.voxel};
    int sizeD[3] = {geom.detect, geom.detect, geom.nProj};

    float u = 0.0f, v = 0.0f;
    Vector3f B(0.0f, 0.0f, 0.0f), G(0.0f, 0.0f, 0.0f);
    rayCasting(u, v, B, G, cond, coord, geom);

    if (!(0.55f < u && u < (float) sizeD[0] - 0.55f && 0.55f < v && v < (float) sizeD[1] - 0.55f))
        return;

    const int n = abs(coord[3]);
    float u_tmp = u - 0.5f, v_tmp = v - 0.5f;
    int intU = floor(u_tmp), intV = floor(v_tmp);
    float c1 = (1.0f - (u_tmp - (float) intU)) * (v_tmp - (float) intV), c2 =
            (u_tmp - (float) intU) * (v_tmp - (float) intV),
            c3 = (u_tmp - (float) intU) * (1.0f - (v_tmp - (float) intV)), c4 =
            (1.0f - (u_tmp - (float) intU)) * (1.0f - (v_tmp - (float) intV));

    for (int i = 0; i < NUM_BASIS_VECTOR; i++) {
        // calculate immutable geometry
        // add scattering coefficient (read paper)
        // B->beam direction unit vector (src2voxel)
        // S->scattering base vector
        // G->grating sensivity vector
        // v_km = (|B_m x S_k|<S_k*G>)^2
        Vector3f S(basisVector[3 * i + 0], basisVector[3 * i + 1], basisVector[3 * i + 2]);
        float vkm = B.cross(S).norm2() * abs(S * G);
        //float vkm = abs(S * G);

        const int idxVoxel = coord[0] + sizeV[0] * coord[2] + i * (sizeV[0] * sizeV[1]);
        const float backForward = vkm * vkm * c1 * devProj[intU + sizeD[0] * (intV + 1) + sizeD[0] * sizeD[1] * n] +
                                  vkm * vkm * c2 *
                                  devProj[(intU + 1) + sizeD[0] * (intV + 1) + sizeD[0] * sizeD[1] * n] +
                                  vkm * vkm * c3 * devProj[(intU + 1) + sizeD[0] * intV + sizeD[0] * sizeD[1] * n] +
                                  vkm * vkm * c4 * devProj[intU + sizeD[0] * intV + sizeD[0] * sizeD[1] * n];

        devVoxelFactor[idxVoxel] += (vkm * vkm);
        devVoxelTmp[idxVoxel] += backForward;
    }
}

__device__ void
rayCasting(float &u, float &v, Vector3f &B, Vector3f &G, int cond, const int coord[4],
           const Geometry &geom) {

    const int n = coord[3];
    int sizeV[3] = {geom.voxel, geom.voxel, geom.voxel};
    int sizeD[3] = {geom.detect, geom.detect, geom.nProj};

    const float theta = 2.0f * (float) M_PI * (float) n / (float) sizeD[2];
    Vector3f offset(INIT_OFFSET[3 * cond + 0], INIT_OFFSET[3 * cond + 1], INIT_OFFSET[3 * cond + 2]);

    // need to modify
    // need multiply Rotate matrix (axis and rotation geom) to vecSod
    Matrix3f Rotate(cosf(theta), -sinf(theta), 0.0f, sinf(theta), cosf(theta), 0.0f, 0.0f, 0.0f, 1.0f);
    // printf("%lf\n", elemR[0]);
    Matrix3f condR(elemR[9 * cond + 0], elemR[9 * cond + 1], elemR[9 * cond + 2],
                   elemR[9 * cond + 3], elemR[9 * cond + 4], elemR[9 * cond + 5],
                   elemR[9 * cond + 6], elemR[9 * cond + 7], elemR[9 * cond + 8]);
    Vector3f t(elemT[3 * cond + 0], elemT[3 * cond + 1], elemT[3 * cond + 2]);

    Rotate = condR * Rotate; // no need
    offset = Rotate * offset;
    Vector3f origin2src(0.0f, geom.sod, 0.0f);
    Vector3f baseU(1.0f, 0.0f, 0.0f);
    Vector3f baseV(0.0f, 0.0f, 1.0f); // 0, 0, -1 is correct

    // this origin is rotation center
    origin2src = Rotate * origin2src;

    Vector3f origin2voxel(
            (2.0f * (float) coord[0] - (float) sizeV[0] + 1.0f) * 0.5f * geom.voxSize - offset[0] - t[0], // -R * offset
            (2.0f * (float) coord[1] - (float) sizeV[1] + 1.0f) * 0.5f * geom.voxSize - offset[1] - t[1],
            (2.0f * (float) coord[2] - (float) sizeV[2] + 1.0f) * 0.5f * geom.voxSize - offset[2] - t[2]);

    // Source to voxel
    Vector3f src2voxel(origin2voxel[0] - origin2src[0],
                       origin2voxel[1] - origin2src[1],
                       origin2voxel[2] - origin2src[2]);

    // src2voxel and plane that have vecSod norm vector
    // p = s + t*d (vector p is on the plane, s is vecSod, d is src2voxel)
    const float coeff = -(origin2src * origin2src) / (origin2src * src2voxel); // -(n * s) / (n * v)
    Vector3f p = origin2src + coeff * src2voxel;

    u = (p * (Rotate * baseU)) * (geom.sdd / geom.sod) / geom.detSize + 0.5f * (float) (sizeD[0]);
    v = (p * (Rotate * baseV)) * (geom.sdd / geom.sod) / geom.detSize + 0.5f * (float) (sizeD[1]);

    B = src2voxel;
    B.normalize();
    G = Rotate * Vector3f(0.0f, 0.0f, 1.0f);
}

__global__ void setup_rand(hiprandState *state, int num_thread, int y) {
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int z = blockIdx.y * blockDim.y + threadIdx.y;
    hiprand_init(1234, z * num_thread + x, 0, &state[z * num_thread + x]);
}



