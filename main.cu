#include "hip/hip_runtime.h"
#include <iostream>
#include <chrono>
#include <volume.h>
#include <params.h>
#include <geometry.h>
#include <reconstruct.cuh>
#include <poisson_cpu.h>

int main() {
    std::string nametag = "phaseCT";
    init_params(nametag);

    Volume<float> sinogram[NUM_PROJ_COND];
    for (auto &e: sinogram)
        e = Volume<float>(NUM_DETECT_U, NUM_DETECT_V, NUM_PROJ);

    // ground truth
    Volume<float> ct[NUM_BASIS_VECTOR];
    Volume<float> md[3];
    for (auto &e: ct)
        e = Volume<float>(NUM_VOXEL, NUM_VOXEL, NUM_VOXEL);
    for (auto &e: md)
        e = Volume<float>(NUM_VOXEL, NUM_VOXEL, NUM_VOXEL);

    Geometry geom(SRC_DETECT_DISTANCE, SRC_OBJ_DISTANCE, DETECTOR_SIZE, NUM_VOXEL, NUM_DETECT_U, NUM_PROJ);

    // load sinogram (relative path)
    for (int i = 0; i < NUM_PROJ_COND; i++) {
        std::string loadfilePath = PROJ_PATH + std::to_string(LOAD_INDEX[i]) + "_" + std::to_string(NUM_DETECT_U)
                + "x" + std::to_string(NUM_DETECT_V) + "x" + std::to_string(NUM_PROJ) + ".raw";

        sinogram[i].load(loadfilePath, NUM_DETECT_U, NUM_DETECT_V, NUM_PROJ);
        sinogram[i].forEach([](float value) -> float { if (value < 0.0) return 1e-8; else return value; });
    }

    Method method = Method::MLEM;
    if (method == Method::MLEM) {
        for (auto &e: ct) {
            e.forEach([](float value) -> float { return 0.01; });
        }
    }

    for (int i = 0; i < NUM_BASIS_VECTOR; i++) {
        std::string loadfilePath = "../volume_bin/nut/sc_os_art_norm" + std::to_string(i + 1) + "_" +
                                   std::to_string(NUM_VOXEL) + "x" + std::to_string(NUM_VOXEL) + "x" +
                                   std::to_string(NUM_VOXEL) + ".raw";

        // ct[i].load(loadfilePath, NUM_VOXEL, NUM_VOXEL, NUM_VOXEL);
        // ct[i].forEach([](float value) -> float { return value * value; });
    }

    // measure clock
    std::chrono::system_clock::time_point start, end;
    start = std::chrono::system_clock::now();

    // main function
    // XTT::newReconstruct(sinogram, ct, md, geom, 40, 1, 30, Rotate::CW, Method::ART, 1e-2);
    // XTT::reconstruct(sinogram, ct, md, geom, 5, 5, Rotate::CW, method, 1e-3);
    // XTT::orthReconstruct(sinogram, ct, md, geom, 15, 15, 5, Rotate::CW, method, 1e-1);
    // XTT::orthTwiceReconstruct(sinogram, ct, md, geom, 4, 10, 5, Rotate::CW, method, 1e-1);
    // IR::reconstruct(sinogram, ct, geom, 6, 5, Rotate::CW, method, 0.01);

    // FDK::gradReconstruct(sinogram, ct, geom, Rotate::CW);
    IR::gradReconstruct(sinogram, ct, geom, 40, 5, Rotate::CW, Method::ART, 2e-2);
    // FDK::reconstruct(sinogram, ct, geom, Rotate::CW);
    // forwardProjOnly(sinogram, ct, geom, Rotate::CW);
    // forwardProjFiber(sinogram, ct, md, Rotate::CW, geom);

    end = std::chrono::system_clock::now();
    double time = static_cast<double>(std::chrono::duration_cast<std::chrono::microseconds>(end - start).count() /
                                      (1000.0 * 1000.0));
    std::cout << "\ntime: " << time << " (s)" << std::endl;

    // save sinogram
    for (int i = 0; i < NUM_PROJ_COND; i++) {
        std::string savefilePathProj =
                "../proj_raw_bin/simulation/proj_13axis_+x+y+z" + std::to_string(i + 1) + "_" + std::to_string(NUM_DETECT_U)
                + "x" + std::to_string(NUM_DETECT_V) + "x" + std::to_string(NUM_PROJ) + ".raw";
        // sinogram[i].save(savefilePathProj);
    }

    // save ct volume
    for (int i = 0; i < NUM_BASIS_VECTOR; i++) {
        std::string savefilePathCT =
                VOLUME_PATH + std::to_string(i + 1) + "_" + std::to_string(NUM_VOXEL) + "x"
                + std::to_string(NUM_VOXEL) + "x" + std::to_string(NUM_VOXEL) + ".raw";
        ct[i].save(savefilePathCT);
    }

    // save direction volume
    for (int i = 0; i < 3; i++) {
        std::string savefilePathCT =
                DIRECTION_PATH + std::to_string(i + 1) + "_" + std::to_string(NUM_VOXEL) + "x" +
                std::to_string(NUM_VOXEL) + "x" + std::to_string(NUM_VOXEL) + ".raw";
        // md[i].save(savefilePathCT);
    }

    return 0;
}


