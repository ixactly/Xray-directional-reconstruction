#include "hip/hip_runtime.h"
#include <iostream>
#include <chrono>
#include <volume.h>
#include <params.h>
#include <geometry.h>
#include <reconstruct.cuh>
#include <pca.cuh>

int main() {
    std::string nametag = "cfrp_7d_13rot";
    init_params(nametag);
    Volume<float> sinogram[NUM_PROJ_COND];
    for (auto &e: sinogram)
        e = Volume<float>(NUM_DETECT_U, NUM_DETECT_V, NUM_PROJ);

    // ground truth
    Volume<float> ct[NUM_BASIS_VECTOR];
    Volume<float> md[3];
    Volume<float> angle[2];
    for (auto &e: ct)
        e = Volume<float>(NUM_VOXEL, NUM_VOXEL, NUM_VOXEL);
    for (auto &e: md)
        e = Volume<float>(NUM_VOXEL, NUM_VOXEL, NUM_VOXEL);
    for (auto &e: angle)
        e = Volume<float>(NUM_VOXEL, NUM_VOXEL, NUM_VOXEL);

    Geometry geom(SRC_DETECT_DISTANCE, SRC_OBJ_DISTANCE, DETECTOR_SIZE, NUM_VOXEL, NUM_DETECT_U, NUM_PROJ);

    // load sinogram (relative path)
    for (int i = 0; i < NUM_PROJ_COND; i++) {
        std::string loadfilePath = PROJ_PATH + std::to_string(LOAD_INDEX[i]) + "_" + std::to_string(NUM_DETECT_U)
                + "x" + std::to_string(NUM_DETECT_V) + "x" + std::to_string(NUM_PROJ) + ".raw";

        sinogram[i].load(loadfilePath, NUM_DETECT_U, NUM_DETECT_V, NUM_PROJ);
        sinogram[i].forEach([](float value) -> float { if (value < 0.0) return 1e-8; else return value; });
    }

    Method method = Method::MLEM;
    if (method == Method::MLEM) {
        for (auto &e: ct) {
            e.forEach([](float value) -> float { return 0.01; });
        }
    }

    for (int i = 0; i < NUM_BASIS_VECTOR; i++) {
        std::string loadfilePath = "../volume_bin/nut/sc_os_art_norm" + std::to_string(i + 1) + "_" +
                                   std::to_string(NUM_VOXEL) + "x" + std::to_string(NUM_VOXEL) + "x" +
                                   std::to_string(NUM_VOXEL) + ".raw";

        // ct[i].load(loadfilePath, NUM_VOXEL, NUM_VOXEL, NUM_VOXEL);
        // ct[i].forEach([](float value) -> float { return value * value; });
    }

    // measure clock
    std::chrono::system_clock::time_point start, end;
    start = std::chrono::system_clock::now();

    // main function
    // XTT::newReconstruct(sinogram, ct, md, geom, 40, 1, 30, Rotate::CW, Method::ART, 1e-2);
    // XTT::reconstruct(sinogram, ct, md, geom, 50, 5, Rotate::CW, method, 1e-3);
    // XTT::orthTwiceReconstruct(sinogram, ct, md, geom, 5, 20, 4, Rotate::CW, method, 1e-1);
    XTT::circleEstReconstruct(sinogram, ct, md, geom, 3, 12, 4, Rotate::CW, method, 1e-1);
    // IR::reconstruct(sinogram, ct, geom, 10, 5, Rotate::CW, method, 0.01);
    // FDK::hilbertReconstruct(sinogram, ct, geom, Rotate::CW);
    // FDK::gradReconstruct(sinogram, ct, geom, Rotate::CW);
    // IR::gradReconstruct(sinogram, ct, geom, 100, 5, Rotate::CW, Method::ART, 8e-2);
    // FDK::reconstruct(sinogram, ct, geom, Rotate::CW);
    // forwardProjOnly(sinogram, ct, geom, Rotate::CW);
    // forwardProjFiber(sinogram, ct, md, Rotate::CW, geom);

    end = std::chrono::system_clock::now();
    double time = static_cast<double>(std::chrono::duration_cast<std::chrono::microseconds>(end - start).count() /
                                      (1000.0 * 1000.0));
    std::cout << "\ntime: " << time << " (s)" << std::endl;

    // save sinogram
    for (int i = 0; i < NUM_PROJ_COND; i++) {
        std::string savefilePathProj =
                VOLUME_PATH + "_proj" + std::to_string(i + 1) + "_" + std::to_string(NUM_DETECT_U) + "x"
                + std::to_string(NUM_DETECT_V) + "x" + std::to_string(NUM_PROJ) + ".raw";
        // sinogram[i].save(savefilePathProj);
    }

    // save ct volume
    for (int i = 0; i < NUM_BASIS_VECTOR; i++) {
        std::string savefilePathCT =
                VOLUME_PATH + std::to_string(i + 1) + "_" + std::to_string(NUM_VOXEL) + "x"
                + std::to_string(NUM_VOXEL) + "x" + std::to_string(NUM_VOXEL) + ".raw";
        ct[i].save(savefilePathCT);
    }

    // save direction volume
    for (int i = 0; i < 3; i++) {
        std::string savefilePathCT =
                DIRECTION_PATH + std::to_string(i + 1) + "_" + std::to_string(NUM_VOXEL) + "x" +
                std::to_string(NUM_VOXEL) + "x" + std::to_string(NUM_VOXEL) + ".raw";
        md[i].save(savefilePathCT);
    }

    /*
    calcAngleFromMD(md, angle, NUM_VOXEL, NUM_VOXEL, NUM_VOXEL);
    angle[0].save(DIRECTION_PATH + "_phi_" + std::to_string(NUM_VOXEL) + "x" +
                  std::to_string(NUM_VOXEL) + "x" + std::to_string(NUM_VOXEL) + ".raw");
    angle[1].save(DIRECTION_PATH + "_theta_" + std::to_string(NUM_VOXEL) + "x" +
                  std::to_string(NUM_VOXEL) + "x" + std::to_string(NUM_VOXEL) + ".raw");
    */
    return 0;
}


