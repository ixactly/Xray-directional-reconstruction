#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <random>
#include <chrono>
#include "ir/Volume.h"
#include "ir/Params.h"
#include "ir/Geometry.h"
#include "ir/mlem.cuh"


int main() {

    Volume<float> sinogram(NUM_DETECT_U, NUM_DETECT_V, NUM_PROJ);
    // ground truth
    Volume<float> ct(NUM_VOXEL, NUM_VOXEL, NUM_VOXEL);
    GeometryCUDA geom(SRC_DETECT_DISTANCE, SRC_OBJ_DISTANCE, DETECTOR_SIZE);
    // sinogram.load("../volume_bin/yukiphantom_float_1024x1024x1000.raw", NUM_DETECT_U, NUM_DETECT_V, NUM_PROJ);
    // ct.load("../volume_bin/yuki_recon2-128x128x128.raw", NUM_VOXEL, NUM_VOXEL, NUM_VOXEL);

    for (int i = NUM_VOXEL / 3; i < NUM_VOXEL * 2 / 3 + 1; i++) {
        for (int j = NUM_VOXEL / 3; j < NUM_VOXEL * 2 / 3 + 1; j++) {
            for (int k = NUM_VOXEL / 3; k < NUM_VOXEL * 2 / 3 + 1; k++) {
                ct(i, j, k) = 1.0;
            }
        }
    }

    // measure clock
    std::chrono::system_clock::time_point start, end;
    start = std::chrono::system_clock::now();

    // main function
    // mlem.forwardproj(sinogram, ctGT, geom, Rotate::CCW);
    bool rotate = true;
    reconstruct(sinogram, ct, geom, 1, 50, rotate);

    end = std::chrono::system_clock::now();
    double time = static_cast<double>(std::chrono::duration_cast<std::chrono::microseconds>(end - start).count() /
                                      (1000.0 * 1000.0));
    std::cout << "\n time: " << time << " (s)" << std::endl;


    std::string savefilePath =
            "../volume_bin/cube_phantom_cuda-" + std::to_string(NUM_DETECT_U) + "x" + std::to_string(NUM_DETECT_V) + "x" +
            std::to_string(NUM_PROJ) + ".raw";
    sinogram.save(savefilePath);

    /*
    std::string savefilePath =
            "../volume_bin/tmp_cuda-" + std::to_string(NUM_VOXEL) + "x" +
            std::to_string(NUM_VOXEL) + "x" + std::to_string(NUM_VOXEL) + ".raw";
    */
    // ct.save(savefilePath);

    return 0;
}


