#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <random>
#include <chrono>
#include <Volume.h>
#include <Params.h>
#include <Geometry.h>
#include <reconstruct.cuh>

int main() {

    Volume<float> sinogram[NUM_PROJ_COND];
    for (auto &e: sinogram)
        e = Volume<float>(NUM_DETECT_U, NUM_DETECT_V, NUM_PROJ);

    // ground truth
    Volume<float> ct[NUM_BASIS_VECTOR];
    Volume<float> md[3];
    for (auto &e: ct)
        e = Volume<float>(NUM_VOXEL, NUM_VOXEL, NUM_VOXEL);
    for (auto &e: md)
        e = Volume<float>(NUM_VOXEL, NUM_VOXEL, NUM_VOXEL);

    Geometry geom(SRC_DETECT_DISTANCE, SRC_OBJ_DISTANCE, DETECTOR_SIZE, NUM_VOXEL, NUM_DETECT_U, NUM_PROJ);

    // load sinogram (relative path)
    for (int i = 0; i < NUM_PROJ_COND; i++) {
        // std::string loadfilePath = "../proj_raw_bin/cfrp_xyz7/SC/CFRP_XYZ7_AXIS" + std::to_string(i + 1) + "_" +
        std::string loadfilePath = "../proj_raw_bin/gfrp_a/SC/gfrp_a_ct" + std::to_string(i + 1) + "_" +
                                   std::to_string(NUM_DETECT_U) + "x" + std::to_string(NUM_DETECT_V) + "x" +
                                   std::to_string(NUM_PROJ) + ".raw";

        sinogram[i].load(loadfilePath, NUM_DETECT_U, NUM_DETECT_V, NUM_PROJ);
        sinogram[i].forEach([](float value) -> float { if (value < 0.0) return 1e-5; else return value; });
    }

    // load volume
    Method method = Method::MLEM;

    if (method == Method::MLEM) {
        for (auto &e : ct) {
            // e.forEach([](float value) -> float { return 0.01; });
        }
    }

    for (int i = 0; i < NUM_BASIS_VECTOR; i++) {
        std::string loadfilePath = "../volume_bin/gfrp_a/xtt_" + std::to_string(i + 1) + "_" +
                                   std::to_string(NUM_VOXEL) + "x" + std::to_string(NUM_VOXEL) + "x" +
                                   std::to_string(NUM_VOXEL) + ".raw";

        ct[i].load(loadfilePath, NUM_VOXEL, NUM_VOXEL, NUM_VOXEL);
        ct[i].forEach([](float value) -> float { return value * value; });
    }

    // measure clock
    std::chrono::system_clock::time_point start, end;
    start = std::chrono::system_clock::now();

    // main function
    // XTT::newReconstruct(sinogram, ct, md, geom, 40, 1, 30, Rotate::CW, Method::ART, 1e-2);
    XTT::reconstruct(sinogram, ct, md, geom, 50, 6, Rotate::CW, method, 9e-3);
    // XTT::reconstruct(sinogram, ct, md, geom, 20, 1, Rotate::CW, Method::MLEM, 9e-3);
    // XTT::orthReconstruct(sinogram, ct, md, geom, 3, 5, 30, Rotate::CW, Method::MLEM, 9e-3);
    // IR::reconstruct(sinogram, ct, geom, 40, 6, Rotate::CW, Method::ART, 5e-2);
    // FDK::reconstruct(sinogram, ct, geom, Rotate::CW);
    // forwardProjOnly(sinogram, ct, geom, Rotate::CW);

    end = std::chrono::system_clock::now();
    double time = static_cast<double>(std::chrono::duration_cast<std::chrono::microseconds>(end - start).count() /
                                      (1000.0 * 1000.0));
    std::cout << "\ntime: " << time << " (s)" << std::endl;

    // save sinogram
    for (int i = 0; i < NUM_PROJ_COND; i++) {
        std::string savefilePathProj =
                "../volume_bin/gfrp_a/proj" + std::to_string(i + 1) + "_" + std::to_string(NUM_DETECT_U) + "x" +
                std::to_string(NUM_DETECT_V) + "x" + std::to_string(NUM_PROJ) + ".raw";
        sinogram[i].save(savefilePathProj);
    }

    // save ct volume
    for (int i = 0; i < NUM_BASIS_VECTOR; i++) {
        std::string savefilePathCT =
                // "../volume_bin/cfrp_xyz7_mark/xtt_cond3" + std::to_string(i + 1) + "_" +
                "../volume_bin/gfrp_a/xtt_" + std::to_string(i + 1) + "_" +
                // "../volume_bin/gfrp_a/xtt_plane" + std::to_string(i + 1) + "_" +
                std::to_string(NUM_VOXEL) + "x" +
                std::to_string(NUM_VOXEL) + "x" + std::to_string(NUM_VOXEL) + ".raw";

        ct[i].save(savefilePathCT);
    }

    // save ct volume
    for (int i = 0; i < 3; i++) {
        std::string savefilePathCT =
                "../volume_bin/gfrp_a/pca/main_direction_xtt_" + std::to_string(i + 1) + "_" +
                std::to_string(NUM_VOXEL) + "x" +
                std::to_string(NUM_VOXEL) + "x" + std::to_string(NUM_VOXEL) + ".raw";
        md[i].save(savefilePathCT);
    }
    return 0;
}


