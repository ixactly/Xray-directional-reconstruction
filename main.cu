#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <random>
#include <chrono>
#include "ir/Volume.h"
#include "ir/Params.h"
#include "ir/Geometry.h"
#include "ir/mlem.cuh"


int main() {

    Volume<float> sinogram(NUM_DETECT_U, NUM_DETECT_V, NUM_PROJ);
    // ground truth
    Volume<float> ct(NUM_VOXEL, NUM_VOXEL, NUM_VOXEL);
    GeometryCUDA geom(SRC_DETECT_DISTANCE, SRC_OBJ_DISTANCE, DETECTOR_SIZE);
    // sinogram.load("../volume_bin/cube_proj_phantom-500x500x500.raw", NUM_DETECT_U, NUM_DETECT_V, NUM_PROJ);
    sinogram.load("../volume_bin/cfrp/ATstack_1000x1000x360.raw", NUM_DETECT_U, NUM_DETECT_V, NUM_PROJ);
    sinogram.forEach([](float value) -> float { if (value < 0.0) return 0.0; else return value;});
    /*
    for (int i = NUM_VOXEL / 3; i < NUM_VOXEL * 2 / 3 + 1; i++) {
        for (int j = NUM_VOXEL / 3; j < NUM_VOXEL * 2 / 3 + 1; j++) {
            for (int k = NUM_VOXEL / 3; k < NUM_VOXEL * 2 / 3 + 1; k++) {
                ct(i, j, k) = 1.0;
            }
        }
    }
    */

    // measure clock
    std::chrono::system_clock::time_point start, end;
    start = std::chrono::system_clock::now();

    // main function

    // if u load ct, turn off initializing of fill 1.0

    ct.forEach([](float value) -> float { return 1.0; });
    /*
    std::string loadfilePath =
            "../volume_bin/cf_at_vol_epoch5-" + std::to_string(NUM_VOXEL) + "x" +
            std::to_string(NUM_VOXEL) + "x" + std::to_string(NUM_VOXEL) + ".raw";
    ct.load(loadfilePath, NUM_VOXEL, NUM_VOXEL, NUM_VOXEL);
    */

    bool rotate = true;
    reconstruct(sinogram, ct, geom, 1, 18, rotate);

    end = std::chrono::system_clock::now();
    double time = static_cast<double>(std::chrono::duration_cast<std::chrono::microseconds>(end - start).count() /
                                      (1000.0 * 1000.0));
    std::cout << "\n time: " << time << " (s)" << std::endl;

    /*
    std::string savefilePath1 =
            "../volume_bin/cube_proj_cube_epoch_one-" + std::to_string(NUM_DETECT_U) + "x" + std::to_string(NUM_DETECT_V) + "x" +
            std::to_string(NUM_PROJ) + ".raw";
    sinogram.save(savefilePath1);
    */

    std::string savefilePath =
            "../volume_bin/cf_at_vol_epoch-" + std::to_string(NUM_VOXEL) + "x" +
            std::to_string(NUM_VOXEL) + "x" + std::to_string(NUM_VOXEL) + ".raw";
    ct.save(savefilePath);

    return 0;
}


