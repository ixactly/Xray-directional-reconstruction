#include "hip/hip_runtime.h"
//
// Created by tomokimori on 22/08/30.
//
#include "Geometry.h"
#include "mlem.cuh"
#include <random>
#include <memory>
#include "Pbar.h"
#include "Params.h"
#include "Volume.h"
#include "omp.h"
#include "reconstruct.cuh"

void reconstruct(Volume<float> *sinogram, Volume<float> *voxel, const Geometry &geom, int epoch, int batch, Rotate dir,
                 IR method) {
    auto forward = (method == IR::MLEM) ? forwardProj : forwardProjXTT;
    auto backward = (method == IR::MLEM) ? backwardProj : backwardProjXTT;
    int rotation = (dir == Rotate::CW) ? 1 : -1;

    int sizeV[3] = {voxel[0].x(), voxel[0].y(), voxel[0].z()};
    int sizeD[3] = {sinogram[0].x(), sinogram[0].y(), sinogram[0].z()};
    int nProj = sizeD[2];

    // hipMalloc
    float *devSino, *devProj, *devVoxel, *devVoxelFactor, *devVoxelTmp;
    const long lenV = sizeV[0] * sizeV[1] * sizeV[2];
    const long lenD = sizeD[0] * sizeD[1] * sizeD[2];

    hipMalloc(&devSino, sizeof(float) * lenD * NUM_PROJ_COND);
    hipMalloc(&devProj, sizeof(float) * lenD * NUM_PROJ_COND); // memory can be small to subsetSize
    hipMalloc(&devVoxel, sizeof(float) * lenV * NUM_BASIS_VECTOR);
    hipMalloc(&devVoxelFactor, sizeof(float) * sizeV[0] * sizeV[1] * NUM_BASIS_VECTOR);
    hipMalloc(&devVoxelTmp, sizeof(float) * sizeV[0] * sizeV[1] * NUM_BASIS_VECTOR);

    for (int i = 0; i < NUM_PROJ_COND; i++)
        hipMemcpy(&devSino[i * lenD], sinogram[i].get(), sizeof(float) * lenD, hipMemcpyHostToDevice);
    for (int i = 0; i < NUM_BASIS_VECTOR; i++)
        hipMemcpy(&devVoxel[i * lenV], voxel[i].get(), sizeof(float) * lenV, hipMemcpyHostToDevice);

    Geometry *devGeom;
    hipMalloc(&devGeom, sizeof(Geometry));
    hipMemcpy(devGeom, &geom, sizeof(Geometry), hipMemcpyHostToDevice);

    // define blocksize
    const int blockSize = 16;
    dim3 blockV(blockSize, blockSize, 1);
    dim3 gridV((sizeV[0] + blockSize - 1) / blockSize, (sizeV[2] + blockSize - 1) / blockSize, 1);
    dim3 blockD(blockSize, blockSize, 1);
    dim3 gridD((sizeD[0] + blockSize - 1) / blockSize, (sizeD[1] + blockSize - 1) / blockSize, 1);

    // forwardProj, divide, backwardProj proj
    int subsetSize = (nProj + batch - 1) / batch;
    std::vector<int> subsetOrder(batch);
    for (int i = 0; i < batch; i++) {
        subsetOrder[i] = i;
    }

    std::vector<float> losses(epoch);

    // progress bar
    progressbar pbar(epoch * batch * (NUM_PROJ_COND * subsetSize + sizeV[1]));

    // main routine

    for (int ep = 0; ep < epoch; ep++) {
        std::mt19937_64 get_rand_mt; // fixed seed
        std::shuffle(subsetOrder.begin(), subsetOrder.end(), get_rand_mt);
        hipMemset(&loss, 0.0, sizeof(float));
        hipMemset(devProj, 0.0f, sizeof(float) * lenD * NUM_PROJ_COND);
        for (int &sub: subsetOrder) {
            // forwardProj and ratio
            for (int i = 0; i < NUM_PROJ_COND; i++) {
                for (int subOrder = 0; subOrder < subsetSize; subOrder++) {
                    int n = rotation * ((sub + batch * subOrder) % nProj);
                    // !!care!! judge from vecSod which plane we chose
                    pbar.update();

                    // forwardProj process
                    for (int y = 0; y < sizeV[1]; y++) {
                        forward<<<gridV, blockV>>>(&devProj[lenD * i], devVoxel, devGeom, i, y, n);
                        hipDeviceSynchronize();
                    }
                    // ratio process
                    projRatio<<<gridD, blockD>>>(&devProj[lenD * i], &devSino[lenD * i], devGeom, n);
                    hipDeviceSynchronize();
                }
            }

            // backwardProj process
            for (int y = 0; y < sizeV[1]; y++) {
                hipMemset(devVoxelFactor, 0, sizeof(float) * sizeV[0] * sizeV[1] * NUM_BASIS_VECTOR);
                hipMemset(devVoxelTmp, 0, sizeof(float) * sizeV[0] * sizeV[1] * NUM_BASIS_VECTOR);
                pbar.update();
                for (int i = 0; i < NUM_PROJ_COND; i++) {
                    for (int subOrder = 0; subOrder < subsetSize; subOrder++) {
                        int n = rotation * ((sub + batch * subOrder) % nProj);
                        backward<<<gridV, blockV>>>(&devProj[lenD * i], devVoxelTmp,
                                                    devVoxelFactor, devGeom, i, y, n);
                        hipDeviceSynchronize();
                    }
                }
                voxelProduct<<<gridV, blockV>>>(devVoxel, devVoxelTmp, devVoxelFactor, devGeom, y);
                hipDeviceSynchronize();
            }
        }
        loss /= static_cast<float>(NUM_DETECT_V * NUM_DETECT_U * NUM_PROJ);
        hipMemcpy(losses.data() + ep, &loss, sizeof(float), hipMemcpyDeviceToHost); // loss
    }

    for (int i = 0; i < NUM_PROJ_COND; i++)
        hipMemcpy(sinogram[i].get(), &devProj[i * lenD], sizeof(float) * lenD, hipMemcpyDeviceToHost);
    for (int i = 0; i < NUM_BASIS_VECTOR; i++)
        hipMemcpy(voxel[i].get(), &devVoxel[i * lenV], sizeof(float) * lenV, hipMemcpyDeviceToHost);

    hipFree(devProj);
    hipFree(devSino);
    hipFree(devVoxel);
    hipFree(devGeom);
    hipFree(devVoxelFactor);
    hipFree(devVoxelTmp);

    std::ofstream ofs("../python/loss.csv");
    for (auto &e: losses)
        ofs << e << ", ";
}

void compareXYZTensorVolume(Volume<float> *voxel, const Geometry &geom) {
    for (int i = 0; i < geom.voxel; i++) {
        for (int j = 0; j < geom.voxel; j++) {
            for (int k = 0; k < geom.voxel; k++) {
                float min = voxel[0](i, j, k);
                int idx = 0;
                for (int n = 1; n < NUM_BASIS_VECTOR; n++) {
                    if (min > voxel[n](i, j, k)) {
                        min = voxel[n](i, j, k);
                        idx = n;
                    }
                }
                for (int n = 0; n < NUM_BASIS_VECTOR; n++) {
                    if (n != idx) {
                        voxel[n](i, j, k) = 0.0f;
                    }
                }
            }
        }
    }
}

/*
__host__ void
reconstructDebugHost(Volume<float> &sinogram, Volume<float> &voxel, const Geometry &geom, const int epoch,
                     const int batch, bool dir) {

    printf("pass");
    CudaVolume<float> sino(sinogram);
    CudaVolume<float> vox(voxel);

    int sizeV[3] = {voxel.x(), voxel.y(), voxel.z()};
    int sizeD[3] = {sinogram.x(), sinogram.y(), sinogram.z()};
    int nProj = sizeD[2];


    // forward, divide, backwardProj proj
    int subsetSize = (nProj + batch - 1) / batch;
    std::vector<int> subsetOrder(batch);
    for (int i = 0; i < batch; i++) {
        subsetOrder[i] = i;
    }

    std::mt19937_64 get_rand_mt; // fixed seed
    std::shuffle(subsetOrder.begin(), subsetOrder.end(), get_rand_mt);

    // main routine
    for (int ep = 0; ep < epoch; ep++) {
        // forward
        for (int n = 15; n < nProj; n++) {

            // forwardProj
            for (int x = 0; x < sizeV[0]; x++) {
                for (int y = 0; y < sizeV[1]; y++) {
                    for (int z = 0; z < sizeV[2]; z++) {
                        int coord[4] = {x, y, z, n};
                        forwardXTTonDevice(coord, sino, &vox, geom);
                    }
                }
            }
        }
    }
}
 */
