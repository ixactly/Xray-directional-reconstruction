#include "hip/hip_runtime.h"
//
// Created by tomokimori on 22/07/20.
//
#include "Geometry.h"
#include "mlem.cuh"
#include <random>

template <typename T>
__device__ __host__ int sign(T val) {
    return (val > T(0)) - (val < T(0));
}

__device__ __host__ void forwardProj(const int coord[4], const int sizeD[3], const int sizeV[3], float *devSino, const float* devVoxel, const GeometryCUDA& geom) {

    // sourceとvoxel座標間の関係からdetのu, vを算出
    // detectorの中心 と 再構成領域の中心 と 光源 のz座標は一致していると仮定
    const int n = coord[3];
    const int x = coord[0], y = coord[1], z = coord[2];
    const float theta = 2.0f * M_PI * n / sizeD[2];

    float offset[3] = {0.0, 0.0, 0.0};
    float vecSod[3] = {sinf(theta) * geom.sod + offset[0], -cosf(theta) * geom.sod + offset[1], 0};

    // Source to voxel center
    float src2cent[3] = {-vecSod[0], -vecSod[1], -vecSod[2]};
    // Source to voxel
    float src2voxel[3] = {(2.0f * (float)coord[0] - (float)sizeV[0] + 1) * 0.5f * geom.voxSize + src2cent[0],
                          (2.0f * (float)coord[1] - (float)sizeV[1] + 1) * 0.5f * geom.voxSize + src2cent[1],
                          (2.0f * (float)coord[2] - (float)sizeV[2] + 1) * 0.5f * geom.voxSize + src2cent[2]};

    const double beta = acos((src2cent[0] * src2voxel[0] + src2cent[1] * src2voxel[1]) /
                            (sqrt(src2cent[0] * src2cent[0] + src2cent[1] * src2cent[1]) *
                             sqrt(src2voxel[0] * src2voxel[0] + src2voxel[1] * src2voxel[1])));
    const double gamma = atan2(src2voxel[2], sqrt(src2voxel[0]*src2voxel[0]+src2voxel[1]*src2voxel[1]));
    const int signU = sign(src2voxel[0] * src2cent[1] - src2voxel[1] * src2cent[0]);

    // src2voxel x src2cent
    // 光線がhitするdetector平面座標の算出(detectorSizeで除算して、正規化済み)
    float u = tanf(signU * beta) * geom.sdd / geom.detSize + (float)sizeD[0] * 0.5f;
    float v = tanf(gamma) * geom.sdd / cosf(beta) / geom.detSize + (float)sizeD[1] * 0.5f; // normalization

    if (!(0.5 < u && u < (float)sizeD[0] - 0.5 && 0.5 < v && v < (float)sizeD[1] - 0.5))
        return;

    float u_tmp = u - 0.5f, v_tmp = v - 0.5f;
    int intU = floor(u_tmp), intV = floor(v_tmp);
    float c1 = (1.0f - (u_tmp - intU)) * (v_tmp - intV), c2 = (u_tmp - intU) * (v_tmp - intV),
            c3 = (u_tmp - intU) * (1.0f - (v_tmp - intV)), c4 =
            (1.0f - (u_tmp - intU)) * (1.0f - (v_tmp - intV));

    const unsigned int idxVoxel = coord[0] + sizeV[0] * coord[1] + sizeV[0] * sizeV[1] * coord[2];
    /*
    atomicAdd(&devSino[intU + sizeD[0] * (intV+1) + sizeD[0] * sizeD[1] * n], c1 * devVoxel[idxVoxel]);
    atomicAdd(&devSino[(intU+1) + sizeD[0] * (intV+1) + sizeD[0] * sizeD[1] * n], c2 * devVoxel[idxVoxel]);
    atomicAdd(&devSino[(intU+1) + sizeD[0] * intV + sizeD[0] * sizeD[1] * n], c3 * devVoxel[idxVoxel]);
    atomicAdd(&devSino[intU + sizeD[0] * intV + sizeD[0] * sizeD[1] * n], c4 * devVoxel[idxVoxel]);
    */

    devSino[intU + sizeD[0] * (intV+1) + sizeD[0] * sizeD[1] * n] += c1 * devVoxel[idxVoxel];
    devSino[(intU+1) + sizeD[0] * (intV+1) + sizeD[0] * sizeD[1] * n] += c2 * devVoxel[idxVoxel];
    devSino[(intU+1) + sizeD[0] * intV + sizeD[0] * sizeD[1] * n] += c3 * devVoxel[idxVoxel];
    devSino[intU + sizeD[0] * intV + sizeD[0] * sizeD[1] * n] += c4 * devVoxel[idxVoxel];
}

__global__ void printKernel() {
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int z = blockIdx.y * blockDim.y + threadIdx.y;
    printf("pass kernel func\n");
}

__global__ void xzPlaneForward(const int* sizeD, const int* sizeV, float *devSino, const float *devVoxel, GeometryCUDA *geom,
                           const int y, const int n) {
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int z = blockIdx.y * blockDim.y + threadIdx.y;

    const int coord[4] = {x, y, z, n};
    forwardProj(coord, sizeD, sizeV, devSino, devVoxel, *geom);
    // printf("pass\n");
}

__global__ void voxelOne(const int* sizeD, const int* sizeV, float *devSino, float *devVoxel, GeometryCUDA *geom,
                               const int y, const int n) {
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int z = blockIdx.y * blockDim.y + threadIdx.y;

    const int coord[4] = {x, y, z, n};
    if (x >= sizeV[0] || y >= sizeV[1] || z >= sizeV[2]) {
        return;
    }
    // printf("%d %d %d\n", x, y, z);
    if (x <= sizeV[0] / 3 && y <= sizeV[1] / 3 && z <= sizeV[2])
    devVoxel[x + sizeV[0] * y + sizeV[0] * sizeV[1] * z] = 1.0f;
    // printf("pass\n");
}

void reconstruct(Volume<float> &sinogram, Volume<float> &voxel, const GeometryCUDA &geom, const int epoch,
                 const int batch, bool dir){
    int sizeV[3] = {voxel.x(), voxel.y(), voxel.z()};
    int sizeD[3] = {sinogram.x(), sinogram.y(), sinogram.z()};
    int nProj = sizeD[2];

    float *devSino, *devVoxel;
    GeometryCUDA *devGeom;
    int *devV, *devD;

    hipMalloc(&devSino, sizeof(float) * sizeD[0] * sizeD[1] * sizeD[2]);
    hipMalloc(&devVoxel, sizeof(float) * sizeV[0] * sizeV[1] * sizeV[2]);
    hipMalloc(&devGeom, sizeof(GeometryCUDA));
    hipMalloc(&devV, sizeof(int) * 3);
    hipMalloc(&devD, sizeof(int) * 3);

    hipMemcpy(devSino, sinogram.getPtr(), sizeof(float) * sizeD[0] * sizeD[1] * sizeD[2], hipMemcpyHostToDevice);
    hipMemcpy(devVoxel, voxel.getPtr(), sizeof(float) * sizeV[0] * sizeV[1] * sizeV[2], hipMemcpyHostToDevice);
    hipMemcpy(devGeom, &geom, sizeof(GeometryCUDA), hipMemcpyHostToDevice);
    hipMemcpy(devV, sizeV, sizeof(int) * 3, hipMemcpyHostToDevice);
    hipMemcpy(devD, sizeD, sizeof(int) * 3, hipMemcpyHostToDevice);

    const int blockSize = 8;
    dim3 block(blockSize, blockSize, 1);
    dim3 grid((sizeV[0] + blockSize - 1) / blockSize, (sizeV[0] + blockSize - 1) / blockSize, 1);

    // forward, divide, backward proj
    int subsetSize = (nProj + batch - 1) / batch;
    std::vector<int> subsetOrder(batch);
    for (int i = 0; i < batch; i++) {
        subsetOrder[i] = i;
    }

    std::mt19937_64 get_rand_mt; // fixed seed
    std::shuffle(subsetOrder.begin(), subsetOrder.end(), get_rand_mt);

    // progress bar
    // progressbar pbar(epoch * nProj);

    // main routine
    for (int ep = 0; ep < epoch; ep++) {
        for (int &sub: subsetOrder) {

            for (int subOrder = 0; subOrder < subsetSize; subOrder++) {
                // pbar.update();
                int n = (sub + batch * subOrder) % nProj;
                /*
                for (int y = 0; y < sizeV[1]; y++) {
                    xzPlaneForward<<<grid, block>>>(devD, devV, devSino, devVoxel, devGeom, y, n);
                    // voxelOne<<<grid, block>>>(devD, devV, devSino, devVoxel, devGeom, y, n);
                    // printKernel<<<grid, block>>>();
                    hipDeviceSynchronize();
                }
                 */
                for (int x = 0; x < sizeV[0]; x++) {
                    for (int y = 0; y < sizeV[1]; y++) {
                        for (int z = 0; z < sizeV[2]; z++) {
                            const int coord[4] = {z, y, x, 0};
                            forwardProj(coord, sizeD, sizeV, sinogram.getPtr(), voxel.getPtr(), geom);
                        }
                    }
                }
            }
        }
    }

    // hipMemcpy(voxel.getPtr(), devVoxel, sizeof(float) * sizeV[0] * sizeV[1] * sizeV[2], hipMemcpyDeviceToHost);
    // hipMemcpy(sinogram.getPtr(), devSino, sizeof(float) * sizeD[0] * sizeD[1] * sizeD[2], hipMemcpyDeviceToHost);

    hipFree(devSino);
    hipFree(devVoxel);
    hipFree(devGeom);
    hipFree(devV);
    hipFree(devD);
}

