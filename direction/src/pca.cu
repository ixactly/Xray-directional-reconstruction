#include "hip/hip_runtime.h"
//
// Created by tomokimori on 22/10/24.
//

#include "pca.cuh"
#include "Params.h"
#include "Vec.h"
#include <Eigen/Dense>
#include <Eigen/LU>


void calcEigenVector(const Volume<float> *ct, Volume<float> *md, Volume<float> *evalue, int x, int y, int z) {

    Eigen::Matrix3f varMatrix;
    varMatrix << 0, 0, 0,
            0, 0, 0,
            0, 0, 0;

    // calclate VarianceCovariance Matrix
    float mu_mean = 0.0f;
    for (int i = 0; i < NUM_BASIS_VECTOR; i++) {
        float mu = (ct[i])(x, y, z);
        mu_mean += mu;
        Eigen::Matrix<float, 3, 1> scat;
        scat << mu * basisVector[3 * i + 0], mu * basisVector[3 * i + 1], mu * basisVector[3 * i + 2];
        // std::cout << basisVector[3 * i + 1] << std::endl;
        varMatrix += scat * scat.transpose();
    }

    mu_mean /= static_cast<float>(NUM_BASIS_VECTOR);
    varMatrix /= static_cast<float>(NUM_BASIS_VECTOR);

    Eigen::SelfAdjointEigenSolver<Eigen::Matrix3f> ES(varMatrix);

    Eigen::Vector3f values = ES.eigenvalues();
    Eigen::Matrix3f vectors = ES.eigenvectors();

    // (temporary) pick up minimum eigenvector, then normalization
    Eigen::Vector3f min = vectors.col(0).normalized();
    float sign = (min.z() >= 0) ? 1.0 : -1.0;

    md[0](x, y, z) = sign * mu_mean * min.x();
    md[1](x, y, z) = sign * mu_mean * min.y();
    md[2](x, y, z) = sign * mu_mean * min.z();

    /*
    md[0](x, y, z) = min.x();
    md[1](x, y, z) = min.y();
    md[2](x, y, z) = min.z();
    */

    evalue[0](x, y, z) = values(0) / (values(0) + values(1) + values(2));
    evalue[1](x, y, z) = values(1) / (values(0) + values(1) + values(2));
    evalue[2](x, y, z) = values(2) / (values(0) + values(1) + values(2));
    /*
    if ((125 < x && x < 135) && y == 189 && z == 171) {
        std::cout << std::endl << varMatrix << std::endl ;
        std::cout << "eigenvalue1: " << values(0) << ", vector1 x: " << vectors.col(0).x() << ", y: " << vectors.col(0).y() << ", z: " << vectors.col(0).z() << std::endl;
        std::cout << "eigenvalue2: " << values(1) << ", vector2 x: " << vectors.col(1).x() << ", y: " << vectors.col(1).y() << ", z: " << vectors.col(1).z() << std::endl;
        std::cout << "eigenvalue3: " << values(2) << ", vector3 x: " << vectors.col(2).x() << ", y: " << vectors.col(2).y() << ", z: " << vectors.col(2).z() << std::endl << std::endl;
    }*/

    // std::cout << md[0](x, y, z) << " " << md[1](x, y, z) << " " << md[2](x, y, z) << std::endl;
    // std::cout << varMatrix << std::endl;
}

void calcPartsAngle(const Volume<float> md[3], Volume<float> angle[2], int size_x, int size_y, int size_z) {
    for (int x = 0; x < size_x; x++) {
        for (int y = 0; y < size_y; y++) {
            for (int z = 0; z < size_z; z++) {
                angle[0](x, y, z) = std::atan2(md[0](x, y, z), -md[1](x, y, z));
                angle[1](x, y, z) = std::atan2(md[2](x, y, z),
                                  std::sqrt(md[1](x, y, z) * md[1](x, y, z) + md[0](x, y, z) * md[0](x, y, z)));
            }
        }
    }
}

void rodriguesRotation(double x, double y, double z, double theta) {
    Eigen::Matrix3d rot1;
    Eigen::Matrix3d rot2;

    double n_x = x / std::sqrt(x * x + y * y + z * z);
    double n_y = y / std::sqrt(x * x + y * y + z * z);
    double n_z = z / std::sqrt(x * x + y * y + z * z);

    Eigen::MatrixXd basis(3, NUM_BASIS_VECTOR);
    rot1 << n_x * n_x, n_x * n_y, n_x * n_z,
            n_x * n_y, n_y * n_y, n_y * n_z,
            n_x * n_z, n_y * n_z, n_z * n_z;
    rot2 << std::cos(theta), -n_z * std::sin(theta), n_y * std::sin(theta),
            n_z * std::sin(theta), std::cos(theta), -n_x * std::sin(theta),
            -n_y * std::sin(theta), n_x * std::sin(theta), std::cos(theta);

    for (int i = 0; i < NUM_BASIS_VECTOR; i++) {
        basis(0, i) = basisVector[3 * i + 0];
        basis(1, i) = basisVector[3 * i + 1];
        basis(2, i) = basisVector[3 * i + 2];
    }

    Eigen::MatrixXd vec = ((1 - std::cos(theta)) * rot1 + rot2) * basis;
    std::cout << vec.transpose() << std::endl;
}
